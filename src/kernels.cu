#include "hip/hip_runtime.h"
#include "kernels.h"
#include "deviceFunc.cu"
#include "base.c"
#include <stdio.h>

/* wall force */
__global__ void calcWallForces (float *fwallx, float *fwally, float *ftmagsum, float *D, int *Injured, float *X, float *Y, wpoint *WP, float *VX, float *VY, parameter *para, int N, int Nw) {
	
	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
		
	int iw;
	float R = para->R; 
	int InjurySwitch = para->InjurySwitch;
	
	int can_see;
	float tmpr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay ;
	
	if (i <= N) {
        for(iw=0; iw<Nw; iw++) {
			
            WallParticleRelation(iw,i,&tmpr,&can_see,Y[i],X[i],WP,para);
			
            if((can_see==1)&&(tmpr<=R)) {

                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* psychological force */
				
                WallPsychForce(iw,i,tmpr,&tmp_fpsx,&tmp_fpsy, D[i], para);
                /* Young and tangential forces */
                if(tmpr<=0.5*D[i]) {
                    
					WallYoungForce(iw,i,tmpr,&tmp_fyox,&tmp_fyoy, D[i], para);
				
                    WallTangForce_FS1(iw,i,tmpr, &tmp_ftax, &tmp_ftay, D[i], VX[i], VY[i], para);

                }
                /* summing wall forces */
                if(Injured[i]==0) {
                    fwallx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fwally[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fwallx[i] += tmp_fyox + tmp_ftax;
                    fwally[i] += tmp_fyoy + tmp_ftay;
                }

                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }

                // measuring x component of touching force exerted on walls left and right from exit 
				/* lasse ich erstmal weg
                if((iw==1)||(iw==7)) {
                    FW_x -= tmp_fyox + tmp_ftax;
                }
				*/
            }
        }
    }
}

__global__ void calcWPointForces (float *fwpointx, float *fwpointy, float *ftmagsum, float *D, int *Injured, float *X, float *Y, wpoint *WP, float *VX, float *VY, parameter *para, int N, int Nwp) {
	
	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
	float R = para->R; 
	int InjurySwitch = para->InjurySwitch;
	int iwp;
	int can_see;
	float tmpr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay;
	
	if (i <=  N) {
        for(iwp=0; iwp<Nwp; iwp++) {
						
            WPointParticleRelation(iwp,i,&tmpr,&can_see, Y[i], X[i], WP);
            if((can_see==1)&&(tmpr<=R)) {

                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* computing forces */
                WPointPsychForce(iwp,i,tmpr,&tmp_fpsx,&tmp_fpsy, X[i], Y[i], D[i], WP[iwp], para);
                if(tmpr<=0.5*D[i]) {
                    
					WPointYoungForce(iwp,i,tmpr,&tmp_fyox,&tmp_fyoy, X[i], Y[i], D[i], WP[iwp], para);

					WPointTangForce_FS1(iwp,i,tmpr,&tmp_ftax,&tmp_ftay, X[i], Y[i], D[i], VX[i], VY[i], WP[iwp], para);
                }

                /* summing forces */
                if(Injured[i]==0) {
                    fwpointx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fwpointy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fwpointx[i] += tmp_fyox + tmp_ftax;
                    fwpointy[i] += tmp_fyoy + tmp_ftay;
                }

                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }

                // measuring x component of touching force exerted on walls left and right from exit
                // erstmal rausgenommen   
				//
                // if((iwp==0)||(iwp==3)) {
                //    FW_x -= tmp_fyox + tmp_ftax;
                // }

            }
        }
    }
}

__global__ void calcColumnForces (float *fcolx, float *fcoly, float *ftmagsum,float *D, int *Injured, float *X, float *Y, float *VX, float *VY, parameter *para, int N) {
    
	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	int InjurySwitch = para->InjurySwitch;
	int ColumnSwitch = para->ColumnSwitch;
	float ColumnCenterX = para-> ColumnCenterX;
	float ColumnCenterY = para-> ColumnCenterY;
	float ColumnD = para-> ColumnD;
	
	float A = para-> A;
	float B = para-> B;
	float C_Young = para-> C_Young;
	float Kappa = para-> Kappa;
	
	float R = para -> R;
		
	// lokale Variable
	float tmprsqr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay, rx, ry, f_over_r, scal_prod_over_rsqr, tmpr;
	
	/* 1.4
     * column
     */
	if (i <= N) {
	
		switch(ColumnSwitch) {
		default:
		case 0: {
				// for(i=0; i<N; i++) {
					fcolx[i] = fcoly[i] = 0.0;
				// }
				 break;
			}
		case 1: {
				// for(i=0; i<N; i++) {
					tmprsqr = SQR(X[i]-ColumnCenterX)+SQR(Y[i]-ColumnCenterY);
					if(tmprsqr<=SQR(R)) {
						tmpr=sqrt(tmprsqr);

						/* init */
						tmp_fpsx = tmp_fpsy = 0.0;
						tmp_fyox = tmp_fyoy = 0.0;
						tmp_ftax = tmp_ftay = 0.0;

						/* computing forces */
						/* psychological */
						f_over_r = A * exp(-(tmpr-0.5*(D[i]+ColumnD))/B) / tmpr;
						tmp_fpsx = (X[i]-ColumnCenterX) * f_over_r;
						tmp_fpsy = (Y[i]-ColumnCenterY) * f_over_r;
						/* touching */
						if(tmpr<=0.5*(D[i]+ColumnD)) {
							/* Young */
							f_over_r = 2.0*C_Young*(0.5*(D[i]+ColumnD)-tmpr) / tmpr;
							tmp_fyox = (X[i]-ColumnCenterX) * f_over_r;
							tmp_fyoy = (Y[i]-ColumnCenterY) * f_over_r;
							/* friction */
							rx = X[i]-ColumnCenterX;
							ry = Y[i]-ColumnCenterY;
							scal_prod_over_rsqr = (ry*VX[i] - rx*VY[i]) / SQR(tmpr);

							tmp_ftax =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
										 * (   ry * scal_prod_over_rsqr );
							tmp_ftay =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
										 * ( - rx * scal_prod_over_rsqr );


						}


						/* summing forces */
						if(Injured[i]==0) {
							fcolx[i] = tmp_fpsx + tmp_fyox + tmp_ftax;
							fcoly[i] = tmp_fpsy + tmp_fyoy + tmp_ftay;
						} else { /* ie. if Injured[i]==1 */
							fcolx[i] = tmp_fyox + tmp_ftax;
							fcoly[i] = tmp_fyoy + tmp_ftay;
						}


						/* sum of magnitude of touching forces */
						if(InjurySwitch==1) {
							ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
						}
					}
				// }
				break;
			}
		}
	}
}

__global__ void calcInjuryForces (float *fsmokex, float *fsmokey, float *VX, float *VY, float *V0of, int *Injured, float *ftmagsum, int N, int UpdNum, float *SimTime, float *Phi, float *X, float *D, parameter *para){

	// lokale Variablen
	float x_smokefront, tmpf,tmpr;
	
	int InjurySwitch = para -> InjurySwitch; 
	float FCrush_over_1m = para -> FCrush_over_1m;
	float SmokeStartTime = para -> SmokeStartTime;
	float VSmoke = para -> VSmoke;
	float R = para -> R ;
	float A_fire = para -> A_fire;
	float B_fire = para -> B_fire;
	
	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
	
	if (i <= N) {
	
		switch(InjurySwitch) {
		case 0: {
				break;
			}
		case 1: {

				/* case: people crushed */
			
				// frisch verletzt
				if((ftmagsum[i]>FCrush_over_1m*PI*D[i])&&(Injured[i]==0)) {
					Injured[i] = 1;
					// NInjured++; wird anschließend neu berechnet
					V0of[i] = 0.0;
				}
				
				break;
			}
		case 2:
		case 3: {

				/* case: smoke front */
				if(SimTime[UpdNum]>=SmokeStartTime) {
					x_smokefront = (SimTime[UpdNum]-SmokeStartTime)*VSmoke;

					
					/* checking position compared to smoke front */
					tmpr = X[i] - x_smokefront;

					/* center of particle behind smoke front: injured */
					
					printf("Index: %d verletzt: %d \n", i, Injured[i]);
					if( tmpr < 0.5*D[i] ) {
						if(Injured[i]==0) {
							Injured[i] = 1;
							
							V0of[i] = 0.0;
							VX[i] = VY[i] = 0.0;
						}
					}
					/* ahead of front but within its interaction range:
					trying to escape */
					if( (tmpr>=0.5*D[i])&&(tmpr<=R) ) {
						tmpf = A_fire*exp(-(tmpr-0.5*D[i])/B_fire);
						fsmokex[i] += cos(Phi[i])*tmpf;
						fsmokey[i] += sin(Phi[i])*tmpf;
					}
					
				}
				break;
			}
		}
	}
}

__global__ void storeOldValues (float* Xprev_d,float* X_d, float* Yprev_d, float*Y_d, int N)
{

    int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
    if (i <= N) {
        Xprev_d[i] = X_d[i];
        Yprev_d[i] = Y_d[i];
    }
}

__global__ void calcNewValues (float* X_d,float* Y_d,float* VY_d,float* VX_d, float tstep, int N)
{
    int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	if (i <= N) {
        X_d[i] += VX_d[i] * tstep;
        Y_d[i] += VY_d[i] * tstep;
    }
}

__global__ void sumUp (const int *summanden, const int countElements, int* sum) {
	float summe = 0;
	int i;
	for (i = 0; i < countElements; i++) {
		summe = summe + summanden[i];
	}
	*sum = summe;
}

__global__ void storeNewVelocity (float *VX, float *VY, float *V, float *Vdir,  float *Phi, const float *X, const float *Y, const float *D, wall *W,  const float *vxnew, const float *vynew, parameter *para, const int N, float YS) {

	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
    
	if (i <= N) {
        VX[i] = vxnew[i];
        VY[i] = vynew[i];
        V[i] = sqrt(SQR(VX[i])+SQR(VY[i]));
        Vdir[i] = atan2(VY[i],VX[i]);
        Phi[i] = DirectionOfExit(X[i], Y[i], D[i], YS, para, W);
    }
}