#include "hip/hip_runtime.h"
#include "kernels.h"
#include "deviceFunc.cu"
#include "base.c"
#include <stdio.h>
__global__ void calcParticelForcesPar (float *fpairx, float *fpairy, float *ftmagsum, float *X, float *Y, float *D, float *VX, float *VY, int *Injured, int N, parameter *para)
{
    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;

    int m;
    float tmprsqr, tmpr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay;

    float R = para -> R;
    int InjurySwitch = para -> InjurySwitch;

    if (i < N) {
        for (m = 0; m < N; m ++) { 	// jedes Partikel überprüfen
            tmprsqr = SQR(X[i]-X[m]) + SQR(Y[i]-Y[m]);
            if( tmprsqr <= SQR(R) ) {
                tmpr = sqrt(tmprsqr);
                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* pair forces */
                /* Force(i,m,...) gives the force exerted by m on i, all forces are symmetric now */
                PP_PsychForce(i,m,tmpr,&tmp_fpsx,&tmp_fpsy, D, X, Y, para);
                if(tmpr<=0.5*(D[i]+D[m])) {
                    PP_YoungForce(i,m,tmpr,&tmp_fyox,&tmp_fyoy, D, X, Y, para);
                    PP_TangForce_FS1(i,m,tmpr,&tmp_ftax,&tmp_ftay, D, X, Y,VX, VY, para);
                }

                /* summing forces */
                if(Injured[i]==0) {
                    fpairx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fpairy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fpairx[i] += tmp_fyox + tmp_ftax;
                    fpairy[i] += tmp_fyoy + tmp_ftay;
                }
                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }
            }
            if (i == m) { // verhindert dass die Werte verfälscht werden, wenn man mit sich selbst prüft
                ftmagsum[i] = 0.0;
                fpairx[i] = 0.0;
                fpairy[i] = 0.0;
            }

        }
    }
}






__global__ void calcParticelForces (float *fpairx, float *fpairy, float *ftmagsum, float *X, float *Y, float *D, float *VX, float *VY, int *Injured, float XS, float YS, int GX, int GY, int G, int *BIndBd, int *BInd, int N, parameter *para)
{
    // int b_ID = blockIdx.x;
    // int i =  b_ID * blockDim.x + threadIdx.x;

    int i;
    int j,k, l,mx, my, m;
    float tmprsqr, tmpr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay;

    float R = para -> R;
    int InjurySwitch = para -> InjurySwitch;
    for(i=0; i<N; i++) {
        // if (i < N) {

        j = (int)floor(X[i]*GX/XS) + G * (int)floor(Y[i]*GY/YS);
        for(k=-1; k<=1; k++) {
            for(l=-1; l<=1; l++) {

                mx = j%G+k;
                my = j/G+l;
                if((mx>=0)&&(mx<GX)&&(my>=0)&&(my<GY)) {

                    m = BIndBd[ (mx+GX)%GX + G * (my%GY) ];
                    /* checking each pair of particles only once */
                    while(m>=i) {
                        m = BInd[m];
                    }
                    if(m!=-1) {
                        do {

                            tmprsqr = SQR(X[i]-X[m]) + SQR(Y[i]-Y[m]);
                            if( tmprsqr <= SQR(R) ) {
                                tmpr = sqrt(tmprsqr);

                                /* init */
                                tmp_fpsx = tmp_fpsy = 0.0;
                                tmp_fyox = tmp_fyoy = 0.0;
                                tmp_ftax = tmp_ftay = 0.0;

                                /* pair forces */
                                /* Force(i,m,...) gives the force exerted by m
                                on i, all forces are symmetric now */
                                PP_PsychForce(i,m,tmpr,&tmp_fpsx,&tmp_fpsy, D, X, Y, para);
                                if(tmpr<=0.5*(D[i]+D[m])) {
                                    PP_YoungForce(i,m,tmpr,&tmp_fyox,&tmp_fyoy, D, X, Y, para);

                                    PP_TangForce_FS1(i,m,tmpr,&tmp_ftax,&tmp_ftay, D, X, Y,VX, VY, para);


                                }

                                /* summing forces */
                                if(Injured[i]==0) {
                                    fpairx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                                    fpairy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                                } else { /* ie. if Injured[i]=1 */
                                    fpairx[i] += tmp_fyox + tmp_ftax;
                                    fpairy[i] += tmp_fyoy + tmp_ftay;
                                }
                                if(Injured[m]==0) {
                                    fpairx[m] -= tmp_fpsx + tmp_fyox + tmp_ftax;
                                    fpairy[m] -= tmp_fpsy + tmp_fyoy + tmp_ftay;
                                } else { /* ie. if Injured[m]=1 */
                                    fpairx[m] -= tmp_fyox + tmp_ftax;
                                    fpairy[m] -= tmp_fyoy + tmp_ftay;
                                }

                                /* sum of magnitude of touching forces */
                                if(InjurySwitch==1) {
                                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                                    ftmagsum[m] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                                }
                            }

                            m = BInd[m];
                            while(m>=i) {
                                m = BInd[m];
                            }

                        } while(m!=-1);
                    }
                }
            }
        }
    }
}

__global__ void calcWallForces (float *fwallx, float *fwally, float *ftmagsum, float *D, int *Injured, float *X, float *Y, wpoint *WP, float *VX, float *VY, parameter *para, int N, int Nw)
{

    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;

    int iw;
    float R = para->R;
    int InjurySwitch = para->InjurySwitch;

    int can_see;
    float tmpr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay ;

    if (i < N) {
        for(iw=0; iw<Nw; iw++) {

            WallParticleRelation(iw,i,&tmpr,&can_see,Y[i],X[i],WP,para);

            if((can_see==1)&&(tmpr<=R)) {

                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* psychological force */

                WallPsychForce(iw,i,tmpr,&tmp_fpsx,&tmp_fpsy, D[i], para);
                /* Young and tangential forces */
                if(tmpr<=0.5*D[i]) {

                    WallYoungForce(iw,i,tmpr,&tmp_fyox,&tmp_fyoy, D[i], para);

                    WallTangForce_FS1(iw,i,tmpr, &tmp_ftax, &tmp_ftay, D[i], VX[i], VY[i], para);

                }
                /* summing wall forces */
                if(Injured[i]==0) {
                    fwallx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fwally[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fwallx[i] += tmp_fyox + tmp_ftax;
                    fwally[i] += tmp_fyoy + tmp_ftay;
                }

                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }

                // measuring x component of touching force exerted on walls left and right from exit
                /* lasse ich erstmal weg
                if((iw==1)||(iw==7)) {
                    FW_x -= tmp_fyox + tmp_ftax;
                }
                */
            }
        }
    }
    // if (i == 1) {
    // printf ("fwallx: &f", fwallx[1]);
    // }

}

__global__ void calcWPointForces (float *fwpointx, float *fwpointy, float *ftmagsum, float *D, int *Injured, float *X, float *Y, wpoint *WP, float *VX, float *VY, parameter *para, int N, int Nwp)
{

    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;


    float R = para->R;
    int InjurySwitch = para->InjurySwitch;
    int iwp;
    int can_see;
    float tmpr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay;

    if (i <=  N) {
        for(iwp=0; iwp<Nwp; iwp++) {

            WPointParticleRelation(iwp,i,&tmpr,&can_see, Y[i], X[i], WP);
            if((can_see==1)&&(tmpr<=R)) {

                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* computing forces */
                WPointPsychForce(iwp,i,tmpr,&tmp_fpsx,&tmp_fpsy, X[i], Y[i], D[i], WP[iwp], para);
                if(tmpr<=0.5*D[i]) {

                    WPointYoungForce(iwp,i,tmpr,&tmp_fyox,&tmp_fyoy, X[i], Y[i], D[i], WP[iwp], para);

                    WPointTangForce_FS1(iwp,i,tmpr,&tmp_ftax,&tmp_ftay, X[i], Y[i], D[i], VX[i], VY[i], WP[iwp], para);
                }

                /* summing forces */
                if(Injured[i]==0) {
                    fwpointx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fwpointy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fwpointx[i] += tmp_fyox + tmp_ftax;
                    fwpointy[i] += tmp_fyoy + tmp_ftay;
                }

                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }

                // measuring x component of touching force exerted on walls left and right from exit
                // erstmal rausgenommen
                //
                // if((iwp==0)||(iwp==3)) {
                //    FW_x -= tmp_fyox + tmp_ftax;
                // }

            }
        }
    }

}

__global__ void calcColumnForces (float *fcolx, float *fcoly, float *ftmagsum,float *D, int *Injured, float *X, float *Y, float *VX, float *VY, parameter *para, int N)
{

    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;

    int InjurySwitch = para->InjurySwitch;
    int ColumnSwitch = para->ColumnSwitch;
    float ColumnCenterX = para-> ColumnCenterX;
    float ColumnCenterY = para-> ColumnCenterY;
    float ColumnD = para-> ColumnD;

    float A = para-> A;
    float B = para-> B;
    float C_Young = para-> C_Young;
    float Kappa = para-> Kappa;

    float R = para -> R;

    // lokale Variable
    float tmprsqr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay, rx, ry, f_over_r, scal_prod_over_rsqr, tmpr;

    /* 1.4
     * column
     */
    if (i < N) {

        switch(ColumnSwitch) {
        default:
        case 0: {

                fcolx[i] = fcoly[i] = 0.0;

                break;
            }
        case 1: {

                tmprsqr = SQR(X[i]-ColumnCenterX)+SQR(Y[i]-ColumnCenterY);
                if(tmprsqr<=SQR(R)) {
                    tmpr=sqrt(tmprsqr);

                    /* init */
                    tmp_fpsx = tmp_fpsy = 0.0;
                    tmp_fyox = tmp_fyoy = 0.0;
                    tmp_ftax = tmp_ftay = 0.0;

                    /* computing forces */
                    /* psychological */
                    f_over_r = A * exp(-(tmpr-0.5*(D[i]+ColumnD))/B) / tmpr;
                    tmp_fpsx = (X[i]-ColumnCenterX) * f_over_r;
                    tmp_fpsy = (Y[i]-ColumnCenterY) * f_over_r;
                    /* touching */
                    if(tmpr<=0.5*(D[i]+ColumnD)) {
                        /* Young */
                        f_over_r = 2.0*C_Young*(0.5*(D[i]+ColumnD)-tmpr) / tmpr;
                        tmp_fyox = (X[i]-ColumnCenterX) * f_over_r;
                        tmp_fyoy = (Y[i]-ColumnCenterY) * f_over_r;
                        /* friction */
                        rx = X[i]-ColumnCenterX;
                        ry = Y[i]-ColumnCenterY;
                        scal_prod_over_rsqr = (ry*VX[i] - rx*VY[i]) / SQR(tmpr);

                        tmp_ftax =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
                                     * (   ry * scal_prod_over_rsqr );
                        tmp_ftay =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
                                     * ( - rx * scal_prod_over_rsqr );


                    }


                    /* summing forces */
                    if(Injured[i]==0) {
                        fcolx[i] = tmp_fpsx + tmp_fyox + tmp_ftax;
                        fcoly[i] = tmp_fpsy + tmp_fyoy + tmp_ftay;
                    } else { /* ie. if Injured[i]==1 */
                        fcolx[i] = tmp_fyox + tmp_ftax;
                        fcoly[i] = tmp_fyoy + tmp_ftay;
                    }


                    /* sum of magnitude of touching forces */
                    if(InjurySwitch==1) {
                        ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                    }
                }
                break;
            }
        }
    }
}

__global__ void calcInjuryForces (float *fsmokex, float *fsmokey, float *VX, float *VY, float *V0of, int *Injured, float *ftmagsum, int N, float SimTime, float *Phi, float *X, float *D, parameter *para)
{

    // lokale Variablen
    float x_smokefront, tmpf,tmpr;

    int InjurySwitch = para -> InjurySwitch;
    float FCrush_over_1m = para -> FCrush_over_1m;
    float SmokeStartTime = para -> SmokeStartTime;
    float VSmoke = para -> VSmoke;
    float R = para -> R ;
    float A_fire = para -> A_fire;
    float B_fire = para -> B_fire;

    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;



    if (i < N) {


        switch(InjurySwitch) {
        case 0: {
                break;
            }
        case 1: {

                /* case: people crushed */

                // frisch verletzt
                if((ftmagsum[i]>FCrush_over_1m*PI*D[i])&&(Injured[i]==0)) {
                    Injured[i] = 1;
                    // NInjured++; wird anschließend neu berechnet
                    V0of[i] = 0.0;
                }

                break;
            }
        case 2:
        case 3: {

                /* case: smoke front */

                if(SimTime>=SmokeStartTime) {
                    x_smokefront = (SimTime-SmokeStartTime)*VSmoke;


                    /* checking position compared to smoke front */
                    tmpr = X[i] - x_smokefront;

                    /* center of particle behind smoke front: injured */


                    if( tmpr < 0.5*D[i] ) {
                        if(Injured[i]==0) {
                            // printf("tmpr: %f, x_smokefront: %f \n", tmpr, x_smokefront);
                            Injured[i] = 1;
                            V0of[i] = 0.0;
                            VX[i] = VY[i] = 0.0;
                        }
                    }
                    /* ahead of front but within its interaction range:
                    trying to escape */
                    if( (tmpr>=0.5*D[i])&&(tmpr<=R) ) {
                        tmpf = A_fire*exp(-(tmpr-0.5*D[i])/B_fire);
                        fsmokex[i] += cos(Phi[i])*tmpf;
                        fsmokey[i] += sin(Phi[i])*tmpf;
                    }

                }
                break;
            }
        }
    }
}

__global__ void sumForces (float *fsumx,float *fsumy,  float *tStepVector, const float sqrt_fact, const float *VX,const float *VY,const float *V0of,const float *Phi,const float *fpairx,const float *fwallx,const float *fwpointx,const float *fpairy,const float *fwally,const float *fwpointy,const float *fsmokex,const float *fsmokey,const float *fcolx,const float *fcoly, const int N, const parameter *para)
{

    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;

    float Tau = para-> Tau;
    float DefaultDeltaT = para -> DefaultDeltaT;
    float V_ChangeLimit = para -> V_ChangeLimit;
    float C_NS = para -> C_NS;
    int InjurySwitch = para -> InjurySwitch;
    int ColumnSwitch = para -> ColumnSwitch;

    float fspx, fspy, ksi, eta;


    if (i < N) {

        /* self-propelling */
        fspx = 1/Tau * (V0of[i]*cos(Phi[i]) - VX[i]);
        fspy = 1/Tau * (V0of[i]*sin(Phi[i]) - VY[i]);



        // noise; die Verwendung habe ich erstmal rausgelassen, siehe erklärung in AnalyseSumForces
        // if(GaTh!=0.0) {
        // ksi = GaussRand(GaMe, GaTh, GaCM);
        // eta = 2.0*PI * rand() / (RAND_MAX+1.0);
        // } else {
        // ksi=0.0;
        // eta=0.0;
        // }

        ksi = 0.0;
        eta = 0.0;


        /* sum of forces */
        fsumx[i] =   fspx + fpairx[i] + fwallx[i] + fwpointx[i] + sqrt_fact * ksi * cos(eta);
        fsumy[i] =   fspy + fpairy[i] + fwally[i] + fwpointy[i] + sqrt_fact * ksi * sin(eta);

        // if (i == 1) {
        // printf ("Kraft in Summe Forces: %f, %f \n",fsumx[i], fsumy[i] );
        // }

        /* adding smoke force */
        if((InjurySwitch==2)||(InjurySwitch==3)) {
            fsumx[i] += fsmokex[i];
            fsumy[i] += fsmokey[i];
        }
        /* adding force of column */
        switch(ColumnSwitch) {
        default:
        case 0: {
                break;
            }
        case 1: {
                fsumx[i] += fcolx[i];
                fsumy[i] += fcoly[i];
                break;
            }
        }

        tStepVector[i] = EulTStep(DefaultDeltaT, sqrt(SQR(fsumx[i])+SQR(fsumy[i])), V_ChangeLimit, C_NS );

        // tStepVector[i] = 0.001;

        // tStepVector[i] = DefaultDeltaT;
        // float f = sqrt(SQR(fsumx[i])+SQR(fsumy[i]));
        // while ( f*(tStepVector[i]) >= V_ChangeLimit ) {
        // tStepVector[i] *= C_NS;

        // }


        // if ((i == 0) && (fsumx[0] < 0)) {
        //

        // }
        // printf ("fsumx: %f, fspx: %f, fpairx: %f, fwallx: %f, fwpointx: %f \n",fsumx[i], fspx, fpairx[i], fwallx[i], fwpointx[i] );
        // printf ("VX: %f, VY: %f \n", VX[0], VY[0]) ;
        // printf ("Phi[i]: %f \n ", Phi[i]);

    }
}


__global__ void NewVelocity (float *vxnew, float *vynew, const float *fsumx, const float *fsumy, const float *VX, const float *VY, const int *Injured, const int N, const float *tStepVector, parameter *para)
{

    /* new velocity */

    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;


    float vnew;
    float Vmax = para-> Vmax;
    int InjurySwitch = para -> InjurySwitch;

    if (i < N) {
        if(  (Injured[i]==1) &&((InjurySwitch==1)||(InjurySwitch==3))) {
            vxnew[i] = 0.0;
            vynew[i] = 0.0;
        } else {
            vxnew[i] = VX[i] + fsumx[i] * tStepVector[i];
            vynew[i] = VY[i] + fsumy[i] * tStepVector[i];
        }

        /* checking new velocity */
        vnew = sqrt( SQR(vxnew[i]) + SQR(vynew[i]) );
        if(vnew > Vmax) {
            vxnew[i] = vxnew[i]/vnew * Vmax;
            vynew[i] = vynew[i]/vnew * Vmax;
        }


    }
    // if (i == 0) {
    // printf ("Berechnung: alte Geschwindigkeit: %f, %f \n",VX[0], VY[0] );
    // printf ("Berechnung: Kraft: %f, %f \n",fsumx[0], fsumy[0] );
    // printf ("Berechnung: neue Geschwindigkeit: %f, %f \n",vxnew[0], vynew[0] );
    // }
}

__global__ void getNewValues (float* Xprev_d,float* X_d, float* Yprev_d, float *Y_d,float *VY_d, float* VX_d, int *NinRoomVektor_d, float tstep, int N, parameter *para)
{
    // speichert die alte Position und berechnet die neue Position der Partikel
    // es wird festgelegt, ob ein Partikel im Raum ist oder nicht
    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;

    float RoomXSize = para -> RoomXSize;

    if (i < N) {
        Xprev_d[i] = X_d[i];
        Yprev_d[i] = Y_d[i];
        X_d[i] += VX_d[i] * tstep;
        Y_d[i] += VY_d[i] * tstep;

        if((Xprev_d[i]>RoomXSize)&&(X_d[i]<=RoomXSize)) {
            NinRoomVektor_d[i] = 1;

        }
        if((Xprev_d[i] <=RoomXSize)&&(X_d[i]>RoomXSize)) {
            NinRoomVektor_d[i] = 0;

        }
        // if (i == 0) {
        // printf ("X_d : %f, Y_d: %f, VX: %f, VY: %f \n", X_d[i], Y_d[i], VX_d[i], VY_d[i]);
        // }

    }
}

__global__ void getMinTimeStep (const float *tStepVector, const int countElements, float *min)
{
    float erg = 10.0;
    int i;


    for (i = 0; i < countElements; i++) {

        if (tStepVector[i] < erg) {
            erg = tStepVector[i];
        }
    }

    *min = erg;
}


__global__ void sumUp (const int *summanden, const int countElements, int* sum)
{
    float summe = 0;
    int i;
    for (i = 0; i < countElements; i++) {
        summe = summe + summanden[i];
    }
    *sum = summe;
    // printf ("summe :%f \n", summe);
}

__global__ void storeNewVelocity (float *VX, float *VY, float *V, float *Vdir,  float *Phi, const float *X, const float *Y, const float *D, wall *W,  const float *vxnew, const float *vynew, parameter *para, const int N, float YS)
{

    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;
    // if (i == 0) {
    // printf ("Phi[i] old : %f \n", Phi[i]);
    // }

    if (i < N) {
        VX[i] = vxnew[i];
        VY[i] = vynew[i];
        V[i] = sqrt(SQR(VX[i])+SQR(VY[i]));
        Vdir[i] = atan2(VY[i],VX[i]);
        Phi[i] = DirectionOfExit(X[i], Y[i], D[i], YS, para, W);
    }
    // if (i == 0) {
    // printf ("Phi[i] new : %f \n", Phi[i]);
    // }
}

__global__ void setV0 (float *V0of, float V0, int N)
{
    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;

    if (i < N) {
        V0of[i] = V0;
    }
}

__global__ void setVdir_Phi (float *Vdir, float *Phi, int N, float *X, float *Y, float *D, float YS,parameter *para, wall *W)
{
    int b_ID = blockIdx.x;
    int i =  b_ID * blockDim.x + threadIdx.x;
    float dir;
    if (i < N) {
        dir = DirectionOfExit(X[i], Y[i], D[i], YS, para, W);
        Vdir[i] = dir;
        Phi[i] = dir;
    }
}

