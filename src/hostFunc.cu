#include "hip/hip_runtime.h"
#include "hostFunc.h"

#include <stdio.h>
#include <string.h>
#include <stddef.h>
#include <stdlib.h>
#include <glog/logging.h>


#define NR_END 1
#define FREE_ARG char*
#define READPAR_EXIT {fprintf(stderr,"readpar EXIT\n");fflush(stderr);exit(-1);}
#define SD_LIB_EXIT {_E("sd_lib.c: Exiting to system.\n");exit(-1);}

#include "Xlib_mod.h"

#define MY_STRLEN 200


void XDrawParticle(int leftxmargin, int upymargin, float magn, float d, float x, float y, int partikelInjured, X11props_t *X11props)
{
    // nötige Props zur Verfügung stellen

    Display *display = X11props -> display;
    GC gc = X11props -> gc;
    Pixmap pix1 = X11props -> pix1;
    int *PaCCode = X11props -> PaCCode;




    /* - drawing the particle  */
    int lxm = leftxmargin, uym = upymargin;

    /* particle color */
    switch(partikelInjured) {
    case 0: {
            XSetForeground( display, gc, PaCCode[0] );
            break;
        }
    case 1: {
            XSetForeground( display, gc, PaCCode[1] );
            break;
        }
    }

    XFillArc(display, pix1, gc,
             (int)floor(lxm + magn * (x - d/2)),
             (int)floor(uym + magn * (y - d/2)),
             (int)floor(magn * d),
             (int)floor(magn * d),
             0, 23040
            );
}
void X11_Pic(float XS, float YS, parameter *para, X11props_t *X11props, int N, int NInRoom, int NInjured, int UpdNum, float* SimTime, int NW, wall *W, float *D, float *X, float *Y, int *Injured)

{

    // Proberties lokal zur Verfügung stellen
    Display *display = X11props -> display;
    GC gc = X11props -> gc;
    Pixmap pix1 = X11props -> pix1;
    int BGCCode = X11props -> BGCCode;
    int ICCode = X11props -> ICCode;
    int X11_WWi = X11props -> X11_WWi;
    int X11_WHe = X11props -> X11_WHe;

    // parameter auslesen
    int X11_Margin = para -> X11_Margin;
    float X11_Magn = para -> X11_Magn;
    int X11_TLH = para -> X11_TLH;
    int Sleep = para -> Sleep;
    int InjurySwitch = para -> InjurySwitch;
    int ColumnSwitch = para -> ColumnSwitch;
    float V0 = para -> V0;
    float SmokeStartTime = para -> SmokeStartTime;
    float VSmoke = para -> VSmoke;
    int X11_InFW = para -> X11_InFW;
    float ColumnCenterX = para -> ColumnCenterX;
    float ColumnD = para -> ColumnD;
    float ColumnCenterY = para -> ColumnCenterY;


    /* 1 cleaning the whole window
       2 drawing particles (smoke front, column)
       3 walls
       4 cleaning the info surface, drawing info
       5 showing it, time delay
       */

    int i,disp_height;
    char disp_str[MY_STRLEN];
    /*  float pmean;*/
    float x;


    /* 1 */
    XSetForeground( display, gc, BGCCode );
    XFillRectangle( display, pix1, gc, 0, 0, X11_WWi, X11_WHe );


    /* 2 */
    for(i=0; i<N; i++) {
        XDrawParticle(X11_InFW, X11_Margin, X11_Magn, D[i], X[i], Y[i], Injured[i], X11props);
    }



    /* 2.B */
    /* smoke front, if needed */
    if(  ((InjurySwitch==2)||(InjurySwitch==3))
            &&(SimTime[UpdNum]>=SmokeStartTime)) {
        XSetForeground( display, gc, ICCode );
        x =   X11_InFW + X11_Magn*(SimTime[UpdNum]-SmokeStartTime)*VSmoke;
        for(i=0; i<=X11_Magn*YS/6.0; i++) {
            XDrawLine(display, pix1, gc,
                      x, X11_Margin + 6*i,
                      x, X11_Margin + 6*i+3
                     );
        }
    }

    /* 2.C */
    /* column */
    switch(ColumnSwitch) {
    default:
    case 0: {
            break;
        }
    case 1: {
            XSetForeground( display, gc, ICCode );
            XDrawArc(display, pix1, gc,
                     (int)floor(X11_InFW+X11_Magn*(ColumnCenterX-0.5*ColumnD)),
                     (int)floor(X11_Margin+X11_Magn*(ColumnCenterY-0.5*ColumnD)),
                     (int)floor(X11_Magn*ColumnD),
                     (int)floor(X11_Magn*ColumnD),
                     0, 23040
                    );
            break;
        }
    }



    /* 3 */
    XSetForeground( display, gc, ICCode );
    for(i=0; i<NW; i++) {
        XDrawLine(display,pix1,gc,
                  (int)floor(X11_InFW+X11_Magn*W[i].x1),
                  (int)floor(X11_Margin+X11_Magn*W[i].y1),
                  (int)floor(X11_InFW+X11_Magn*W[i].x2),
                  (int)floor(X11_Margin+X11_Magn*W[i].y2)
                 );
    }

    /* 4 */
    XSetForeground( display, gc, BGCCode );

    // cleaning the x=XS end of the field to allow particles leave the screen gradually
    XFillRectangle( display, pix1, gc,
                    (int)floor(X11_InFW+X11_Magn*XS), 0,
                    (int)floor(X11_WWi-X11_InFW-X11_Magn*XS), X11_WHe );

    /* writing info */
    XSetForeground( display, gc, ICCode );
    disp_height = X11_Margin + X11_TLH;

    disp_height += X11_TLH;
    sprintf( disp_str, "t [%6d] = %.1f", UpdNum, SimTime[UpdNum] );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );

    disp_height += X11_TLH;
    sprintf( disp_str, "N = %d", N );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );

    disp_height += X11_TLH;
    sprintf( disp_str, "N_injured = %d", NInjured );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );

    disp_height += X11_TLH;
    sprintf( disp_str, "V0 = %g", V0 );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );




    /* 5 */
    h_show(X11_WWi,X11_WHe, X11props);
    sleep(Sleep);
}


bool needToDraw (int UpdNum, int  DrawUN, float DrawST, float *SimTime)
{
    if(  (UpdNum==0)
            ||(  (UpdNum>0)
                 &&(  (  (DrawUN != 0)
                         &&(UpdNum % DrawUN == 0)
                      )
                      ||(  (DrawUN == 0)
                           &&(   floor( SimTime[UpdNum] / DrawST )
                                 > floor( SimTime[UpdNum-1] / DrawST )
                             )
                        )
                   )
              )
      ) {

        return true;

    }
    return false;
}



void Save_Demo(int UpdNum, int  SaveUN, float SaveST, float *SimTime,  float AyS, int Mb, int Me, float *E)
{
    LOG (INFO) << "Save Demo gestartet. ";
    float simtime_now,simtime_now_minus_1,e_now,e_now_minus_1;

    if(  (UpdNum==0)
            ||(  (UpdNum>0)
                 &&(  (  (SaveUN != 0)
                         &&(UpdNum % SaveUN == 0)
                      )
                      ||(  (SaveUN == 0)
                           &&(   floor( SimTime[UpdNum] / SaveST )
                                 > floor( SimTime[UpdNum-1] / SaveST )
                             )
                        )
                   )
              )
      ) {




        // closing present time window, opening new time window
        if(UpdNum>0) {
            LOG (INFO) << "closing present time window, opening new time window";
            simtime_now = SimTime[UpdNum];
            simtime_now_minus_1 = SimTime[UpdNum-1];
            e_now = E[UpdNum];
            e_now_minus_1 = E[UpdNum-1];
            LOG (INFO) << "Werte berechnet.";

            free_vector(SimTime,Mb,Me);
            free_vector(E,Mb,Me);
            LOG (INFO) << "Vektoren freigegeben";

            Mb = UpdNum-1;
            Me = UpdNum-1 + AyS-1;
            SimTime = vector(Mb,Me);
            SimTime[UpdNum-1] = simtime_now_minus_1;
            SimTime[UpdNum] = simtime_now;
            E = vector(Mb,Me);
            E[UpdNum-1] = e_now_minus_1;
            E[UpdNum] = e_now;
            LOG (INFO) << "neue Vektoren erzeugt.";
        }
    }
    LOG (INFO) << "Save Demo beendet.";
}

void X11_init(parameter *para, float XS, float YS, X11props_t *X11props)
{
    LOG (INFO) << "x11 init gestartet.";
    int ii,last_ok;
    XColor sdef,edef;

    // benötige Proberties lokal zur Verfügung stellen
    Display *display = X11props -> display;
    Colormap cmap = X11props -> cmap;



    LOG (INFO) << "Props lokal zur Verfügung gestellt.";

    // parameter auslesen
    int X11_InFW = para -> X11_InFW;
    int X11_InFH = para -> X11_InFH;
    int X11_Margin = para -> X11_Margin;
    int X11_GrFH = para -> X11_GrFH;

    float X11_Magn = para -> X11_Magn;

    char X11_FontName [MY_STRLEN];
    strcpy (X11_FontName, para->X11_FontName);
    char BackGroundColorName [MY_STRLEN];
    strcpy (BackGroundColorName, para -> BackGroundColorName);
    char InfoColorName[MY_STRLEN];
    strcpy (InfoColorName, para -> InfoColorName);


    LOG (INFO) << "Parameter ausgelesen.";

    char *ParticleColorName[]= {"yellow", "grey"};
    

    int BGCCode, ICCode, PaCNum, *PaCCode;
    LOG (INFO) << "lokale Werte initialisiert." ;

    /* general */
    int X11_WWi = X11_InFW + (int)(X11_Magn*XS) + X11_Margin;
    int X11_WHe = (int)MAX( X11_InFH, X11_Magn*YS+X11_GrFH + 3*X11_Margin );

    g_win( "open", " PanicSimulator", "PanSim", 0, 0, X11_WWi, X11_WHe, 4, X11props);
    LOG (INFO) << "g_win durchgeführt.";
    g_font( "open", X11_FontName, X11props);
    LOG (INFO) << "g_font durchgeführt.";

    // nach den Aufrufen von g_win und g_font müßen display und cmap neu gesetzt werden
    display = X11props -> display;
    cmap = X11props -> cmap;

    /* colors */
    if( !XAllocNamedColor(display,cmap,BackGroundColorName,&edef,&sdef) ) {
        fprintf(stderr,"Error: couldn't allocate color: %s\n", BackGroundColorName);
        SD_LIB_EXIT;
    }
    BGCCode = sdef.pixel;

    if( !XAllocNamedColor(display,cmap,InfoColorName,&edef,&sdef) ) {
        fprintf(stderr,"Error: couldn't allocate color: %s\n",InfoColorName);
        SD_LIB_EXIT;
    }
    ICCode = sdef.pixel;


    PaCNum = sizeof(ParticleColorName)/sizeof(char*);
    PaCCode = ivector(0,PaCNum-1);
    for(ii=0,last_ok=0; ii<PaCNum; ii++) {
        if( !XAllocNamedColor(display,cmap,ParticleColorName[ii],&edef,&sdef) ) {
            fprintf(stderr,"WARNING: couldn't allocate color: %s\n",
                    ParticleColorName[ii]);
            fprintf(stderr,"Using %s instead\n",ParticleColorName[last_ok]);
            PaCCode[ii]=PaCCode[last_ok];
        } else {
            PaCCode[ii] = sdef.pixel;
            last_ok=ii;
        }

    }

    // lokale Veränderungen speichern
    X11props -> display = display;
    X11props -> cmap = cmap ;

	
	
    X11props -> BGCCode = BGCCode;
    X11props -> ICCode = ICCode;
    X11props -> PaCNum = PaCNum;
    X11props -> PaCCode = PaCCode;
    
    X11props -> X11_WWi = X11_WWi;
    X11props -> X11_WHe = X11_WHe;
}

void PlaceParticel (float *X, float *Y, float *D,parameter *para, float EPSILON)
{

    int i,j,ok_flag;
    int N = para-> N0;
    int ColumnSwitch = para -> ColumnSwitch;
    float ColumnCenterX = para -> ColumnCenterX;
    float ColumnCenterY = para -> ColumnCenterY;
    float ColumnD = para -> ColumnD;
    float Dmean = para-> Dmean;
    float deltaD = para -> deltaD;
    float H = para -> H;
    float RoomXSize = para -> RoomXSize;
    float RoomYSize = para -> RoomYSize;


    for(i=0; i<N; i++) {
        D[i] =   (Dmean + deltaD) - 2.0*deltaD * rand()/(RAND_MAX+1.0);
        X[i] =   0.5*H*D[i]+EPSILON + (RoomXSize-H*D[i]-2.0*EPSILON)*rand()/(RAND_MAX+1.0);
        Y[i] =   0.5*H*D[i]+EPSILON + (RoomYSize-H*D[i]-2.0*EPSILON)*rand()/(RAND_MAX+1.0);


        /* checking whether far enough from the column */
        ok_flag = 1;
        switch(ColumnSwitch) {
        default:
        case 0: {
                break;
            }
        case 1: {
                if(   SQR(X[i]-ColumnCenterX)+SQR(Y[i]-ColumnCenterY)
                        <= SQR(0.5*(D[i]+ColumnD))+EPSILON
                  ) {
                    ok_flag = 0;
                    i--;
                }
                break;
            }
        }

        /* checking distances to already existing particles */
        if(ok_flag==1) {
            for(j=0; j<i; j++) {
                if(     SQR(X[j] - X[i])
                        + SQR(Y[j] - Y[i])
                        <= SQR( 0.5*H*(D[i]+D[j]) ) + EPSILON
                  ) {
                    i = i - 1;
                    j = i - 1;
                }
            }
        }
    }
}


void InitRoom (wall *W,wpoint *WP, parameter *para, float XS)
{

    float RoomXSize = para -> RoomXSize;
    float RoomYSize = para -> RoomYSize;
    float DoorWidth = para -> DoorWidth;
    float WallWidth = para -> WallWidth;



    /* every wall rotated by PI/2 points towards the inside of the room */

    /* upper part */
    W[0].x1 = 0.0;
    W[0].y1 = 0.0;
    W[0].x2 = XS;
    W[0].y2 = 0.0;

    W[1].x1 = RoomXSize;
    W[1].y1 = 0.0;
    W[1].x2 = W[2].x1 = WP[0].x = RoomXSize;
    W[1].y2 = W[2].y1 = WP[0].y = 0.5*RoomYSize-0.5*DoorWidth;
    W[2].x2 = W[3].x1 = WP[1].x = RoomXSize+WallWidth;
    W[2].y2 = W[3].y1 = WP[1].y = 0.5*RoomYSize-0.5*DoorWidth;
    W[3].x2 = RoomXSize+WallWidth;
    W[3].y2 = 0.0;


    /* lower part */
    W[4].x1 = XS;
    W[4].y1 = RoomYSize;
    W[4].x2 = 0.0;
    W[4].y2 = RoomYSize;

    W[5].x1 = RoomXSize+WallWidth;
    W[5].y1 = RoomYSize;

    W[5].x2 = W[6].x1 = WP[2].x = RoomXSize+WallWidth;
    W[5].y2 = W[6].y1 = WP[2].y = 0.5*RoomYSize+0.5*DoorWidth;
    W[6].x2 = W[7].x1 = WP[3].x = RoomXSize;
    W[6].y2 = W[7].y1 = WP[3].y = 0.5*RoomYSize+0.5*DoorWidth;
    W[7].x2 = RoomXSize;
    W[7].y2 = RoomYSize;


    /* left wall of the room */
    W[8].x1 = 0.0;
    W[8].y1 = RoomYSize;
    W[8].x2 = 0.0;
    W[8].y2 = 0.0;
}

void InitBookKeeping (int G, int N, float GX, float GY, float XS, float YS, int *BIndBd, int *BInd, float *X, float *Y)
{
    int i, j;
    for(i=0; i<SQR(G); i++) {
        BIndBd[i] = -1;
    }
    for(i=0; i<N; i++) {
        BInd[i] = -1;
    }

    for(i=0; i<N; i++) {
        j = (int)floor(X[i]*GX/XS) + G * (int)floor(Y[i]*GY/YS);

        if(BIndBd[j]==-1) {
            BIndBd[j] = i;
        } else {
            j = BIndBd[j];
            while(BInd[j]!=-1) {
                j = BInd[j];
            }
            BInd[j] = i;
        }
    }
}






void prepareParameter (parameter* p)
{
    LOG(INFO) << "pepareParamter aufgerufen";
    // die Parameter werden eingelesen und in die passende Struct eingefügt
    int N0, InjurySwitch, ColumnSwitch, X11_Margin, X11_InFW, X11_InFH, X11_TLH, X11_GrFH, X11_RightRim, SaveUN, DrawUN, Sleep, Draw, RndSeed, MaxUpdNum, AyS;

    float RoomXSize, RoomYSize, DoorWidth, WallWidth, Dmean, deltaD, A, B, A_fire, B_fire, Kappa, C_Young, R, R_fire, V0, Tau, GaMe, GaTh, GaCM, SmokeStartTime, VSmoke, FCrush_over_1m, ColumnCenterX, ColumnCenterY, ColumnD, X11_Magn, SaveST, DrawST, DrawDMult, MaxSimTime, Vmax, H, DefaultDeltaT, C_NS, V_ChangeLimit;

    char BackGroundColorName[MY_STRLEN], InfoColorName[MY_STRLEN], X11_FontName[MY_STRLEN];
    LOG(INFO) << "Variablen deklariert.";

    int *IPar[]= {&N0, &InjurySwitch, &ColumnSwitch, &X11_Margin, &X11_InFW, &X11_InFH, &X11_TLH, &X11_GrFH, &X11_RightRim, &SaveUN, &DrawUN, &Sleep, &Draw, &RndSeed, &MaxUpdNum, &AyS };
    char *IParName[]= {"N0", "InjurySwitch",
                       "ColumnSwitch", "X11_Margin",
                       "X11_InFW", "X11_InFH", "X11_TLH", "X11_GrFH",
                       "X11_RightRim", "SaveUN", "DrawUN", "Sleep",
                       "Draw", "RndSeed", "MaxUpdNum", "AyS"
                      };

    float *FPar[]= {&RoomXSize, &RoomYSize, &DoorWidth, &WallWidth, &Dmean,
                    &deltaD, &A, &B, &A_fire, &B_fire, &Kappa,
                    &C_Young, &R, &R_fire, &V0, &Tau,
                    &GaMe, &GaTh, &GaCM, &SmokeStartTime, &VSmoke,
                    &FCrush_over_1m,
                    &ColumnCenterX, &ColumnCenterY, &ColumnD,
                    &X11_Magn, &SaveST, &DrawST,
                    &DrawDMult, &MaxSimTime, &Vmax, &H, &DefaultDeltaT,
                    &C_NS, &V_ChangeLimit
                   };
    char *FParName[]= {"RoomXSize", "RoomYSize", "DoorWidth", "WallWidth",
                       "Dmean", "deltaD", "A", "B", "A_fire", "B_fire",
                       "Kappa", "C_Young",
                       "R", "R_fire", "V0", "Tau", "GaMe", "GaTh", "GaCM",
                       "SmokeStartTime", "VSmoke",
                       "FCrush_over_1m",
                       "ColumnCenterX", "ColumnCenterY", "ColumnD",
                       "X11_Magn", "SaveST",
                       "DrawST", "DrawDMult",
                       "MaxSimTime", "Vmax", "H",
                       "DefaultDeltaT", "C_NS", "V_ChangeLimit"
                      };
    char *SPar[]= {BackGroundColorName, InfoColorName, X11_FontName};
    char *SParName[]= {"BackGroundColorName", "InfoColorName", "X11_FontName"};

    LOG(INFO) << "Arrays intialisiert.";

    int IParNum = sizeof(IPar)/sizeof(int*),
        FParNum = sizeof(FPar)/sizeof(float*),
        SParNum = sizeof(SPar)/sizeof(char*);

    LOG(INFO) << "ReadPar wird aufgerufen.";

    readpar ( "start", "panic.par", IPar, IParName, IParNum, FPar, FParName, FParNum, SPar, SParName, SParNum );

    LOG(INFO) << "ReadPar fertig.";

    // paras in Struct kopieren
    // int Para;
    p->N0	=	N0	;
    p->InjurySwitch	=	 InjurySwitch	;
    p->ColumnSwitch	=	 ColumnSwitch	;
    p->X11_Margin	=	 X11_Margin	;
    p->X11_InFW	=	 X11_InFW	;
    p->X11_InFH	=	 X11_InFH	;
    p->X11_TLH	=	 X11_TLH	;
    p->X11_GrFH	=	 X11_GrFH	;
    p->X11_RightRim	=	 X11_RightRim	;
    p->SaveUN	=	 SaveUN	;
    p->DrawUN	=	 DrawUN	;
    p->Sleep	=	 Sleep	;
    p->Draw	=	 Draw	;
    p->RndSeed	=	 RndSeed	;
    p->MaxUpdNum	=	 MaxUpdNum	;
    p->AyS	=	 AyS	;
    LOG(INFO) << "IntPar kopiert.";
    // float Parameter
    p->RoomXSize	=	RoomXSize	;
    p->RoomYSize	=	 RoomYSize	;
    p->DoorWidth	=	 DoorWidth	;
    p->WallWidth	=	 WallWidth	;
    p->Dmean	=	 Dmean	;
    p->deltaD	=	 deltaD	;
    p->A	=	 A	;
    p->B	=	 B	;
    p->A_fire	=	 A_fire	;
    p->B_fire	=	 B_fire	;
    p->Kappa	=	 Kappa	;
    p->C_Young	=	 C_Young	;
    p->R	=	 R	;
    p->R_fire	=	 R_fire	;
    p->V0	=	 V0	;
    p->Tau	=	 Tau	;
    p->GaMe	=	 GaMe	;
    p->GaTh	=	 GaTh	;
    p->GaCM	=	 GaCM	;
    p->SmokeStartTime	=	 SmokeStartTime	;
    p->VSmoke	=	 VSmoke	;
    p->FCrush_over_1m	=	 FCrush_over_1m	;
    p->ColumnCenterX	=	 ColumnCenterX	;
    p->ColumnCenterY	=	 ColumnCenterY	;
    p->ColumnD	=	 ColumnD	;
    p->X11_Magn	=	 X11_Magn	;
    p->SaveST	=	 SaveST	;
    p->DrawST	=	 DrawST	;
    p->DrawDMult	=	 DrawDMult	;
    p->MaxSimTime	=	 MaxSimTime	;
    p->Vmax	=	 Vmax	;
    p->H	=	 H	;
    p->DefaultDeltaT	=	 DefaultDeltaT	;
    p->C_NS	=	 C_NS	;
    p->V_ChangeLimit	=	 V_ChangeLimit	;
    LOG(INFO) << "Float Parameter kopiert.";

    // char parameter
    strcpy (p->BackGroundColorName, BackGroundColorName);
    strcpy (p->InfoColorName, InfoColorName);
    strcpy (p->X11_FontName, X11_FontName);
    LOG(INFO) << "Char Parameter kopiert.";

}

/* readpar_v_2000_03_02.c

   reading parameter file with this format:
   1st column     #
   2nd            one of these characters: i f s (integer/float/string)
   3rd            one of these characters: 0 1 (1:interactive parameter,0:not)
   4th            parameter name
   5th            parameter value
*/


void readpar ( char *sw, char *ifn,
               int *intValue[], char *intName[], int intNum,
               float *floatValue[], char *floatName[], int floatNum,
               char *stringValue[], char *stringName[], int stringNum )
{

    /* sw: switch = "start" or "re"
       ifn: input file name
       each array starts with the 0. element
       */


    FILE *ifp;
    int ii,i,*intFound,*floatFound,*stringFound,exitFlag,
        interactive,tmpInt;
    char tmpString[100], tmpStringVal[100],c;
    float tmpFloat;


    /* malloc, init, etc */
    intFound = ivector(0,intNum-1);
    floatFound = ivector(0,floatNum-1);
    stringFound = ivector(0,stringNum-1);
    for(i=0; i<intNum; i++) {
        intFound[i]=0;
    }
    for(i=0; i<floatNum; i++) {
        floatFound[i]=0;
    }
    for(i=0; i<stringNum; i++) {
        stringFound[i]=0;
    }

    if( !(ifp = fopen(ifn, "r")) ) {
        fprintf(stderr,"readpar: Couldn't open \"%s\" for reading\n",ifn);
        READPAR_EXIT;
    }


    for(ii=0; ii<intNum+floatNum+stringNum; ii++) {
        while(getc(ifp)!=0x23) {}; /* 0x23 = '#' */

        /* type of parameter */
        fscanf(ifp,"%s",&c);
        switch(c) {
        case 0x69: { /* 0x69 = 'i' */
                exitFlag=0;
                fscanf(ifp,"%d %s %d",&interactive,tmpString,&tmpInt);
                for(i=0; (exitFlag==0)&&(i<intNum); i++) {
                    if(strcmp(intName[i],tmpString)==0) {
                        intFound[i]=1;
                        exitFlag=1;
                        if(  (strcmp(sw,"start")==0)
                                ||((strcmp(sw,"re")==0)&&(interactive==1))
                          ) {
                            *(intValue[i])=tmpInt;
                        }
                    }
                }
                if(exitFlag==0) {
                    fprintf(stderr,"readpar WARNING: don't need this integer parameter: %s\n",tmpString);
                }
                break;
            }
        case 0x66: { /* 0x66 = 'f' */
                exitFlag=0;
                fscanf(ifp,"%d %s %f",&interactive,tmpString,&tmpFloat);
                for(i=0; (exitFlag==0)&&(i<floatNum); i++) {
                    if(strcmp(floatName[i],tmpString)==0) {
                        floatFound[i]=1;
                        exitFlag=1;
                        if(  (strcmp(sw,"start")==0)
                                ||((strcmp(sw,"re")==0)&&(interactive==1))
                          ) {
                            *(floatValue[i])=tmpFloat;
                        }
                    }
                }
                if(exitFlag==0) {
                    fprintf(stderr,"readpar WARNING: don't need this float parameter: %s\n",tmpString);
                }
                break;
            }
        case 0x73: { /* 0x73 = 's' */
                exitFlag=0;
                fscanf(ifp,"%d %s %s",&interactive,tmpString,tmpStringVal);
                for(i=0; (exitFlag==0)&&(i<stringNum); i++) {
                    if(strcmp(stringName[i],tmpString)==0) {
                        stringFound[i]=1;
                        exitFlag=1;
                        if(  (strcmp(sw,"start")==0)
                                ||((strcmp(sw,"re")==0)&&(interactive==1))
                          ) {
                            strcpy(stringValue[i],tmpStringVal);
                        }
                    }
                }
                if(exitFlag==0) {
                    fprintf(stderr,"readpar WARNING: don't need this string parameter: %s\n",tmpString);
                }
                break;
            }
        }
    }


    /* checking whether all parameters have been found */
    for(i=0; i<intNum; i++) {
        if(intFound[i]==0) {
            fprintf(stderr,"readpar ERROR: integer parameter %s not found in %s\n",intName[i],ifn);
            READPAR_EXIT;
        }
    }
    for(i=0; i<floatNum; i++) {
        if(floatFound[i]==0) {
            fprintf(stderr,"readpar ERROR: float parameter %s not found in %s\n",floatName[i],ifn);
            READPAR_EXIT;
        }
    }
    for(i=0; i<stringNum; i++) {
        if(stringFound[i]==0) {
            fprintf(stderr,"readpar ERROR: stringparameter %s not found in %s\n",stringName[i],ifn);
            READPAR_EXIT;
        }
    }

    free_ivector(intFound,0,intNum-1);
    free_ivector(floatFound,0,floatNum-1);
    free_ivector(stringFound,0,stringNum-1);
}


void nrerror(char error_text[])
/* Numerical Recipes standard error handler */
{
    fprintf(stderr,"Numerical Recipes run-time error...\n");
    fprintf(stderr,"%s\n",error_text);
    fprintf(stderr,"...now exiting to system...\n");
    exit(1);
}

float *vector(long nl, long nh)
/* allocate a float vector with subscript range v[nl..nh] */
{
    float *v;

    v=(float *)malloc((size_t) ((nh-nl+1+NR_END)*sizeof(float)));
    if (!v) {
        nrerror("allocation failure in vector()");
    }
    return v-nl+NR_END;
}

int *ivector(long nl, long nh)
/* allocate an int vector with subscript range v[nl..nh] */
{
    int *v;

    v=(int *)malloc((size_t) ((nh-nl+1+NR_END)*sizeof(int)));
    if (!v) {
        nrerror("allocation failure in ivector()");
    }
    return v-nl+NR_END;
}

void free_vector(float *v, long nl, long nh)
/* free a float vector allocated with vector() */
{
    free((FREE_ARG) (v+nl-NR_END));
}

void free_ivector(int *v, long nl, long nh)
/* free an int vector allocated with ivector() */
{
    free((FREE_ARG) (v+nl-NR_END));
}