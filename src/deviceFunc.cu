#include "hip/hip_runtime.h"
#include "deviceFunc.h"
#include "base.c"

#include <stdio.h>

__device__ void PP_TangForce_FS1(int i1, int i2, float r, float *fx, float *fy, float *D, float *X, float *Y,float *VX, float *VY, parameter *para)
{
    /* exerted by particle i2 on particle i1 */
    float Kappa = para -> Kappa;
    float rx,ry,vx,vy,scal_prod_over_rsqr;

    rx = X[i1]-X[i2];
    ry = Y[i1]-Y[i2];
    vx = VX[i1]-VX[i2];
    vy = VY[i1]-VY[i2];
    scal_prod_over_rsqr = (ry*vx - rx*vy) / SQR(r);
    *fx = -Kappa * (0.5*(D[i1]+D[i2])-r) * (   ry * scal_prod_over_rsqr );
    *fy = -Kappa * (0.5*(D[i1]+D[i2])-r) * ( - rx * scal_prod_over_rsqr );
}

__device__ void PP_PsychForce(int i1, int i2, float r, float *fx, float *fy, float *D, float *X, float *Y, parameter *para)
{

    float A = para -> A;
    float B = para -> B;

    float f_over_r;

    f_over_r = A*exp(-(r-0.5*(D[i1]+D[i2]))/B) / r;
    *fx = (X[i1]-X[i2]) * f_over_r;
    *fy = (Y[i1]-Y[i2]) * f_over_r;
}

__device__ void PP_YoungForce(int i1, int i2, float r, float *fx, float *fy, float *D, float *X, float *Y, parameter *para)
{
    float C_Young = para -> C_Young;
    float f_over_r;

    f_over_r = 2.0*C_Young*(0.5*(D[i1]+D[i2])-r) / r;
    *fx = (X[i1]-X[i2]) * f_over_r;
    *fy = (Y[i1]-Y[i2]) * f_over_r;
}


__device__ void WallParticleRelation(int iw, int i, float *r, int *can_see, float yCoor, float xCoor, wpoint *WP, parameter *para)
{
    // can_see: whether partice i is within the range of wall iw;  r: distance

    float RoomYSize = para-> RoomYSize;
    float R = para -> R;

    switch(iw) {
    case 0: {
            *r = yCoor;
            break;
        }
    case 1: {
            *r = WP[0].x -xCoor;
            break;
        }
    case 2: {
            *r = yCoor-WP[0].y;
            break;
        }
    case 3: {
            *r = xCoor-WP[1].x;
            break;
        }
    case 4: {
            *r = RoomYSize-yCoor;
            break;
        }
    case 5: {
            *r = xCoor-WP[2].x;
            break;
        }
    case 6: {
            *r = WP[2].y-yCoor;
            break;
        }
    case 7: {
            *r = WP[3].x-xCoor;
            break;
        }
    case 8: {
            *r = xCoor;
            break;
        }
    }


    switch(iw) {
    case 0: {
            if(yCoor<=R) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 1: {
            if((xCoor>=WP[0].x-R)&&(xCoor<=WP[0].x)&&(yCoor<=WP[0].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 2: {
            if((xCoor>=WP[0].x)&&(xCoor<=WP[1].x)&&(yCoor<=WP[0].y+R)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 3: {
            if((xCoor>=WP[1].x)&&(xCoor<=WP[1].x+R)&&(yCoor<=WP[1].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 4: {
            if(yCoor>=RoomYSize-R) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 5: {
            if((xCoor>=WP[2].x)&&(xCoor<=WP[2].x+R)&&(yCoor>=WP[2].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 6: {
            if((xCoor>=WP[3].x)&&(xCoor<=WP[2].x)&&(yCoor>=WP[2].y-R)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 7: {
            if((xCoor<=WP[3].x)&&(xCoor>=WP[3].x-R)&&(yCoor>=WP[3].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 8: {
            if(xCoor<=R) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    }
}




__device__ void WallTangForce_FS1( int iw, int i, float r, float *fx, float *fy, float diameter, float VelocityX_Dir, float VelocityY_Dir, parameter *para )
{
    float Kappa = para-> Kappa;
#define tmp_delta_r (0.5*diameter-r)

    /* friction forces */
    switch(iw) {
    case 0:
    case 2:
    case 4:
    case 6: {
            *fx = -Kappa*tmp_delta_r*VelocityX_Dir;
            *fy = 0.0;
            break;
        }
    case 1:
    case 3:
    case 5:
    case 7:
    case 8: {
            *fx = 0.0;
            *fy = -Kappa*tmp_delta_r*VelocityY_Dir;
            break;
        }
    }

#undef tmp_delta_r
}

__device__ void WallPsychForce(int iw, int i, float r, float *fx, float *fy, float diameter, parameter *para)
{
    float A = para-> A;
    float B = para -> B;
#define tmp_f (A*exp(-(r-0.5*diameter)/B))

    switch(iw) {
    case 0: {
            *fx = 0.0;
            *fy = tmp_f;
            break;
        }
    case 1: {
            *fx = - tmp_f;
            *fy = 0.0;
            break;
        }
    case 2: {
            *fx = 0.0;
            *fy = tmp_f;
            break;
        }
    case 3: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    case 4: {
            *fx = 0.0;
            *fy = - tmp_f;
            break;
        }
    case 5: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    case 6: {
            *fx = 0.0;
            *fy = - tmp_f;
            break;
        }
    case 7: {
            *fx = - tmp_f;
            *fy = 0.0;
            break;
        }
    case 8: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    }

#undef tmp_f
}

__device__ void WallYoungForce(int iw, int i, float r, float *fx, float *fy, float diameter, parameter *para)
{

    float C_Young = para-> C_Young;
#define tmp_f (2.0*C_Young*(0.5*diameter-r))

    switch(iw) {
    case 0: {
            *fx = 0.0;
            *fy = tmp_f;
            break;
        }
    case 1: {
            *fx = - tmp_f;
            *fy = 0.0;
            break;
        }
    case 2: {
            *fx = 0.0;
            *fy = tmp_f;
            break;
        }
    case 3: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    case 4: {
            *fx = 0.0;
            *fy = - tmp_f;
            break;
        }
    case 5: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    case 6: {
            *fx = 0.0;
            *fy = - tmp_f;
            break;
        }
    case 7: {
            *fx = - tmp_f;
            *fy = 0.0;
            break;
        }
    case 8: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    }

#undef tmp_f
}

__device__ void WPointYoungForce(int iwp, int i, float r, float *fx, float *fy, float xCoor, float yCoor, float diameter, wpoint WP, parameter *para )

{
    /* exerted by wpoint iwp on particle i */
    float C_Young = para-> C_Young;
    float rx,ry;

#define tmp_f_over_r ( 2.0*C_Young*(0.5*diameter-r) / r)

    rx=WP.x-xCoor;
    ry=WP.y-yCoor;
    *fx = - rx * tmp_f_over_r;
    *fy = - ry * tmp_f_over_r;

#undef tmp_f_over_r
}

__device__ void WPointPsychForce(int iwp, int i, float r, float *fx, float *fy, float xCoor, float yCoor, float diameter, wpoint WP, parameter *para )

{
    /* exerted by wpoint iwp on particle i */
    float A = para-> A;
    float B = para-> B;
#define tmp_f_over_r (A*exp(-(r-0.5*diameter)/B)/r)

    *fx = (xCoor-WP.x) * tmp_f_over_r;
    *fy = (yCoor-WP.y) * tmp_f_over_r;

#undef tmp_f_over_r
}


__device__ void WPointParticleRelation(int iwp, int i, float *r, int *can_see, float yCoor, float xCoor, wpoint *WP)
{
    /* can_see: whether partice i is within the range of wpoint iwp r: distance */

    *r = sqrt(SQR(WP[iwp].x-xCoor)+SQR(WP[iwp].y-yCoor));

    switch(iwp) {
    case 0: {
            if((xCoor<=WP[0].x)&&(yCoor>=WP[0].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 1: {
            if((xCoor>=WP[1].x)&&(yCoor>=WP[1].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 2: {
            if((xCoor>=WP[2].x)&&(yCoor<=WP[2].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 3: {
            if((xCoor<=WP[3].x)&&(yCoor<=WP[3].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    }
}

__device__ void WPointTangForce_FS1(int iwp, int i, float r, float *fx, float *fy, float xCoor, float yCoor, float diameter, float VelocityX_Dir, float VelocityY_Dir, wpoint WP, parameter *para )

{
    /* exerted by wpoint iwp on particle i */
    float Kappa = para-> Kappa;
    float rx,ry,scal_prod_over_rsqr;

    rx = xCoor-WP.x;
    ry = yCoor-WP.y;
    scal_prod_over_rsqr = (ry*VelocityX_Dir - rx*VelocityY_Dir) / SQR(r);
    *fx = -Kappa * (0.5*diameter-r) * (   ry * scal_prod_over_rsqr );
    *fy = -Kappa * (0.5*diameter-r) * ( - rx * scal_prod_over_rsqr );
}

__device__ float EulTStep(float tmpTimeStep, float f, float V_ChangeLimit, float C_NS )
{
    /* adjusts the time step in a way that the force (fx,fy) doesn't change the velocity of particle i by more than V_ChangeLimit */

    while( f*(tmpTimeStep) >= V_ChangeLimit ) {
        tmpTimeStep *= C_NS;
    }

    return tmpTimeStep;
}


__device__ float DirectionOfExit(float xCoor, float yCoor, float diameter, float YS, parameter *para, wall *W)
{
    float DoorWidth = para-> DoorWidth;
    float RoomXSize = para -> RoomXSize;
    float EPSILON = 1.0e-5;

    // printf ("Dir of Exit: x: %f, y: %f, d: %f, YS: %f \n", xCoor, yCoor, diameter, YS);
    /* direction of exit for particle i */

    float dsqr, /* sqr of particle center - door-post distance */
          rsqr; /* sqr of particle's radius */


    /* behind the upper door-post */
    if((yCoor<=0.5*YS-0.5*DoorWidth+0.5*diameter+EPSILON)&&(xCoor<=RoomXSize)) {

        dsqr = SQR(W[1].x2-xCoor) + SQR(W[1].y2-yCoor);
        rsqr = SQR(0.5*diameter)+EPSILON;
        if(dsqr<=rsqr) {
            /* very close to the door-post */
            if(yCoor<=0.5*YS-0.5*DoorWidth) {
                return( 0.5*PI );
            } else {
                return(   0.5*PI
                          + atan2( W[1].y2-yCoor,W[1].x2-xCoor )
                      );
            }
        } else {
            /* well apart from the door-post */
            return(   atan2( 1.0, sqrt(dsqr/rsqr-1.0) )
                      + atan2( W[1].y2-yCoor,W[1].x2-xCoor )
                  );
        }
    }


    /* behind the lower door-post */
    else if((yCoor>=0.5*YS+0.5*DoorWidth-0.5*diameter-EPSILON)&&(xCoor<=RoomXSize)) {

        dsqr = SQR(W[6].x2-xCoor) + SQR(W[6].y2-yCoor);
        rsqr = SQR(0.5*diameter)+EPSILON;
        if(dsqr<=rsqr) {
            /* very close to the door-post */
            if(yCoor>=0.5*YS+0.5*DoorWidth) {
                return( -0.5*PI );
            } else {
                return( - 0.5*PI
                        + atan2( W[6].y2-yCoor,W[6].x2-xCoor )
                      );
            }
        } else {
            /* well apart from the door-post */
            return( - atan2( 1.0, sqrt(dsqr/rsqr-1.0) )
                    + atan2( W[6].y2-yCoor,W[6].x2-xCoor )
                  );
        }
    }


    /* in the center or outside */
    else {
        return 0.0;
    }
}