#include "hip/hip_runtime.h"
// library file for sd / panic used by sd.c and sd_crunch.c

int MainSwitch_DEFAULT = 0;
char *IFN_DEFAULT = "sd.par";
char *OFN_DEFAULT = "sd.dat";
char *OF2N_DEFAULT = "sd.dat2";

/********  global constants **********/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/stat.h>
#include <unistd.h>

#include "sd_lib.h"

#include "nrutil.c"
#include "base.c"
#include "readpar.c"
#include "Xlibext.c"



#define SD_LIB_EXIT {_E("sd_lib.c: Exiting to system.\n");exit(-1);}
#define MY_STRLEN 200
float EPSILON = 1.0e-5;

char *ParticleColorName[]= {"yellow", "grey"};
char *SmokeColorName="grey";


/* number of walls and wpoints */
#define NW 9
#define NWP 4
typedef struct wall {
    float x1, y1, x2, y2;
} wall;
typedef struct wpoint {
    float x, y;
} wpoint;


/********* global parameters -- to be read from parameter file **********/

static int N0, InjurySwitch, ColumnSwitch,
       X11_Margin, X11_InFW, X11_InFH, X11_TLH, X11_GrFH, X11_RightRim,
       SaveUN, DrawUN, Sleep, Draw,
       RndSeed, MaxUpdNum, AyS;

static float RoomXSize, RoomYSize, DoorWidth, WallWidth, Dmean,
       deltaD, A, B, A_fire, B_fire, Kappa, C_Young, R, R_fire, V0, Tau,
       GaMe, GaTh, GaCM,
       SmokeStartTime, VSmoke, FCrush_over_1m,
       ColumnCenterX, ColumnCenterY, ColumnD,
       X11_Magn, SaveST, DrawST, DrawDMult, MaxSimTime, Vmax, H, DefaultDeltaT, C_NS, V_ChangeLimit;
static char BackGroundColorName[MY_STRLEN], InfoColorName[MY_STRLEN],
       X11_FontName[MY_STRLEN];


int *IPar[]= {&N0, &InjurySwitch, &ColumnSwitch,
              &X11_Margin, &X11_InFW,
              &X11_InFH, &X11_TLH, &X11_GrFH, &X11_RightRim, &SaveUN,
              &DrawUN, &Sleep, &Draw,
              &RndSeed, &MaxUpdNum, &AyS
             };
char *IParName[]= {"N0", "InjurySwitch",
                   "ColumnSwitch", "X11_Margin",
                   "X11_InFW", "X11_InFH", "X11_TLH", "X11_GrFH",
                   "X11_RightRim", "SaveUN", "DrawUN", "Sleep",
                   "Draw", "RndSeed", "MaxUpdNum", "AyS"
                  };
float *FPar[]= {&RoomXSize, &RoomYSize, &DoorWidth, &WallWidth, &Dmean,
                &deltaD, &A, &B, &A_fire, &B_fire, &Kappa,
                &C_Young, &R, &R_fire, &V0, &Tau,
                &GaMe, &GaTh, &GaCM, &SmokeStartTime, &VSmoke,
                &FCrush_over_1m,
                &ColumnCenterX, &ColumnCenterY, &ColumnD,
                &X11_Magn, &SaveST, &DrawST,
                &DrawDMult, &MaxSimTime, &Vmax, &H, &DefaultDeltaT,
                &C_NS, &V_ChangeLimit
               };
char *FParName[]= {"RoomXSize", "RoomYSize", "DoorWidth", "WallWidth",
                   "Dmean", "deltaD", "A", "B", "A_fire", "B_fire",
                   "Kappa", "C_Young",
                   "R", "R_fire", "V0", "Tau", "GaMe", "GaTh", "GaCM",
                   "SmokeStartTime", "VSmoke",
                   "FCrush_over_1m",
                   "ColumnCenterX", "ColumnCenterY", "ColumnD",
                   "X11_Magn", "SaveST",
                   "DrawST", "DrawDMult",
                   "MaxSimTime", "Vmax", "H",
                   "DefaultDeltaT", "C_NS", "V_ChangeLimit"
                  };
char *SPar[]= {BackGroundColorName, InfoColorName, X11_FontName};
char *SParName[]= {"BackGroundColorName", "InfoColorName",
                   "X11_FontName"
                  };


int IParNum = sizeof(IPar)/sizeof(int*),
    FParNum = sizeof(FPar)/sizeof(float*),
    SParNum = sizeof(SPar)/sizeof(char*);



/******* global variables ************/

int MainSwitch, UpdNum, N, GX, GY, G, Mb, Me, *BIndBd, *BInd, GaussFlag, NInRoom, *Injured, NInjured;
int X11_WWi, X11_WHe, BGCCode, ICCode, PaCNum, *PaCCode, SmokeCCode;

float *SimTime, XS, YS, *D, *Phi, *X, *Y, *Xprev, *Yprev, *V, *VX, *VY, *E, *Vdir, GaussSet1, GaussSet2, *V0of;
float FW_x;

XColor BGC_sdef,IC_sdef,PC_sdef[100],SmokeColor_sdef;

char IFN[MY_STRLEN], OFN[MY_STRLEN], OF2N[MY_STRLEN];
FILE *OFP,*OF2P;
wall *W;
wpoint *WP;

struct stat IFStatBuf;
long int IFModTime;


/********************* Funktionen *****************/

// void WallParticleRelation(int iw, int i, float *r, int *can_see)
// {
    // can_see: whether partice i is within the range of wall iw;  r: distance


    // switch(iw) {
    // case 0: {
            // *r = Y[i];
            // break;
        // }
    // case 1: {
            // *r = WP[0].x-X[i];
            // break;
        // }
    // case 2: {
            // *r = Y[i]-WP[0].y;
            // break;
        // }
    // case 3: {
            // *r = X[i]-WP[1].x;
            // break;
        // }
    // case 4: {
            // *r = RoomYSize-Y[i];
            // break;
        // }
    // case 5: {
            // *r = X[i]-WP[2].x;
            // break;
        // }
    // case 6: {
            // *r = WP[2].y-Y[i];
            // break;
        // }
    // case 7: {
            // *r = WP[3].x-X[i];
            // break;
        // }
    // case 8: {
            // *r = X[i];
            // break;
        // }
    // }


    // switch(iw) {
    // case 0: {
            // if(Y[i]<=R) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 1: {
            // if((X[i]>=WP[0].x-R)&&(X[i]<=WP[0].x)&&(Y[i]<=WP[0].y)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 2: {
            // if((X[i]>=WP[0].x)&&(X[i]<=WP[1].x)&&(Y[i]<=WP[0].y+R)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 3: {
            // if((X[i]>=WP[1].x)&&(X[i]<=WP[1].x+R)&&(Y[i]<=WP[1].y)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 4: {
            // if(Y[i]>=RoomYSize-R) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 5: {
            // if((X[i]>=WP[2].x)&&(X[i]<=WP[2].x+R)&&(Y[i]>=WP[2].y)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 6: {
            // if((X[i]>=WP[3].x)&&(X[i]<=WP[2].x)&&(Y[i]>=WP[2].y-R)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 7: {
            // if((X[i]<=WP[3].x)&&(X[i]>=WP[3].x-R)&&(Y[i]>=WP[3].y)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 8: {
            // if(X[i]<=R) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // }
// }
/*------------------------------*/

// void WallPsychForce(int iw, int i, float r, float *fx, float *fy)
// {

// #define tmp_f (A*exp(-(r-0.5*D[i])/B))

    // switch(iw) {
    // case 0: {
            // *fx = 0.0;
            // *fy = tmp_f;
            // break;
        // }
    // case 1: {
            // *fx = - tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // case 2: {
            // *fx = 0.0;
            // *fy = tmp_f;
            // break;
        // }
    // case 3: {
            // *fx = tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // case 4: {
            // *fx = 0.0;
            // *fy = - tmp_f;
            // break;
        // }
    // case 5: {
            // *fx = tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // case 6: {
            // *fx = 0.0;
            // *fy = - tmp_f;
            // break;
        // }
    // case 7: {
            // *fx = - tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // case 8: {
            // *fx = tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // }

// #undef tmp_f
// }

// /*------------------------------*/

// void WallYoungForce(int iw, int i, float r, float *fx, float *fy)
// {

// #define tmp_f (2.0*C_Young*(0.5*D[i]-r))

    // switch(iw) {
    // case 0: {
            // *fx = 0.0;
            // *fy = tmp_f;
            // break;
        // }
    // case 1: {
            // *fx = - tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // case 2: {
            // *fx = 0.0;
            // *fy = tmp_f;
            // break;
        // }
    // case 3: {
            // *fx = tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // case 4: {
            // *fx = 0.0;
            // *fy = - tmp_f;
            // break;
        // }
    // case 5: {
            // *fx = tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // case 6: {
            // *fx = 0.0;
            // *fy = - tmp_f;
            // break;
        // }
    // case 7: {
            // *fx = - tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // case 8: {
            // *fx = tmp_f;
            // *fy = 0.0;
            // break;
        // }
    // }

// #undef tmp_f
// }


// /*------------------------------*/

// void WallTangForce_FS1( int iw, int i, float r, float *fx, float *fy )
// {

// #define tmp_delta_r (0.5*D[i]-r)

    // /* friction forces */
    // switch(iw) {
    // case 0:
    // case 2:
    // case 4:
    // case 6: {
            // *fx = -Kappa*tmp_delta_r*VX[i];
            // *fy = 0.0;
            // break;
        // }
    // case 1:
    // case 3:
    // case 5:
    // case 7:
    // case 8: {
            // *fx = 0.0;
            // *fy = -Kappa*tmp_delta_r*VY[i];
            // break;
        // }
    // }

// #undef tmp_delta_r
// }


/*------------------------------------*/

// void WPointParticleRelation(int iwp, int i, float *r, int *can_see)
// {
    // /* can_see: whether partice i is within the range of wpoint iwp
       // r: distance */

    // *r = sqrt(SQR(WP[iwp].x-X[i])+SQR(WP[iwp].y-Y[i]));

    // switch(iwp) {
    // case 0: {
            // if((X[i]<=WP[0].x)&&(Y[i]>=WP[0].y)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 1: {
            // if((X[i]>=WP[1].x)&&(Y[i]>=WP[1].y)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 2: {
            // if((X[i]>=WP[2].x)&&(Y[i]<=WP[2].y)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // case 3: {
            // if((X[i]<=WP[3].x)&&(Y[i]<=WP[3].y)) {
                // *can_see=1;
            // } else {
                // *can_see=0;
            // }
            // break;
        // }
    // }
// }

// /*------------------------------*/

// void WPointPsychForce(int iwp, int i, float r, float *fx, float *fy)
// {
    // /* exerted by wpoint iwp on particle i */

// #define tmp_f_over_r (A*exp(-(r-0.5*D[i])/B)/r)

    // *fx = (X[i]-WP[iwp].x) * tmp_f_over_r;
    // *fy = (Y[i]-WP[iwp].y) * tmp_f_over_r;

// #undef tmp_f_over_r
// }

// /*------------------------------*/

// void WPointYoungForce(int iwp, int i, float r, float *fx, float *fy)
// {
    // /* exerted by wpoint iwp on particle i */

    // float rx,ry;

// #define tmp_f_over_r ( 2.0*C_Young*(0.5*D[i]-r) / r)

    // rx=WP[iwp].x-X[i];
    // ry=WP[iwp].y-Y[i];
    // *fx = - rx * tmp_f_over_r;
    // *fy = - ry * tmp_f_over_r;

// #undef tmp_f_over_r
// }

// /*------------------------------*/

// void WPointTangForce_FS1(int iwp, int i, float r, float *fx, float *fy)
// {
    // /* exerted by wpoint iwp on particle i */

    // float rx,ry,scal_prod_over_rsqr;

    // rx = X[i]-WP[iwp].x;
    // ry = Y[i]-WP[iwp].y;
    // scal_prod_over_rsqr = (ry*VX[i] - rx*VY[i]) / SQR(r);
    // *fx = -Kappa * (0.5*D[i]-r) * (   ry * scal_prod_over_rsqr );
    // *fy = -Kappa * (0.5*D[i]-r) * ( - rx * scal_prod_over_rsqr );
// }

/*------------------------------*/

void PP_PsychForce(int i1, int i2, float r, float *fx, float *fy)
{

    float f_over_r;

    f_over_r = A*exp(-(r-0.5*(D[i1]+D[i2]))/B) / r;
    *fx = (X[i1]-X[i2]) * f_over_r;
    *fy = (Y[i1]-Y[i2]) * f_over_r;
}

/*------------------------------*/

void PP_YoungForce(int i1, int i2, float r, float *fx, float *fy)
{

    float f_over_r;

    f_over_r = 2.0*C_Young*(0.5*(D[i1]+D[i2])-r) / r;
    *fx = (X[i1]-X[i2]) * f_over_r;
    *fy = (Y[i1]-Y[i2]) * f_over_r;
}

/*---------------------------*/

void PP_TangForce_FS1(int i1, int i2, float r, float *fx, float *fy)
{
    /* exerted by particle i2 on particle i1 */

    float rx,ry,vx,vy,scal_prod_over_rsqr;

    rx = X[i1]-X[i2];
    ry = Y[i1]-Y[i2];
    vx = VX[i1]-VX[i2];
    vy = VY[i1]-VY[i2];
    scal_prod_over_rsqr = (ry*vx - rx*vy) / SQR(r);
    *fx = -Kappa * (0.5*(D[i1]+D[i2])-r) * (   ry * scal_prod_over_rsqr );
    *fy = -Kappa * (0.5*(D[i1]+D[i2])-r) * ( - rx * scal_prod_over_rsqr );
}

/*---------------------------*/

float DirectionOfExit( int i )
{
    /* direction of exit for particle i */

    float dsqr, /* sqr of particle center - door-post distance */
          rsqr; /* sqr of particle's radius */


    /* behind the upper door-post */
    if((Y[i]<=0.5*YS-0.5*DoorWidth+0.5*D[i]+EPSILON)&&(X[i]<=RoomXSize)) {

        dsqr = SQR(W[1].x2-X[i]) + SQR(W[1].y2-Y[i]);
        rsqr = SQR(0.5*D[i])+EPSILON;
        if(dsqr<=rsqr) {
            /* very close to the door-post */
            if(Y[i]<=0.5*YS-0.5*DoorWidth) {
                return( 0.5*PI );
            } else {
                return(   0.5*PI
                          + atan2( W[1].y2-Y[i],W[1].x2-X[i] )
                      );
            }
        } else {
            /* well apart from the door-post */
            return(   atan2( 1.0, sqrt(dsqr/rsqr-1.0) )
                      + atan2( W[1].y2-Y[i],W[1].x2-X[i] )
                  );
        }
    }


    /* behind the lower door-post */
    else if((Y[i]>=0.5*YS+0.5*DoorWidth-0.5*D[i]-EPSILON)&&(X[i]<=RoomXSize)) {

        dsqr = SQR(W[6].x2-X[i]) + SQR(W[6].y2-Y[i]);
        rsqr = SQR(0.5*D[i])+EPSILON;
        if(dsqr<=rsqr) {
            /* very close to the door-post */
            if(Y[i]>=0.5*YS+0.5*DoorWidth) {
                return( -0.5*PI );
            } else {
                return( - 0.5*PI
                        + atan2( W[6].y2-Y[i],W[6].x2-X[i] )
                      );
            }
        } else {
            /* well apart from the door-post */
            return( - atan2( 1.0, sqrt(dsqr/rsqr-1.0) )
                    + atan2( W[6].y2-Y[i],W[6].x2-X[i] )
                  );
        }
    }


    /* in the center or outside */
    else {
        return 0.0;
    }
}

/*-----------------------------------------------*/

void RemoveParticle( int *n, int i )
{
    /* *n: number of particles now
       i: index of particle to be removed */

    int j;


    /* (a) particle i (which is off-board now)
       is removed from the book-keeping
       (block determined by previous coordinates)
       (b) if i != *n-1
           (b1) particle *n - 1 is removed from the book-keeping
          (block determined by previous coordinates)
           (b2) copying all values of particle *n-1 into i's place
           (b3) inserting particle i (that used to be indexed *n-1) into the
                book-keeping, into the block given by the previous
          coordinates (Xprev[i],Yprev[i]), and not into the block
          given by (X[i],Y[i])
          . reason: after this substitution (*n-1 -> i)
          particle i will be looked for in the block of
          (Xprev[i],Yprev[i]) in Upd
          because no one tells the main cycle (located in Upd),
          whether this particle is the result of a substitution
          or not
       (c) decrement particle number  ( *n to *n - 1 ) */


    /* a */
    j = (int)floor(Xprev[i]*GX/XS) + G*(int)floor(Yprev[i]*GY/YS);
    if(BIndBd[j]==i) {
        BIndBd[j] = BInd[i];
    } else {
        j = BIndBd[j];
        while(BInd[j]!=i) {
            j = BInd[j];
        }
        BInd[j] = BInd[i];
    }



    /* b */
    if(i!=*n-1) {

        /* b1 */

        j = (int)floor(Xprev[*n-1]*GX/XS) + G*(int)floor(Yprev[*n-1]*GY/YS);
        if(BIndBd[j]==*n-1) {
            BIndBd[j] = BInd[*n-1];
        } else {
            j = BIndBd[j];
            while(BInd[j]!=*n-1) {
                j = BInd[j];
            }
            BInd[j] = BInd[*n-1];
        }



        /* b2 */
        D[i] = D[*n-1];
        Phi[i] = Phi[*n-1];
        X[i] = X[*n-1];
        Y[i] = Y[*n-1];
        V[i] = V[*n-1];
        VX[i] = VX[*n-1];
        VY[i] = VY[*n-1];
        Xprev[i] = Xprev[*n-1];
        Yprev[i] = Yprev[*n-1];
        Vdir[i]=Vdir[*n-1];
        Injured[i]=Injured[*n-1];
        V0of[i]=V0of[*n-1];



        /* b3 */
        j = (int)floor(Xprev[i]*GX/XS) + G*(int)floor(Yprev[i]*GY/YS);
        if(BIndBd[j]==-1) {
            BIndBd[j] = i;
            BInd[i] = -1;
        } else {
            j = BIndBd[j];
            while(BInd[j]!=-1) {
                j = BInd[j];
            }
            BInd[j] = i;
            BInd[i] = -1;
        }
    }




    /* c */
    (*n)--;
}

/*-----------------------------------------------------------------------*/

void EulTStep( float *tstep, float f )
{
    /* adjusts the time step in a way that the force (fx,fy) doesn't change the velocity of particle i by more than V_ChangeLimit */

    while( f*(*tstep) >= V_ChangeLimit ) {
        *tstep *= C_NS;
    }
}

/********************************/

void Start_Bare( int narg, char *argstr[] )
{
    /* reading command line parameters and parameter file */


    /* 1 */

    switch(narg) {
    case 1: {
            MainSwitch = MainSwitch_DEFAULT;
            strcpy(IFN,IFN_DEFAULT);
            strcpy(OFN,OFN_DEFAULT);
            strcpy(OF2N,OF2N_DEFAULT);
            break;
        }
    case 3: {
            MainSwitch = atoi(argstr[1]);
            strcpy(IFN,argstr[2]);
            strcpy(OFN,OFN_DEFAULT);
            strcpy(OF2N,OF2N_DEFAULT);
            break;
        }

    default: {
            _E("Usage:  either \"sd <MainSwitch> <input file name>\",");
            _E("        or     \"sd\" .\n");
            SD_LIB_EXIT;
        }
    }
    fprintf(stderr,"(default values are:\n MainSwitch = %d, input = %s, output = %s, %s)\n", MainSwitch,IFN,OFN,OF2N);
    fflush(stderr);

    /* 2 */

    /* reading parameters */
    readpar ( "start", IFN, IPar, IParName, IParNum,
              FPar, FParName, FParNum,
              SPar, SParName, SParNum );


    /* 3 */

}

/*----------------------------------------*/

void Init_Bare( char *init_switch )
{
    /* 1 global vars, mem.alloc.
       2 walls, wpoints
       3 particles
     */

    int i,j,ok_flag;



    /* 1 */
    stat(IFN, &IFStatBuf);
    IFModTime = IFStatBuf.st_mtime;
    UpdNum = 0;
    Mb = 0;
    Me = AyS-1;
    SimTime = vector( Mb, Me );
    SimTime[0] = 0.0;
    srand(RndSeed);

    XS = RoomXSize+WallWidth+X11_RightRim+EPSILON;
    YS = RoomYSize;
    N = N0;
    NInRoom = N0;

    GX = (int)MAX(1.0,floor(XS/R));
    GY = (int)MAX(1.0,floor(YS/R));
    G = (int)MAX(GX,GY);

    BIndBd = ivector( 0, SQR(G)-1 );
    BInd = ivector( 0, N0-1 );
    D = vector( 0, N0-1 );
    Phi = vector( 0, N0-1 );
    X = vector( 0, N0-1 );
    Y = vector( 0, N0-1 );
    Xprev = vector( 0, N0-1 );
    Yprev = vector( 0, N0-1 );
    V = vector( 0, N0-1 );
    VX = vector( 0, N0-1 );
    Vdir = vector(0,N0-1);
    VY = vector( 0, N0-1 );
    V0of = vector( 0, N0-1 );
    E = vector( Mb, Me );
    E[0] = 1.0;
    Injured = ivector(0,N0-1);



    /* 2 walls, wpoints */
    /* allocating memory:
     * if there's a column at the door,
     * the four faces and corners of the column have to be initialized,
     * too
     */
    W = (wall*)calloc(NW,sizeof(wall));
    WP = (wpoint*)calloc(NWP,sizeof(wpoint));



    /* every wall rotated by PI/2 points towards the inside of the room */

    /* upper part */
    W[0].x1 = 0.0;
    W[0].y1 = 0.0;
    W[0].x2 = XS;
    W[0].y2 = 0.0;

    W[1].x1 = RoomXSize;
    W[1].y1 = 0.0;
    W[1].x2 = W[2].x1 = WP[0].x = RoomXSize;
    W[1].y2 = W[2].y1 = WP[0].y = 0.5*RoomYSize-0.5*DoorWidth;
    W[2].x2 = W[3].x1 = WP[1].x = RoomXSize+WallWidth;
    W[2].y2 = W[3].y1 = WP[1].y = 0.5*RoomYSize-0.5*DoorWidth;
    W[3].x2 = RoomXSize+WallWidth;
    W[3].y2 = 0.0;


    /* lower part */
    W[4].x1 = XS;
    W[4].y1 = RoomYSize;
    W[4].x2 = 0.0;
    W[4].y2 = RoomYSize;

    W[5].x1 = RoomXSize+WallWidth;
    W[5].y1 = RoomYSize;

    W[5].x2 = W[6].x1 = WP[2].x = RoomXSize+WallWidth;
    W[5].y2 = W[6].y1 = WP[2].y = 0.5*RoomYSize+0.5*DoorWidth;
    W[6].x2 = W[7].x1 = WP[3].x = RoomXSize;
    W[6].y2 = W[7].y1 = WP[3].y = 0.5*RoomYSize+0.5*DoorWidth;
    W[7].x2 = RoomXSize;
    W[7].y2 = RoomYSize;


    /* left wall of the room */
    W[8].x1 = 0.0;
    W[8].y1 = RoomYSize;
    W[8].x2 = 0.0;
    W[8].y2 = 0.0;




    /* 3 */

    /* diameters and coordinates */
    for(i=0; i<N; i++) {
        D[i] =   (Dmean + deltaD)
                 - 2.0*deltaD * rand()/(RAND_MAX+1.0);
        X[i] =   0.5*H*D[i]+EPSILON
                 + (RoomXSize-H*D[i]-2.0*EPSILON)*rand()/(RAND_MAX+1.0);
        Y[i] =   0.5*H*D[i]+EPSILON
                 + (RoomYSize-H*D[i]-2.0*EPSILON)*rand()/(RAND_MAX+1.0);


        /* checking whether far enough from the column */
        ok_flag = 1;
        switch(ColumnSwitch) {
        default:
        case 0: {
                break;
            }
        case 1: {
                if(   SQR(X[i]-ColumnCenterX)+SQR(Y[i]-ColumnCenterY)
                        <= SQR(0.5*(D[i]+ColumnD))+EPSILON
                  ) {
                    ok_flag = 0;
                    i--;
                }
                break;
            }
        }


        /* checking distances to already existing particles */
        if(ok_flag==1) {
            for(j=0; j<i; j++) {
                if(     SQR(X[j] - X[i])
                        + SQR(Y[j] - Y[i])
                        <= SQR( 0.5*H*(D[i]+D[j]) ) + EPSILON
                  ) {
                    i = i - 1;
                    j = i - 1;
                }
            }
        }
    }


    /* book-keeping */
    for(i=0; i<SQR(G); i++) {
        BIndBd[i] = -1;
    }
    for(i=0; i<N; i++) {
        BInd[i] = -1;
    }

    for(i=0; i<N; i++) {
        j = (int)floor(X[i]*GX/XS) + G * (int)floor(Y[i]*GY/YS);

        if(BIndBd[j]==-1) {
            BIndBd[j] = i;
        } else {
            j = BIndBd[j];
            while(BInd[j]!=-1) {
                j = BInd[j];
            }
            BInd[j] = i;
        }
    }



    /* injuries, velocities and preferred directions */
    NInjured = 0;
    for(i=0; i<N; i++) {
        Injured[i] = 0;
    }

    for(i=0; i<N; i++) {
        Phi[i] = DirectionOfExit( i );
        Vdir[i] = Phi[i];
        V[i]=0.0;
        V0of[i]=V0;
        VX[i]=0.0;
        VY[i]=0.0;
    }
}

/*------------------------------*/

float GaussRand( float gmean, float gtheta, float gcutmult )
{
    /* generates a random number (x) with
       P(x) = exp[- (x-gmean)^2 / (2*gtheta)], if x is in
              [gmean - gcutmult*sqrt(gtheta), gmean + gcutmult*sqrt(gtheta)]
            = 0                              , if not */

    if( (GaussFlag==1) && (fabs(GaussSet2-gmean) <= gcutmult*sqrt(gtheta)) ) {
        GaussFlag = 0;
        return GaussSet2;
    } else {
        float v1,v2,rsq,fac;

        GaussFlag = 0;
        do {
            do {
                v1 = 1.0 - 2.0*(rand()/(RAND_MAX+1.0));
                v2 = 1.0 - 2.0*(rand()/(RAND_MAX+1.0));
            } while((rsq=v1*v1+v2*v2) >= 1.0);
            fac = sqrt(-2.0*gtheta*log(rsq)/rsq);
            GaussSet1 = v1*fac;
            GaussSet2 = v2*fac;
        } while(    (fabs(GaussSet1-gmean) > gcutmult*sqrt(gtheta))
                    && (fabs(GaussSet2-gmean) > gcutmult*sqrt(gtheta)) );

        if(fabs(GaussSet1-gmean) <= gcutmult*sqrt(gtheta)) {
            GaussFlag = 1;
            return GaussSet1;
        } else {
            GaussFlag = 0;
            return GaussSet2;
        }
    }
}

/*------------------------------*/

float EMean( char* sw, int unfreq, float stfreq )
{
    /* calculates the mean value of the efficiency of the system for the last
       few update steps -- NOTE: use this function only when UpdNum > 0

       if unfreq != 0, the average will be calculated for the last unfreq
       updates (the present one included)
       if unfreq == 0, the average will be calculated for the shortest
       possible time interval exceeding stfreq */

    int i, start;
    float e_mean, f;


    if(strcmp(sw,"un")==0) {
        start = UpdNum - unfreq;
    } else { /* i.e. if(strcmp(sw,"st")==0) */
        start = Mb; /* start from beginning of present time window */
        f = floor( SimTime[UpdNum] / stfreq );
        while( f - floor( SimTime[start] / stfreq ) > 1.0 ) {
            start++;
        }
        if( start==UpdNum ) {
            start--;
        }
    }
    e_mean = 0.0;
    for(i=start+1; i<=UpdNum; i++) {
        e_mean += E[i] * ( SimTime[i] - SimTime[i-1] );
    }
    e_mean /= SimTime[UpdNum] - SimTime[start];


    e_mean /= V0;
    return e_mean;
}


/*==============================*/



/*--------------------------------------------------*/

void Init_Demo()
{
    /* 1 general
       2 special
       */


    _E("Initializing, please wait... \n");


    /* 1 */
    Init_Bare("demo");



    /* 2 */

    /* opening files */
    if(!(OFP=fopen(OFN,"w"))) {
        fprintf(stderr,"sd_lib.c: Couldn't open %s for writing.\n",OFN);
        SD_LIB_EXIT;
    }
    fprintf(OFP,"UpdNum, SimTime, N, <E>\n");
    fflush(OFP);

    if(!(OF2P=fopen(OF2N,"w"))) {
        fprintf(stderr,"sd_lib.c: Couldn't open %s for writing.\n",OF2N);
        SD_LIB_EXIT;
    }
    fprintf(OF2P,"\n");
    fflush(OF2P);



    /* init visual or data output */
    X11_init();

    _E("... finished.\n");
}

/*------------------------------*/

void X11_init()
{
    int ii,last_ok;
    XColor sdef,edef;


    /* general */
    X11_WWi = X11_InFW + (int)(X11_Magn*XS) + X11_Margin;
    X11_WHe = (int)MAX( X11_InFH, X11_Magn*YS+X11_GrFH + 3*X11_Margin );
    g_win( "open", " self-driven", "sd", 0, 0, X11_WWi, X11_WHe, 4);
    g_font( "open", X11_FontName );


    /* colors */
    if( !XAllocNamedColor(display,cmap,BackGroundColorName,&edef,&sdef) ) {
        fprintf(stderr,"Error: couldn't allocate color: %s\n",
                BackGroundColorName);
        SD_LIB_EXIT;
    }
    BGCCode = sdef.pixel;

    if( !XAllocNamedColor(display,cmap,InfoColorName,&edef,&sdef) ) {
        fprintf(stderr,"Error: couldn't allocate color: %s\n",InfoColorName);
        SD_LIB_EXIT;
    }
    ICCode = sdef.pixel;


    PaCNum = sizeof(ParticleColorName)/sizeof(char*);
    PaCCode = ivector(0,PaCNum-1);
    for(ii=0,last_ok=0; ii<PaCNum; ii++) {
        if( !XAllocNamedColor(display,cmap,ParticleColorName[ii],&edef,&sdef) ) {
            fprintf(stderr,"WARNING: couldn't allocate color: %s\n",
                    ParticleColorName[ii]);
            fprintf(stderr,"Using %s instead\n",ParticleColorName[last_ok]);
            PaCCode[ii]=PaCCode[last_ok];
        } else {
            PaCCode[ii] = sdef.pixel;
            last_ok=ii;
        }
    }
}



/*------------------------------------------*/

void Pic()
{

    if(  (UpdNum==0)
            ||(  (UpdNum>0)
                 &&(  (  (DrawUN != 0)
                         &&(UpdNum % DrawUN == 0)
                      )
                      ||(  (DrawUN == 0)
                           &&(   floor( SimTime[UpdNum] / DrawST )
                                 > floor( SimTime[UpdNum-1] / DrawST )
                             )
                        )
                   )
              )
      ) {

        X11_Pic();

    }
}

/*------------------------------*/

void X11_Pic()
{
    /* 1 cleaning the whole window
       2 drawing particles (smoke front, column)
       3 walls
       4 cleaning the info surface, drawing info
       5 showing it, time delay
       */

    int i,disp_height;
    char disp_str[MY_STRLEN];
    /*  float pmean;*/
    float x;


    /* 1 */
    XSetForeground( display, gc, BGCCode );
    XFillRectangle( display, pix1, gc, 0, 0, X11_WWi, X11_WHe );


    /* 2 */
    for(i=0; i<N; i++) {
        XDrawParticle( i, X11_InFW, X11_Margin, X11_Magn);
    }



    /* 2.B */
    /* smoke front, if needed */
    if(  ((InjurySwitch==2)||(InjurySwitch==3))
            &&(SimTime[UpdNum]>=SmokeStartTime)) {
        XSetForeground( display, gc, ICCode );
        x =   X11_InFW + X11_Magn*(SimTime[UpdNum]-SmokeStartTime)*VSmoke;
        for(i=0; i<=X11_Magn*YS/6.0; i++) {
            XDrawLine(display, pix1, gc,
                      x, X11_Margin + 6*i,
                      x, X11_Margin + 6*i+3
                     );
        }
    }

    /* 2.C */
    /* column */
    switch(ColumnSwitch) {
    default:
    case 0: {
            break;
        }
    case 1: {
            XSetForeground( display, gc, ICCode );
            XDrawArc(display, pix1, gc,
                     (int)floor(X11_InFW+X11_Magn*(ColumnCenterX-0.5*ColumnD)),
                     (int)floor(X11_Margin+X11_Magn*(ColumnCenterY-0.5*ColumnD)),
                     (int)floor(X11_Magn*ColumnD),
                     (int)floor(X11_Magn*ColumnD),
                     0, 23040
                    );
            break;
        }
    }



    /* 3 */
    XSetForeground( display, gc, ICCode );
    for(i=0; i<NW; i++) {
        XDrawLine(display,pix1,gc,
                  (int)floor(X11_InFW+X11_Magn*W[i].x1),
                  (int)floor(X11_Margin+X11_Magn*W[i].y1),
                  (int)floor(X11_InFW+X11_Magn*W[i].x2),
                  (int)floor(X11_Margin+X11_Magn*W[i].y2)
                 );
    }





    /* 4 */
    XSetForeground( display, gc, BGCCode );

    /* cleaning the x=XS end of the field to allow particles
       leave the screen gradually */
    XFillRectangle( display, pix1, gc,
                    (int)floor(X11_InFW+X11_Magn*XS), 0,
                    (int)floor(X11_WWi-X11_InFW-X11_Magn*XS), X11_WHe );

    /* writing info */
    XSetForeground( display, gc, ICCode );
    disp_height = X11_Margin + X11_TLH;

    disp_height += X11_TLH;
    sprintf( disp_str, "t [%6d] = %.1f", UpdNum, SimTime[UpdNum] );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );

    disp_height += X11_TLH;
    sprintf( disp_str, "N (in room) = %d", NInRoom );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );

    disp_height += X11_TLH;
    sprintf( disp_str, "N_injured = %d", NInjured );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );

    disp_height += X11_TLH;
    sprintf( disp_str, "V0 = %g", V0 );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );

    disp_height += X11_TLH;
    sprintf( disp_str, "FWall_x = %.2f", FW_x );
    XDrawString( display, pix1, gc, X11_Margin, disp_height,
                 disp_str, (signed int)strlen(disp_str) );



    /* 5 */
    h_show(X11_WWi,X11_WHe);
    sleep(Sleep);
}


/*--------------------------------------*/



void XDrawParticle( int i, int leftxmargin, int upymargin, float magn)
{

    /* - drawing the particle  */

    int lxm = leftxmargin, uym = upymargin;
    float d,x,y;



    /* particle color */
    switch(Injured[i]) {
    case 0: {
            XSetForeground( display, gc, PaCCode[0] );
            break;
        }
    case 1: {
            XSetForeground( display, gc, PaCCode[1] );
            break;
        }
    }




    /* drawing particle */
    switch( Draw ) {
    default:
    case 0: {

            d = D[i];
            x = X[i];
            y = Y[i];

            XFillArc(display, pix1, gc,
                     (int)floor(lxm + magn * (x - d/2)),
                     (int)floor(uym + magn * (y - d/2)),
                     (int)floor(magn * d),
                     (int)floor(magn * d),
                     0, 23040
                    );

            break;
        }
    case 1: {

            d = D[i];
            x = X[i];
            y = Y[i];

            XFillArc(display, pix1, gc,
                     (int)floor(lxm + magn * (x - d/2)),
                     (int)floor(uym + magn * (y - d/2)),
                     (int)floor(magn * d),
                     (int)floor(magn * d),
                     0, 23040
                    );

            XSetForeground( display, gc, ICCode );
            XDrawArc(display, pix1, gc,
                     (int)floor(lxm + magn * (x - d/2)),
                     (int)floor(uym + magn * (y - d/2)),
                     (int)floor(magn * d),
                     (int)floor(magn * d),
                     0, 23040
                    );

            break;
        }
    case 2: {

            d = DrawDMult * D[i];
            x = X[i];
            y = Y[i];

            XFillArc(display, pix1, gc,
                     (int)floor(lxm + magn * (x - d/2)),
                     (int)floor(uym + magn * (y - d/2)),
                     (int)floor(magn * d),
                     (int)floor(magn * d),
                     0, 23040
                    );

            XSetForeground( display, gc, ICCode );
            XDrawArc(display, pix1, gc,
                     (int)floor(lxm + magn * (x - d/2)),
                     (int)floor(uym + magn * (y - d/2)),
                     (int)floor(magn * d),
                     (int)floor(magn * d),
                     0, 23040
                    );

            break;
        }
    case 3: {

            d = D[i];
            x = X[i];
            y = Y[i];

            if(sqrt(SQR(VX[i])+SQR(VY[i]))>=0.5) {
                XFillArc(display, pix1, gc,
                         (int)floor(lxm + magn * (x - d/2)),
                         (int)floor(uym + magn * (y - d/2)),
                         (int)floor(magn * d),
                         (int)floor(magn * d),
                         0, 23040
                        );
            }

            break;
        }
    }

}

/*-----------------------------------------------*/

void Save_Demo()
{

    char sw[MY_STRLEN];
    float simtime_now,simtime_now_minus_1,e_now,e_now_minus_1,
          e_mean;

    if(  (UpdNum==0)
            ||(  (UpdNum>0)
                 &&(  (  (SaveUN != 0)
                         &&(UpdNum % SaveUN == 0)
                      )
                      ||(  (SaveUN == 0)
                           &&(   floor( SimTime[UpdNum] / SaveST )
                                 > floor( SimTime[UpdNum-1] / SaveST )
                             )
                        )
                   )
              )
      ) {


        if(UpdNum>0) {
            if(SaveUN!=0) {
                strcpy(sw,"un");
            } else { /* ie. if(SaveUN==0) */
                strcpy(sw,"st");
            }
            e_mean=EMean(sw,SaveUN,SaveST);
        } else { /* ie. if(UpdNum==0) */
            e_mean=1.0;
        }
        fprintf(OFP,"%d\t%g\t%d\t%g\n",
                UpdNum,SimTime[UpdNum],N,e_mean);
        fflush(OFP);



        /* closing present time window, opening new time window */
        if(UpdNum>0) {

            simtime_now = SimTime[UpdNum];
            simtime_now_minus_1 = SimTime[UpdNum-1];
            e_now = E[UpdNum];
            e_now_minus_1 = E[UpdNum-1];

            free_vector(SimTime,Mb,Me);
            free_vector(E,Mb,Me);

            Mb = UpdNum-1;
            Me = UpdNum-1 + AyS-1;
            SimTime = vector(Mb,Me);
            SimTime[UpdNum-1] = simtime_now_minus_1;
            SimTime[UpdNum] = simtime_now;
            E = vector(Mb,Me);
            E[UpdNum-1] = e_now_minus_1;
            E[UpdNum] = e_now;
        }
    }
}

/*------------------------------*/

void Shutdown_Demo() {}

/*------------------------------*/

