#include "hip/hip_runtime.h"
#include <glog/logging.h>
#include <stdio.h>

#include "types.h"
#include "hostFunc.h"
#include "kernels.h"

#include "base.c"

int main( int argc, char **argv)
{
    google::InitGoogleLogging(argv[0]);

    // A) Parameter einlesen
    parameter *para_h;
    para_h = (parameter*) malloc (sizeof(parameter));
    prepareParameter (para_h);
    LOG (INFO) << "Parameter eingelsen.";

    // B) globale Werte setzen
    int N0 = para_h-> N0;
    float EPSILON = 1.0e-5;
    float AyS = para_h -> AyS;
    int UpdNum = 0;
    int Mb = 0;
    int Me = AyS-1;
    float *SimTime = vector( Mb, Me );
    SimTime[0] = 0.0;
    srand(para_h->RndSeed);
    float XS = (para_h->RoomXSize)+(para_h->WallWidth)+(para_h->X11_RightRim)+EPSILON;
    float YS = para_h->RoomYSize;
    int N = N0;
    int NInRoom = N0;
    int NInjured = 0;
    int GX = (int)MAX(1.0,floor(XS/para_h->R));
    int GY = (int)MAX(1.0,floor(YS/para_h->R));
    float MaxSimTime = para_h -> MaxSimTime;
    int MaxUpdNum = para_h -> MaxUpdNum;
    float DefaultDeltaT = para_h -> DefaultDeltaT;
	float V0 = para_h-> V0;

    int DrawUN = para_h -> DrawUN;
    float DrawST = para_h -> DrawST;

    float sqrt_fact;
	float tstep;

    const int NW = 9; // Anzahl der Wände
    const int NWP = 4; // Anzahl der wpoints (Ecken)
	
	// C) alle device Pointer deklarieren
    float *Xprev_d, *Yprev_d, *X_d, *Y_d, *V_d, *VX_d, *VY_d, *Vdir_d, *fwallx_d, *fwally_d,*fwpointx_d, *fwpointy_d, *ftmagsum_d, *D_d, *fcolx_d, *fcoly_d, *fsmokex_d, *fsmokey_d, *V0of_d, *SimTime_d, *Phi_d, *fsumx_d, *fsumy_d, *tStepVector_d, *fpairx_d, *fpairy_d, *timeStep_d, *vxnew_d, *vynew_d;

    int *Injured_d, *NInjured_d, *NInRoom_d, *NinRoomVektor_d;

    wpoint *WP_d; wall *W_d; parameter *para_d;
	
	hipError_t error;
	
    int sizeFloatVector = N0 * sizeof(float);
	int siteIntVector = N0 * sizeof(int);
	
	// D) Host Variablen für Init der Simulation
    
    float *D, *X, *Y, *VX, *VY;
	int *Injured;
	
	error = hipHostMalloc (&D,sizeFloatVector,0); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipHostMalloc (&X,sizeFloatVector,0); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipHostMalloc (&Y,sizeFloatVector,0); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipHostMalloc (&VY,sizeFloatVector,0); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipHostMalloc (&VX,sizeFloatVector,0); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipHostMalloc (&Injured,siteIntVector,0); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

	float *Phi = vector( 0, N0-1 );	
    float *Xprev = vector( 0, N0-1 );
    float *Yprev = vector( 0, N0-1 );
    float *V = vector( 0, N0-1 );
    float *Vdir = vector(0,N0-1);
    float *V0of = vector( 0, N0-1 );
    float *E = vector( Mb, Me );
    E[0] = 1.0;
    
    LOG (INFO) << "globale Werte gesetzt.";
	
	
    dim3 dimBlock(256);
    dim3 dimGrid((N0 + dimBlock.x - 1) / dimBlock.x);

    LOG(INFO) << "Block dimensions: " << dimBlock.x << " " << dimBlock.y << " " << dimBlock.z;
    LOG(INFO) << "Grid dimensions: " << dimGrid.x << " " << dimGrid.y << " " << dimGrid.z;
	
	

    // E) walls, wpoints initalisieren
    wall *W;
    wpoint *WP;
    W = (wall*)calloc(NW,sizeof(wall));
    WP = (wpoint*)calloc(NWP,sizeof(wpoint));
    InitRoom (W,WP, para_h, XS);
    LOG (INFO) << "Wände initialisiert.";

    // F) partikel plazieren
    PlaceParticel (X, Y, D,para_h, EPSILON);
    LOG (INFO) << "Partikel erzeugt & plaziert.";

    // InitBookKeeping (G, N0, GX, GY, XS, YS, BIndBd, BInd, X, Y); LOG (INFO) << "Initiales Bookkeeping durchgeführt.";


	// G DeviceVektoren erzeugen
    // G1 .. der Partikeleigenschaften erzeugen
    error = hipMalloc (&Injured_d,N0 * sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&D_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&vxnew_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&vynew_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&V0of_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Xprev_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Yprev_d, sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&X_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Y_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&VY_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&VX_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&V_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Vdir_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Phi_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // G2 .. der globalen Werte
    error = hipMalloc (&timeStep_d,sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // TimeStep auf dem Device
    error = hipMalloc (&NInRoom_d,sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Anzahl der Personen im Raum auf dem Device
    error = hipMalloc (&NInjured_d,sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Anzahl der Verletzten
    error = hipMalloc (&tStepVector_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Hilfsvektor um den neuen minimalen Timestep zu finden
    error = hipMalloc (&NinRoomVektor_d,N0 * sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Hilfsvektor um den die neue Anzahl der Personen im Raum zu ermitteln
    error = hipMalloc (&SimTime_d, sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // G3 .. für die Szene
    error = hipMalloc (&W_d,NW * sizeof(wall));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&WP_d,NWP * sizeof(wpoint));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&para_d,sizeof(parameter));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // G4 .. der Teilkräfte
    error = hipMalloc (&fwallx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fwally_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fwpointx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fwpointy_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fpairx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fpairy_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fcolx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fcoly_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fsmokex_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fsmokey_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fsumy_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fsumx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&ftmagsum_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Device Pointer alloziert.";

    // H) Vektoren der Szene kopieren
    error = hipMemcpy(W_d, W, NW * sizeof(wall), hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(WP_d, WP, NWP * sizeof(wpoint), hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(para_d, para_h, sizeof(parameter), hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Parameter und Wände hochkopiert.";

    // I Eigenschaftsvektoren auf vorbereiten 
    // I1. leere Intialisierung
    error = hipMemset (Injured_d, 0, N * sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (vxnew_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (vynew_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (Xprev_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (Yprev_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (VY_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (VX_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (V_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Leere Eigenschaftsvektoren wurden initialisiert.";

    // I2 erzeugte Werte hochkopieren
    error = hipMemcpy(X_d, X, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(Y_d, Y, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(D_d, D, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "x Koordinaten, y Koordinaten und Durchmesser wurden kopiert.";

    // I3 StartWert für V0 setzen
    setV0 <<<dimGrid, dimBlock>>> (V0of_d, V0, N0);
    hipDeviceSynchronize(); error = hipGetLastError (); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Das Ergebniss von setV0 ist: " << hipGetErrorString(error);

    // I4 StartWerte für Vdir und Phi setzten
    setVdir_Phi <<<dimGrid, dimBlock>>> (Vdir_d, Phi_d, N, X_d, Y_d, D_d, YS,para_d, W_d);
    hipDeviceSynchronize(); error = hipGetLastError (); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Das Ergebniss von setVdir_Phi ist: " << hipGetErrorString(error);

    // I5 DevicePointer der globalen Werte mit initialen Werten setzen
    error = hipMemset (NInRoom_d, 0, sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (NInjured_d, 0, sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (timeStep_d, 0, sizeof(float)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (tStepVector_d, 0, N * sizeof(float)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (NinRoomVektor_d, 0, N * sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (SimTime_d, 0, N * sizeof(float)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "DevicePointer der globalen Werte mit initialen Werten gesetzt.";

    // J) X11 Props intialisieren und X11 Window init
    X11props_t *X11props;
    int sizeX11Porps = (int) sizeof(X11props_t);
    X11props = (X11props_t*) malloc (sizeX11Porps);
    X11props->display = NULL; X11props->vis = NULL; // enthaltene Pointer initalsieren
    X11_init(para_h, XS, YS, X11props);
    LOG (INFO) << "X11 Fenster initialisiert";
	
	hipStream_t CopyDownStream;
	hipStream_t initStream;
	error = hipStreamCreate (&CopyDownStream);
	error = hipStreamCreate (&initStream);
	CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    
	LOG (WARNING) << "Schleife gestartet.";
    do {
        
        LOG (INFO) << "UpDate Nr: " << UpdNum;
		// kräftevektoren neu initialsieren
		error = hipMemsetAsync(fpairy_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fpairx_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fwallx_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fwally_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fcolx_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fcoly_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fsmokex_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fsmokey_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fwpointx_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fwpointy_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(ftmagsum_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fsumx_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemsetAsync(fsumy_d, 0, sizeFloatVector,initStream);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        LOG (INFO) << "temporäre Vektoren initialisiert.";

 

        if (needToDraw (UpdNum, DrawUN, DrawST, SimTime)) {
			// Werte zum zeichnen runterkopieren		
			LOG (INFO) << "Start Zeichnen";
			error = hipMemcpyAsync(X, X_d, sizeFloatVector, hipMemcpyDeviceToHost,CopyDownStream);
			CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
			error = hipMemcpyAsync(Y, Y_d, sizeFloatVector, hipMemcpyDeviceToHost,CopyDownStream);
			CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
			error = hipMemcpyAsync(D, D_d, sizeFloatVector, hipMemcpyDeviceToHost,CopyDownStream);
			CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
			error = hipMemcpyAsync(Injured, Injured_d, N0 * sizeof(int), hipMemcpyDeviceToHost,CopyDownStream);
			CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
			error = hipMemcpyAsync(VX, VX_d, sizeFloatVector, hipMemcpyDeviceToHost,CopyDownStream);
			CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
			error = hipMemcpyAsync(VY, VY_d, sizeFloatVector, hipMemcpyDeviceToHost,CopyDownStream);
			CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
			LOG (INFO) << "Werte zum Zeichnen zurückkopiert.";
            X11_Pic(XS, YS, para_h, X11props, N, NInRoom, NInjured, UpdNum, SimTime, NW, W, D, X, Y, Injured);
        }
        LOG (INFO) << "Fertig mit zeichnen." ;

        // Update Schritt
        tstep = DefaultDeltaT;
        calcWallForces<<<dimGrid, dimBlock>>> (fwallx_d, fwally_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, WP_d, VX_d, VY_d, para_d, N, NW); hipDeviceSynchronize(); error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        calcWPointForces <<<dimGrid, dimBlock>>> (fwpointx_d, fwpointy_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, WP_d, VX_d, VY_d, para_d, N, NWP); 
		hipDeviceSynchronize(); error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        LOG (INFO) << "W und WP Forces berechnet. Start CalcPartikelForces" ;

        //  particle-particle forces berechnen
        calcParticelForcesPar <<<dimGrid, dimBlock>>> (fpairx_d, fpairy_d, ftmagsum_d,X_d, Y_d, D_d, VX_d, VY_d, Injured_d, N, para_d);        
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        LOG (INFO) << "calcParticelForcesOnHost fertig" ;

		// Coloum Forces berechnen
        calcColumnForces <<<dimGrid, dimBlock>>> (fcolx_d, fcoly_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, VX_d, VY_d, para_d, N); 
		hipDeviceSynchronize(); error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

		// InjuryForces berechnen und Anzahl der Verletzten neu bestimmen und auf Host speichern
        calcInjuryForces <<<dimGrid, dimBlock>>> (fsmokex_d, fsmokey_d, VX_d, VY_d, V0of_d, Injured_d,ftmagsum_d, N, SimTime[UpdNum], Phi_d, X_d, D_d, para_d); 
		hipDeviceSynchronize (); error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        sumUp<<<1,1>>> (Injured_d,N, NInjured_d); hipDeviceSynchronize();  error = hipGetLastError ();
        error = hipMemcpy(&NInjured, NInjured_d, sizeof(int), hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        // Summe der Kräfte berechnen
        sqrt_fact = sqrt(tstep/DefaultDeltaT);
        sumForces<<<dimGrid, dimBlock >>> (fsumx_d, fsumy_d, tStepVector_d, sqrt_fact, VX_d, VY_d, V0of_d, Phi_d, fpairx_d, fwallx_d, fwpointx_d, fpairy_d, fwally_d, fwpointy_d, fsmokex_d, fsmokey_d, fcolx_d, fcoly_d,N, para_d);
        hipDeviceSynchronize(); error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        // neuen timeStep bestimmen und auf Host speichern
        getMinTimeStep <<<1,1>>> (tStepVector_d,N, timeStep_d);  hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemcpy(&tstep, timeStep_d, sizeof(float), hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
  
		// neue Geschwindigkeit ermitteln
        NewVelocity <<<dimGrid, dimBlock >>> (vxnew_d,  vynew_d,   fsumx_d,   fsumy_d,   VX_d,   VY_d,   Injured_d, N, tStepVector_d , para_d); 
		hipDeviceSynchronize(); error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

		// neue Positionen bestimmen
        getNewValues <<<dimGrid, dimBlock >>> (Xprev_d, X_d, Yprev_d, Y_d, VY_d, VX_d, NinRoomVektor_d, tstep, N, para_d); hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        // storeNewVelocity
        storeNewVelocity <<<dimGrid, dimBlock >>> (VX_d, VY_d, V_d, Vdir_d,  Phi_d, X_d, Y_d, D_d, W_d,  vxnew_d, vynew_d, para_d, N, YS); 
		hipDeviceSynchronize(); error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        
        SimTime[UpdNum+1] = SimTime[UpdNum] + tstep; UpdNum++;
		LOG (INFO) << "Updateschritt beendet." ;
        
    } while(( UpdNum < MaxUpdNum ) &&( SimTime[UpdNum] < MaxSimTime ));
    LOG (WARNING) << "Schleife beendet.";
    
	hipStreamDestroy(CopyDownStream);
	hipStreamDestroy(initStream);
	
    return 0;
}