#include "hip/hip_runtime.h"
#include <glog/logging.h>
#include <stdio.h>

#include "types.h"
#include "hostFunc.h"
#include "kernels.h"

#include "base.c"




int main( int argc, char **argv)
{
    google::InitGoogleLogging(argv[0]);

    // A) Parameter einlesen
    parameter *para_h;
    para_h = (parameter*) malloc (sizeof(parameter));
    prepareParameter (para_h);
    LOG (INFO) << "Parameter eingelsen.";

    // globale Werte setzen
    int N0 = para_h-> N0;
    float EPSILON = 1.0e-5;
    float AyS = para_h -> AyS;
    int UpdNum = 0;
    int Mb = 0;
    int Me = AyS-1;
    float *SimTime = vector( Mb, Me );
    SimTime[0] = 0.0;
    srand(para_h->RndSeed);
    float XS = (para_h->RoomXSize)+(para_h->WallWidth)+(para_h->X11_RightRim)+EPSILON;
    float YS = para_h->RoomYSize;
    int N = N0;
    int NInRoom = N0;
    int NInjured = 0;
    int GX = (int)MAX(1.0,floor(XS/para_h->R));
    int GY = (int)MAX(1.0,floor(YS/para_h->R));
    int G = (int)MAX(GX,GY);
    float MaxSimTime = para_h -> MaxSimTime;
    int MaxUpdNum = para_h -> MaxUpdNum;
    float DefaultDeltaT = para_h -> DefaultDeltaT;

    int DrawUN = para_h -> DrawUN;
    float DrawST = para_h -> DrawST;

    float sqrt_fact;

    const int NW = 9; // Anzahl der Wände
    const int NWP = 4; // Anzahl der wpoints (Ecken)

    int *BIndBd = ivector( 0, SQR(G)-1 );
    int *BInd = ivector( 0, N0-1 );
    float *D = vector( 0, N0-1 );
    float *Phi = vector( 0, N0-1 );
    float *X = vector( 0, N0-1 );
    float *Y = vector( 0, N0-1 );
    float *Xprev = vector( 0, N0-1 );
    float *Yprev = vector( 0, N0-1 );
    float *V = vector( 0, N0-1 );
    float *VX = vector( 0, N0-1 );
    float *Vdir = vector(0,N0-1);
    float *VY = vector( 0, N0-1 );
    float *V0of = vector( 0, N0-1 );
    float *E = vector( Mb, Me );
    E[0] = 1.0;
    int *Injured = ivector(0,N0-1);
    LOG (INFO) << "globale Werte gesetzt.";

    dim3 dimBlock(256);
    dim3 dimGrid((N0 + dimBlock.x - 1) / dimBlock.x);

    LOG(INFO) << "Block dimensions: " << dimBlock.x << " " << dimBlock.y << " " << dimBlock.z;
    LOG(INFO) << "Grid dimensions: " << dimGrid.x << " " << dimGrid.y << " " << dimGrid.z;


    /* 2 walls, wpoints initalisieren */
    wall *W;
    wpoint *WP;
    W = (wall*)calloc(NW,sizeof(wall));
    WP = (wpoint*)calloc(NWP,sizeof(wpoint));
    InitRoom (W,WP, para_h, XS);
    LOG (INFO) << "Wände initialisiert.";

    // partikel plazieren
    PlaceParticel (X, Y, D,para_h, EPSILON);
    LOG (INFO) << "Partikel erzeugt & plaziert.";

    InitBookKeeping (G, N0, GX, GY, XS, YS, BIndBd, BInd, X, Y);
    LOG (INFO) << "Initiales Bookkeeping durchgeführt.";




    // alle device Pointer erzeugen;
    float *Xprev_d, *Yprev_d, *X_d, *Y_d, *V_d, *VX_d, *VY_d, *Vdir_d, *fwallx_d, *fwally_d,*fwpointx_d, *fwpointy_d, *ftmagsum_d, *D_d, *fcolx_d, *fcoly_d, *fsmokex_d, *fsmokey_d, *V0of_d, *SimTime_d, *Phi_d, *fsumx_d, *fsumy_d, *tStepVector_d, *fpairx_d, *fpairy_d, *timeStep_d, *vxnew_d, *vynew_d;

    int *BIndBd_d, *BInd_d;
    int sizeBIndBd_d = SQR (G) -1;



    int *Injured_d, *NInjured_d;

    int *NInRoom_d, *NinRoomVektor_d;

    wpoint *WP_d;

    wall *W_d;

    parameter *para_d;

    int sizeFloatVector = N0 * sizeof(float);

    hipError_t error;

    // DeviceVektoren der Partikeleigenschaften
    error = hipMalloc (&BInd_d,N0 * sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&BIndBd_d,sizeBIndBd_d * sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&Injured_d,N0 * sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&D_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&vxnew_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&vynew_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&V0of_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&Xprev_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Yprev_d, sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&X_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Y_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&VY_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&VX_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&V_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Vdir_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&Phi_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // DevicePointer der globalen Werte
    error = hipMalloc (&timeStep_d,sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // TimeStep auf dem Device
    error = hipMalloc (&NInRoom_d,sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Anzahl der Personen im Raum auf dem Device
    error = hipMalloc (&NInjured_d,sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Anzahl der Verletzten
    error = hipMalloc (&tStepVector_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Hilfsvektor um den neuen minimalen Timestep zu finden
    error = hipMalloc (&NinRoomVektor_d,N0 * sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Hilfsvektor um den die neue Anzahl der Personen im Raum zu ermitteln

    error = hipMalloc (&SimTime_d, sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // DeviceVektoren für die Szene
    error = hipMalloc (&W_d,NW * sizeof(wall));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&WP_d,NWP * sizeof(wpoint));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&para_d,sizeof(parameter));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // Device Vektoren der Teilkräfte
    error = hipMalloc (&fwallx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fwally_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&fwpointx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fwpointy_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&fpairx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fpairy_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&fcolx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fcoly_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&fsmokex_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fsmokey_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&fsumy_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&fsumx_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&ftmagsum_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    LOG (INFO) << "Device Pointer alloziert.";


    // nötige Elemente hochkopieren u.A. die Parameter auf Device kopieren  *****
    // Vektoren für die Szene hochkopieren

    error = hipMemcpy(W_d, W, NW * sizeof(wall), hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(WP_d, WP, NWP * sizeof(wpoint), hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(para_d, para_h, sizeof(parameter), hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Parameter und Wände hochkopiert.";

    // Eigenschaftsvektoren kopieren bzw. auf Device initialisieren

    // die folgenden leer initialiseren
    error = hipMemset (Injured_d, 0, N * sizeof(int));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (vxnew_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (vynew_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (Xprev_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (Yprev_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (VY_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (VX_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (V_d, 0, N * sizeof(float));
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Leere Eigenschaftsvektoren wurden initialisiert.";

    // erzeugte Werte hochkopieren
    error = hipMemcpy(X_d, X, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(Y_d, Y, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(D_d, D, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(BInd_d, BInd, N * sizeof(int), hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(BIndBd_d, BIndBd, sizeBIndBd_d * sizeof(int), hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);



    LOG (INFO) << "x Koordinaten, y Koordinaten und Durchmesser wurden kopiert.";


    // StartWert für V0 setzen
    float V0 = para_h-> V0;
    setV0 <<<dimGrid, dimBlock>>> (V0of_d, V0, N0);
    hipDeviceSynchronize(); error = hipGetLastError (); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Das Ergebniss von setV0 ist: " << hipGetErrorString(error);


    // StartWerte für Vdir und Phi setzten --> beide mit direction of Exit initialisieren
    setVdir_Phi <<<dimGrid, dimBlock>>> (Vdir_d, Phi_d, N, X_d, Y_d, D_d, YS,para_d, W_d);
    hipDeviceSynchronize(); error = hipGetLastError (); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "Das Ergebniss von setVdir_Phi ist: " << hipGetErrorString(error);

    // DevicePointer der globalen Werte mit initialen Werten setzen
    error = hipMemset (NInRoom_d, 0, sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (NInjured_d, 0, sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (timeStep_d, 0, sizeof(float)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (tStepVector_d, 0, N * sizeof(float)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (NinRoomVektor_d, 0, N * sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset (SimTime_d, 0, N * sizeof(float)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "DevicePointer der globalen Werte mit initialen Werten gesetzt.";

    // temporäre Kraftverktoren initialisiert
    error = hipMemset(fpairy_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fpairx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fwallx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fwally_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fcolx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fcoly_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fsmokex_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fsmokey_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fwpointx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fwpointy_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(ftmagsum_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fsumx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemset(fsumy_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    LOG (INFO) << "temporäre Kraftverktoren initialisiert.";


    // X11 Props intialisieren und X11 Window init
    X11props_t *X11props;
    int sizeX11Porps = (int) sizeof(X11props_t);
    X11props = (X11props_t*) malloc (sizeX11Porps);
    // enthaltene Pointer initalsieren
    X11props->display = NULL;
    X11props->vis = NULL;

    LOG (INFO) << "sizeX11Porps: " << sizeX11Porps;
    X11_init(para_h, XS, YS, X11props);
    LOG (INFO) << "X11 Fenster initialisiert";



    
    float tstep;
    LOG (WARNING) << "Schleife gestartet.";
    do {
        
        LOG (INFO) << "UpDate Nr: " << UpdNum;
        error = hipMemset(fpairy_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fpairx_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fwallx_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fwally_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fcolx_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fcoly_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fsmokex_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fsmokey_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fwpointx_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fwpointy_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(ftmagsum_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fsumx_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemset(fsumy_d, 0, sizeFloatVector);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        LOG (INFO) << "temporäre Vektoren initialisiert.";


        // Werte zum zeichnen runterkopieren
        error = hipMemcpy(X, X_d, sizeFloatVector, hipMemcpyDeviceToHost);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemcpy(Y, Y_d, sizeFloatVector, hipMemcpyDeviceToHost);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemcpy(D, D_d, sizeFloatVector, hipMemcpyDeviceToHost);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemcpy(Injured, Injured_d, N0 * sizeof(int), hipMemcpyDeviceToHost);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemcpy(VX, VX_d, sizeFloatVector, hipMemcpyDeviceToHost);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        error = hipMemcpy(VY, VY_d, sizeFloatVector, hipMemcpyDeviceToHost);
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


        LOG (INFO) << "Werte zum Zeichnen zurückkopiert.";

 


        if (needToDraw (UpdNum, DrawUN, DrawST, SimTime)) {
            X11_Pic(XS, YS, para_h, X11props, N, NInRoom, NInjured, UpdNum, SimTime, NW, W, D, X, Y, Injured);
            // printf (", gezeichnet");
        }
        LOG (INFO) << "Fertig mit zeichnen." ;


        // Update Schritt
        tstep = DefaultDeltaT;
        calcWallForces<<<dimGrid, dimBlock>>> (fwallx_d, fwally_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, WP_d, VX_d, VY_d, para_d, N, NW); hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        calcWPointForces <<<dimGrid, dimBlock>>> (fwpointx_d, fwpointy_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, WP_d, VX_d, VY_d, para_d, N, NWP); hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        LOG (INFO) << "W und WP Forces berechnet. Start CalcPartikelForces" ;

        //  particle-particle forces berechnen
        
        calcParticelForcesPar <<<dimGrid, dimBlock>>> (fpairx_d, fpairy_d, ftmagsum_d,X_d, Y_d, D_d, VX_d, VY_d, Injured_d, N, para_d);        
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        LOG (INFO) << "calcParticelForcesOnHost fertig" ;


        calcColumnForces <<<dimGrid, dimBlock>>> (fcolx_d, fcoly_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, VX_d, VY_d, para_d, N); hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        calcInjuryForces <<<dimGrid, dimBlock>>> (fsmokex_d, fsmokey_d, VX_d, VY_d, V0of_d, Injured_d,ftmagsum_d, N, SimTime[UpdNum], Phi_d, X_d, D_d, para_d); hipDeviceSynchronize ();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        // Anzahl der Verletzten neu bestimmen und auf Host speichern
        sumUp<<<1,1>>> (Injured_d,N, NInjured_d); hipDeviceSynchronize();  error = hipGetLastError ();
        error = hipMemcpy(&NInjured, NInjured_d, sizeof(int), hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        // Summe der Kräfte berechnen
        sqrt_fact = sqrt(tstep/DefaultDeltaT);
        sumForces<<<dimGrid, dimBlock >>> (fsumx_d, fsumy_d, tStepVector_d, sqrt_fact, VX_d, VY_d, V0of_d, Phi_d, fpairx_d, fwallx_d, fwpointx_d, fpairy_d, fwally_d, fwpointy_d, fsmokex_d, fsmokey_d, fcolx_d, fcoly_d,N, para_d);
        hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        // neuen timeStep bestimmen und auf Host speichern
        getMinTimeStep <<<1,1>>> (tStepVector_d,N, timeStep_d);  hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        error = hipMemcpy(&tstep, timeStep_d, sizeof(float), hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        // LOG(WARNING) << "newTimeStep :" << tstep;


        NewVelocity <<<dimGrid, dimBlock >>> (vxnew_d,  vynew_d,   fsumx_d,   fsumy_d,   VX_d,   VY_d,   Injured_d, N, tStepVector_d , para_d); hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        getNewValues <<<dimGrid, dimBlock >>> (Xprev_d, X_d, Yprev_d, Y_d, VY_d, VX_d, NinRoomVektor_d, tstep, N, para_d); hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        // benötigte Werte vorher runterkopieren
        // calcNewBookKeepingOnHost ()
        // veränderte Werte wieder hochkopieren

        // storeNewVelocity
        storeNewVelocity <<<dimGrid, dimBlock >>> (VX_d, VY_d, V_d, Vdir_d,  Phi_d, X_d, Y_d, D_d, W_d,  vxnew_d, vynew_d, para_d, N, YS); hipDeviceSynchronize();
        error = hipGetLastError(); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
        LOG (INFO) << "Updateschritt beendet." ;
        
		// Ende des Update Schrittes
        SimTime[UpdNum+1] = SimTime[UpdNum] + tstep;
        UpdNum++;
        
    } while(( UpdNum < MaxUpdNum ) &&( SimTime[UpdNum] < MaxSimTime ));
    LOG (WARNING) << "Schleife beendet.";
    
    return 0;
}