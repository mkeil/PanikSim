#include "hip/hip_runtime.h"


#include "update.h"
#include <glog/logging.h>
#include <stdio.h>

__global__ void storeOldValues (float* Xprev_d,float* X_d, float* Yprev_d, float*Y_d)
{

    int i = threadIdx.x;
    // printf ("%d\n", i);
    if (i <= 200) {
        Xprev_d[i] = X_d[i];
        Yprev_d[i] = Y_d[i];
    }
}

__global__ void calcNewValues (float* X_d,float* Y_d,float* VY_d,float* VX_d, float tstep)
{


    int i = threadIdx.x;
    if (i <= 200) {
        X_d[i] += VX_d[i] * tstep;
        Y_d[i] += VY_d[i] * tstep;
    }

}
void Upd()
{

    int allocN,i,j,k,l,mx,my,m,can_see,iwp,iw,j_old,j_new;
    float *fwallx,*fwally,*fwpointx,*fwpointy,*fpairx,*fpairy,
          *fspx,*fspy,*fsumx,*fsumy,*vxnew,*vynew,tstep,tmpr,
          tmp_fpsx,tmp_fpsy,tmp_fyox,tmp_fyoy,tmp_ftax,tmp_ftay,
          tmprsqr,sqrt_fact,ksi,eta,vnew,*ftmagsum,*fsmokex,*fsmokey,
          x_smokefront,tmpf,f_over_r,scal_prod_over_rsqr,rx,ry,
          *fcolx,*fcoly;



    /* 0 */
    allocN=N;
    fwallx=vector(0,allocN-1);
    fwally=vector(0,allocN-1);
    fwpointx=vector(0,allocN-1);
    fwpointy=vector(0,allocN-1);
    fpairx=vector(0,allocN-1);
    fpairy=vector(0,allocN-1);
    fsmokex=vector(0,allocN-1);
    fsmokey=vector(0,allocN-1);

    fspx=vector(0,allocN-1);
    fspy=vector(0,allocN-1);
    fsumx=vector(0,allocN-1);
    fsumy=vector(0,allocN-1);
    vxnew=vector(0,allocN-1);
    vynew=vector(0,allocN-1);
    ftmagsum=vector(0,allocN-1);
    fcolx=vector(0,allocN-1);
    fcoly=vector(0,allocN-1);



    /* 1 */

    /* 1.0 */
    /* default values */
    tstep = DefaultDeltaT;
    for(i=0; i<N; i++) {
        fwallx[i] = 0.0;
        fwally[i] = 0.0;
        fwpointx[i] = 0.0;
        fwpointy[i] = 0.0;
        fpairx[i] = 0.0;
        fpairy[i] = 0.0;
        fsmokex[i] = 0.0;
        fsmokey[i] = 0.0;

        fspx[i] = 0.0;
        fspy[i] = 0.0;
        fsumx[i] = 0.0;
        fsumy[i] = 0.0;
        ftmagsum[i] = 0.0;
        fcolx[i] = 0.0;
        fcoly[i] = 0.0;
    }
    FW_x=0.0;






    /* 1.1 */
    /* wall force */
    for(i=0; i<N; i++) {
        for(iw=0; iw<NW; iw++) {

            WallParticleRelation(iw,i,&tmpr,&can_see);
            if((can_see==1)&&(tmpr<=R)) {

                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* psychological force */
                WallPsychForce(iw,i,tmpr,&tmp_fpsx,&tmp_fpsy);
                /* Young and tangential forces */
                if(tmpr<=0.5*D[i]) {
                    WallYoungForce(iw,i,tmpr,&tmp_fyox,&tmp_fyoy);

                    WallTangForce_FS1(iw,i,tmpr,&tmp_ftax,&tmp_ftay);

                }
                /* summing wall forces */
                if(Injured[i]==0) {
                    fwallx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fwally[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fwallx[i] += tmp_fyox + tmp_ftax;
                    fwally[i] += tmp_fyoy + tmp_ftay;
                }

                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }

                /* measuring x component of touching force exerted
                on walls left and right from exit
                 -- only in demo mode */

                if((iw==1)||(iw==7)) {
                    FW_x -= tmp_fyox + tmp_ftax;
                }
            }
        }
    }



    /* 1.2 */
    /* wpoint force */
    for(i=0; i<N; i++) {
        for(iwp=0; iwp<NWP; iwp++) {

            WPointParticleRelation(iwp,i,&tmpr,&can_see);
            if((can_see==1)&&(tmpr<=R)) {

                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* computing forces */
                WPointPsychForce(iwp,i,tmpr,&tmp_fpsx,&tmp_fpsy);
                if(tmpr<=0.5*D[i]) {
                    WPointYoungForce(iwp,i,tmpr,&tmp_fyox,&tmp_fyoy);

                    WPointTangForce_FS1(iwp,i,tmpr,&tmp_ftax,&tmp_ftay);

                }

                /* summing forces */
                if(Injured[i]==0) {
                    fwpointx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fwpointy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fwpointx[i] += tmp_fyox + tmp_ftax;
                    fwpointy[i] += tmp_fyoy + tmp_ftay;
                }

                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }

                /* measuring x component of touching force exerted
                   on walls left and right from exit
                   -- only in demo mode */

                if((iwp==0)||(iwp==3)) {
                    FW_x -= tmp_fyox + tmp_ftax;
                }

            }
        }
    }



    /* 1.3 */
    /* particle-particle forces */
    for(i=0; i<N; i++) {

        j = (int)floor(X[i]*GX/XS) + G * (int)floor(Y[i]*GY/YS);
        for(k=-1; k<=1; k++) {
            for(l=-1; l<=1; l++) {

                mx = j%G+k;
                my = j/G+l;
                if((mx>=0)&&(mx<GX)&&(my>=0)&&(my<GY)) {

                    m = BIndBd[ (mx+GX)%GX + G * (my%GY) ];
                    /* checking each pair of particles only once */
                    while(m>=i) {
                        m = BInd[m];
                    }
                    if(m!=-1) {
                        do {

                            tmprsqr = SQR(X[i]-X[m]) + SQR(Y[i]-Y[m]);
                            if( tmprsqr <= SQR(R) ) {
                                tmpr = sqrt(tmprsqr);

                                /* init */
                                tmp_fpsx = tmp_fpsy = 0.0;
                                tmp_fyox = tmp_fyoy = 0.0;
                                tmp_ftax = tmp_ftay = 0.0;

                                /* pair forces */
                                /* Force(i,m,...) gives the force exerted by m
                                on i, all forces are symmetric now */
                                PP_PsychForce(i,m,tmpr,&tmp_fpsx,&tmp_fpsy);
                                if(tmpr<=0.5*(D[i]+D[m])) {
                                    PP_YoungForce(i,m,tmpr,&tmp_fyox,&tmp_fyoy);
                                    PP_TangForce_FS1(i,m,tmpr,&tmp_ftax,&tmp_ftay);

                                }

                                /* summing forces */
                                if(Injured[i]==0) {
                                    fpairx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                                    fpairy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                                } else { /* ie. if Injured[i]=1 */
                                    fpairx[i] += tmp_fyox + tmp_ftax;
                                    fpairy[i] += tmp_fyoy + tmp_ftay;
                                }
                                if(Injured[m]==0) {
                                    fpairx[m] -= tmp_fpsx + tmp_fyox + tmp_ftax;
                                    fpairy[m] -= tmp_fpsy + tmp_fyoy + tmp_ftay;
                                } else { /* ie. if Injured[m]=1 */
                                    fpairx[m] -= tmp_fyox + tmp_ftax;
                                    fpairy[m] -= tmp_fyoy + tmp_ftay;
                                }

                                /* sum of magnitude of touching forces */
                                if(InjurySwitch==1) {
                                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                                    ftmagsum[m] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                                }
                            }

                            m = BInd[m];
                            while(m>=i) {
                                m = BInd[m];
                            }

                        } while(m!=-1);
                    }
                }
            }
        }
    }



    /* 1.4
     * column
     */
    switch(ColumnSwitch) {
    default:
    case 0: {
            for(i=0; i<N; i++) {
                fcolx[i] = fcoly[i] = 0.0;
            }
            break;
        }
    case 1: {
            for(i=0; i<N; i++) {
                tmprsqr = SQR(X[i]-ColumnCenterX)+SQR(Y[i]-ColumnCenterY);
                if(tmprsqr<=SQR(R)) {
                    tmpr=sqrt(tmprsqr);

                    /* init */
                    tmp_fpsx = tmp_fpsy = 0.0;
                    tmp_fyox = tmp_fyoy = 0.0;
                    tmp_ftax = tmp_ftay = 0.0;

                    /* computing forces */
                    /* psychological */
                    f_over_r = A * exp(-(tmpr-0.5*(D[i]+ColumnD))/B) / tmpr;
                    tmp_fpsx = (X[i]-ColumnCenterX) * f_over_r;
                    tmp_fpsy = (Y[i]-ColumnCenterY) * f_over_r;
                    /* touching */
                    if(tmpr<=0.5*(D[i]+ColumnD)) {
                        /* Young */
                        f_over_r = 2.0*C_Young*(0.5*(D[i]+ColumnD)-tmpr) / tmpr;
                        tmp_fyox = (X[i]-ColumnCenterX) * f_over_r;
                        tmp_fyoy = (Y[i]-ColumnCenterY) * f_over_r;
                        /* friction */
                        rx = X[i]-ColumnCenterX;
                        ry = Y[i]-ColumnCenterY;
                        scal_prod_over_rsqr = (ry*VX[i] - rx*VY[i]) / SQR(tmpr);

                        tmp_ftax =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
                                     * (   ry * scal_prod_over_rsqr );
                        tmp_ftay =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
                                     * ( - rx * scal_prod_over_rsqr );


                    }


                    /* summing forces */
                    if(Injured[i]==0) {
                        fcolx[i] = tmp_fpsx + tmp_fyox + tmp_ftax;
                        fcoly[i] = tmp_fpsy + tmp_fyoy + tmp_ftay;
                    } else { /* ie. if Injured[i]==1 */
                        fcolx[i] = tmp_fyox + tmp_ftax;
                        fcoly[i] = tmp_fyoy + tmp_ftay;
                    }


                    /* sum of magnitude of touching forces */
                    if(InjurySwitch==1) {
                        ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                    }
                }
            }
            break;
        }
    }



    /* 1.5 */
    /* injuries */

    switch(InjurySwitch) {
    case 0: {
            break;
        }
    case 1: {

            /* case: people crushed */
            for(i=0; i<N; i++) {

                /* newly injured */
                if((ftmagsum[i]>FCrush_over_1m*PI*D[i])&&(Injured[i]==0)) {
                    Injured[i] = 1;
                    NInjured++;
                    V0of[i] = 0.0;
                }
            }
            break;
        }
    case 2:
    case 3: {

            /* case: smoke front */
            if(SimTime[UpdNum]>=SmokeStartTime) {
                x_smokefront = (SimTime[UpdNum]-SmokeStartTime)*VSmoke;

                for(i=0; i<N; i++) {
                    /* checking position compared to smoke front */
                    tmpr = X[i] - x_smokefront;

                    /* center of particle behind smoke front: injured */
                    if( tmpr < 0.5*D[i] ) {
                        if(Injured[i]==0) {
                            Injured[i] = 1;
                            NInjured++;
                            V0of[i] = 0.0;
                            VX[i] = VY[i] = 0.0;
                        }
                    }
                    /* ahead of front but within its interaction range:
                    trying to escape */
                    if( (tmpr>=0.5*D[i])&&(tmpr<=R) ) {
                        tmpf = A_fire*exp(-(tmpr-0.5*D[i])/B_fire);
                        fsmokex[i] += cos(Phi[i])*tmpf;
                        fsmokey[i] += sin(Phi[i])*tmpf;
                    }
                }
            }
            break;
        }
    }



    /* 2 */

    /* 2.1 preparing update of the eq. of motion */

    sqrt_fact = sqrt(tstep/DefaultDeltaT);
    for(i=0; i<N; i++) {

        /* self-propelling */
        fspx[i] = 1/Tau * (V0of[i]*cos(Phi[i]) - VX[i]);
        fspy[i] = 1/Tau * (V0of[i]*sin(Phi[i]) - VY[i]);

        /* noise */
        if(GaTh!=0.0) {
            ksi = GaussRand(GaMe, GaTh, GaCM);
            eta = 2.0*PI * rand() / (RAND_MAX+1.0);
        } else {
            ksi=0.0;
            eta=0.0;
        }


        /* sum of forces */
        fsumx[i] =   fspx[i] + fpairx[i] + fwallx[i] + fwpointx[i]
                     + sqrt_fact * ksi * cos(eta);
        fsumy[i] =   fspy[i] + fpairy[i] + fwally[i] + fwpointy[i]
                     + sqrt_fact * ksi * sin(eta);


        /* adding smoke force */
        if((InjurySwitch==2)||(InjurySwitch==3)) {
            fsumx[i] += fsmokex[i];
            fsumy[i] += fsmokey[i];
        }
        /* adding force of column */
        switch(ColumnSwitch) {
        default:
        case 0: {
                break;
            }
        case 1: {
                fsumx[i] += fcolx[i];
                fsumy[i] += fcoly[i];
                break;
            }
        }


        /* time step adjustment for velocity change */
        EulTStep( &tstep, sqrt(SQR(fsumx[i])+SQR(fsumy[i])) );


        /* new velocity */
        if(  (Injured[i]==1)
                &&((InjurySwitch==1)||(InjurySwitch==3))
          ) {
            vxnew[i] = 0.0;
            vynew[i] = 0.0;
        } else {
            vxnew[i] = VX[i] + fsumx[i] * tstep;
            vynew[i] = VY[i] + fsumy[i] * tstep;
        }


        /* checking new velocity */
        vnew = sqrt( SQR(vxnew[i]) + SQR(vynew[i]) );
        if(vnew > Vmax) {
            vxnew[i] = vxnew[i]/vnew * Vmax;
            vynew[i] = vynew[i]/vnew * Vmax;
        }
    }




    /* 3 */
    float *Xprev_d, *Yprev_d, *X_d, *Y_d, *VX_d, *VY_d;
    int sizeFloatVector = N * sizeof(float);

    hipError_t error = hipMalloc (&Xprev_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


    error = hipMalloc (&Yprev_d, sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


    error = hipMalloc (&X_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


    error = hipMalloc (&Y_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&VY_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&VX_d,sizeFloatVector);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // alte Werte hochkopieren
    error = hipMemcpy(X_d, X, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Y_d, Y, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(VY_d, VY, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(VX_d, VX, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


    storeOldValues <<<1, 200>>> (Xprev_d,X_d,Yprev_d,Y_d);
    hipDeviceSynchronize();


    calcNewValues <<<1, N>>> (X_d,Y_d,VY_d,VX_d,tstep);
    hipDeviceSynchronize();

    // neue Werte zurückkopieren
    error = hipMemcpy(Xprev, Xprev_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Yprev, Yprev_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(X, X_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Y, Y_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    for(i=0; i<N; i++) {

        /* .1 wurde ersetzt durch storeOldValues */
        // Xprev[i] = X[i];
        // Yprev[i] = Y[i];

        /* .2 wurde erstzt durch calcNewValues*/
        // X[i] += VX[i] * tstep;
        // Y[i] += VY[i] * tstep;

        if((Xprev[i]>RoomXSize)&&(X[i]<=RoomXSize)) {
            NInRoom++;
        }
        if((Xprev[i]<=RoomXSize)&&(X[i]>RoomXSize)) {
            NInRoom--;

        }
    }


    /* .3 and .4 */
    for(i=0; i<N; i++) {

        /* (a) if the particle is on the board, its book-keeping
           arrays are modified only if its block has changed during
           the last update
           (b) if the particle is off-board, it will be removed */


        /* a */
        if(X[i]<XS) {
            j_old =   (int)floor(Xprev[i]*GX/XS)
                      + G*(int)floor(Yprev[i]*GY/YS);
            j_new = (int)floor(X[i]*GX/XS) + G*(int)floor(Y[i]*GY/YS);
            if( j_new != j_old ) {

                /* deleting particle i from its old block */
                j = j_old;
                if(BIndBd[j]==i) {
                    BIndBd[j] = BInd[i];
                } else {
                    j = BIndBd[j];
                    while(BInd[j]!=i) {
                        j = BInd[j];
                    }
                    BInd[j] = BInd[i];
                }


                /* inserting particle i into its new block */
                j = j_new;
                if(BIndBd[j]==-1) {
                    BIndBd[j] = i;
                    BInd[i] = -1;
                } else {
                    j = BIndBd[j];
                    while(BInd[j]!=-1) {
                        j = BInd[j];
                    }
                    BInd[j] = i;
                    BInd[i] = -1;
                }
            }
        } else {
            RemoveParticle( &N, i );
            i--;
        }
    }



    /* 4 */

    /* 4.1 */
    E[UpdNum+1] = 0.0;
    for(i=0; i<N; i++) {
        E[UpdNum+1] += VX[i] * cos(Phi[i]) + VY[i] * sin(Phi[i]);
    }
    if(N>0) {
        E[UpdNum+1] /= N;
    }


    /* 4.2 */
    for(i=0; i<N; i++) {
        VX[i] = vxnew[i];
        VY[i] = vynew[i];
        V[i] = sqrt(SQR(VX[i])+SQR(VY[i]));
        Vdir[i] = atan2(VY[i],VX[i]);
        Phi[i] = DirectionOfExit( i );
    }


    /* 4.3 */
    SimTime[UpdNum+1] = SimTime[UpdNum] + tstep;
    UpdNum++;


    /*
    if(NInjured>0){
    fprintf(stdout,"t[%d]=%g\n",UpdNum,SimTime[UpdNum]);
    fflush(stdout);
    }
    */


    /* 5 */
    free_vector(fwallx,0,allocN-1);
    free_vector(fwally,0,allocN-1);
    free_vector(fwpointx,0,allocN-1);
    free_vector(fwpointy,0,allocN-1);
    free_vector(fpairx,0,allocN-1);
    free_vector(fpairy,0,allocN-1);
    free_vector(fsmokex,0,allocN-1);
    free_vector(fsmokey,0,allocN-1);

    free_vector(fspx,0,allocN-1);
    free_vector(fspy,0,allocN-1);
    free_vector(fsumx,0,allocN-1);
    free_vector(fsumy,0,allocN-1);
    free_vector(vxnew,0,allocN-1);
    free_vector(vynew,0,allocN-1);
    free_vector(ftmagsum,0,allocN-1);
    free_vector(fcolx,0,allocN-1);
    free_vector(fcoly,0,allocN-1);


    /* 6 */
}
