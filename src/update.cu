#include "hip/hip_runtime.h"
#pragma once

#include <glog/logging.h>
#include <stdio.h>

#include "types.h"
#include "update.h"
#include "prepareParameter.h"


 /* wall force */
__global__ void calcWallForces (float *fwallx, float *fwally, float *ftmagsum, float *D, int *Injured, float *X, float *Y, wpoint *WP, float *VX, float *VY, parameter *para) {
	
	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
	int iw;
	int N = 200; // für den ersten Test reicht das, später muß es mit übergeben werden 
	int Nw = 9; // für den ersten Test reicht das, später muß es mit übergeben werden 
	float R = para->R; 
	int InjurySwitch = para->InjurySwitch;
	
	int can_see;
	float tmpr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay ;
	
	if (i <= N) {
        for(iw=0; iw<Nw; iw++) {
			
            WallParticleRelation(iw,i,&tmpr,&can_see,Y[i],X[i],WP,para);
			
            if((can_see==1)&&(tmpr<=R)) {

                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* psychological force */
				
                WallPsychForce(iw,i,tmpr,&tmp_fpsx,&tmp_fpsy, D[i], para);
                /* Young and tangential forces */
                if(tmpr<=0.5*D[i]) {
                    
					WallYoungForce(iw,i,tmpr,&tmp_fyox,&tmp_fyoy, D[i], para);
				
                    WallTangForce_FS1(iw,i,tmpr, &tmp_ftax, &tmp_ftay, D[i], VX[i], VY[i], para);

                }
                /* summing wall forces */
                if(Injured[i]==0) {
                    fwallx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fwally[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fwallx[i] += tmp_fyox + tmp_ftax;
                    fwally[i] += tmp_fyoy + tmp_ftay;
                }

                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }

                // measuring x component of touching force exerted on walls left and right from exit 
				/* lasse ich erstmal weg
                if((iw==1)||(iw==7)) {
                    FW_x -= tmp_fyox + tmp_ftax;
                }
				*/
            }
        }
    }
}

__global__ void calcWPointForces (float *fwpointx, float *fwpointy, float *ftmagsum, float *D, int *Injured, float *X, float *Y, wpoint *WP, float *VX, float *VY, parameter *para) {
	
	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
	int N = 200; // für den ersten Test reicht das, später muß es mit übergeben werden 
	int Nwp = 4; // für den ersten Test reicht das, später muß es mit übergeben werden 
	
	float R = para->R; 
	int InjurySwitch = para->InjurySwitch;
	int iwp;
	int can_see;
	float tmpr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay;
	
	if (i <=  N) {
        for(iwp=0; iwp<Nwp; iwp++) {
			
			
			
            WPointParticleRelation(iwp,i,&tmpr,&can_see, Y[i], X[i], WP);
            if((can_see==1)&&(tmpr<=R)) {

                /* init */
                tmp_fpsx = tmp_fpsy = 0.0;
                tmp_fyox = tmp_fyoy = 0.0;
                tmp_ftax = tmp_ftay = 0.0;

                /* computing forces */
                WPointPsychForce(iwp,i,tmpr,&tmp_fpsx,&tmp_fpsy, X[i], Y[i], D[i], WP[iwp], para);
                if(tmpr<=0.5*D[i]) {
                    
					WPointYoungForce(iwp,i,tmpr,&tmp_fyox,&tmp_fyoy, X[i], Y[i], D[i], WP[iwp], para);

					WPointTangForce_FS1(iwp,i,tmpr,&tmp_ftax,&tmp_ftay, X[i], Y[i], D[i], VX[i], VY[i], WP[iwp], para);

                }

                /* summing forces */
                if(Injured[i]==0) {
                    fwpointx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    fwpointy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                } else { /* ie. if Injured[i]=1 */
                    fwpointx[i] += tmp_fyox + tmp_ftax;
                    fwpointy[i] += tmp_fyoy + tmp_ftay;
                }

                /* sum of magnitude of touching forces */
                if(InjurySwitch==1) {
                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                }

                // measuring x component of touching force exerted on walls left and right from exit
                // erstmal rausgenommen   
				//
                // if((iwp==0)||(iwp==3)) {
                //    FW_x -= tmp_fyox + tmp_ftax;
                // }

            }
        }
    }
}

__global__ void calcColumnForces (float *fcolx, float *fcoly, float *ftmagsum,float *D, int *Injured, float *X, float *Y, float *VX, float *VY, parameter *para) {
    
	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
	
	int N = 200; // für den ersten Test reicht das, später muß es mit übergeben werden 
	
	int InjurySwitch = para->InjurySwitch;
	int ColumnSwitch = para->ColumnSwitch;
	float ColumnCenterX = para-> ColumnCenterX;
	float ColumnCenterY = para-> ColumnCenterY;
	float ColumnD = para-> ColumnD;
	
	float A = para-> A;
	float B = para-> B;
	float C_Young = para-> C_Young;
	float Kappa = para-> Kappa;
	
	float R = para -> R;
		
	// lokale Variable
	float tmprsqr, tmp_fpsx, tmp_fpsy, tmp_fyox, tmp_fyoy, tmp_ftax, tmp_ftay, rx, ry, f_over_r, scal_prod_over_rsqr, tmpr;
	
	/* 1.4
     * column
     */
	if (i <= N) {
	
		switch(ColumnSwitch) {
		default:
		case 0: {
				// for(i=0; i<N; i++) {
					fcolx[i] = fcoly[i] = 0.0;
				// }
				 break;
			}
		case 1: {
				// for(i=0; i<N; i++) {
					tmprsqr = SQR(X[i]-ColumnCenterX)+SQR(Y[i]-ColumnCenterY);
					if(tmprsqr<=SQR(R)) {
						tmpr=sqrt(tmprsqr);

						/* init */
						tmp_fpsx = tmp_fpsy = 0.0;
						tmp_fyox = tmp_fyoy = 0.0;
						tmp_ftax = tmp_ftay = 0.0;

						/* computing forces */
						/* psychological */
						f_over_r = A * exp(-(tmpr-0.5*(D[i]+ColumnD))/B) / tmpr;
						tmp_fpsx = (X[i]-ColumnCenterX) * f_over_r;
						tmp_fpsy = (Y[i]-ColumnCenterY) * f_over_r;
						/* touching */
						if(tmpr<=0.5*(D[i]+ColumnD)) {
							/* Young */
							f_over_r = 2.0*C_Young*(0.5*(D[i]+ColumnD)-tmpr) / tmpr;
							tmp_fyox = (X[i]-ColumnCenterX) * f_over_r;
							tmp_fyoy = (Y[i]-ColumnCenterY) * f_over_r;
							/* friction */
							rx = X[i]-ColumnCenterX;
							ry = Y[i]-ColumnCenterY;
							scal_prod_over_rsqr = (ry*VX[i] - rx*VY[i]) / SQR(tmpr);

							tmp_ftax =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
										 * (   ry * scal_prod_over_rsqr );
							tmp_ftay =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
										 * ( - rx * scal_prod_over_rsqr );


						}


						/* summing forces */
						if(Injured[i]==0) {
							fcolx[i] = tmp_fpsx + tmp_fyox + tmp_ftax;
							fcoly[i] = tmp_fpsy + tmp_fyoy + tmp_ftay;
						} else { /* ie. if Injured[i]==1 */
							fcolx[i] = tmp_fyox + tmp_ftax;
							fcoly[i] = tmp_fyoy + tmp_ftay;
						}


						/* sum of magnitude of touching forces */
						if(InjurySwitch==1) {
							ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
						}
					}
				// }
				break;
			}
		}
	}
}

__global__ void calcInjuryForces (float *fsmokex, float *fsmokey, float *VX, float *VY, float *V0of, int *Injured, float *ftmagsum, int N, int UpdNum, float *SimTime, float *Phi, float *X, float *D, parameter *para){

	// lokale Variablen
	float x_smokefront, tmpf,tmpr;
	
	int InjurySwitch = para -> InjurySwitch; 
	float FCrush_over_1m = para -> FCrush_over_1m;
	float SmokeStartTime = para -> SmokeStartTime;
	float VSmoke = para -> VSmoke;
	float R = para -> R ;
	float A_fire = para -> A_fire;
	float B_fire = para -> B_fire;
	
	int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
	
	if (i <= N) {
	
		switch(InjurySwitch) {
		case 0: {
				break;
			}
		case 1: {

				/* case: people crushed */
				// for(i=0; i<N; i++) {

					// frisch verletzt
					if((ftmagsum[i]>FCrush_over_1m*PI*D[i])&&(Injured[i]==0)) {
						Injured[i] = 1;
						// NInjured++; wird anschließend neu berechnet
						V0of[i] = 0.0;
					}
				// }
				break;
			}
		case 2:
		case 3: {

				/* case: smoke front */
				if(SimTime[UpdNum]>=SmokeStartTime) {
					x_smokefront = (SimTime[UpdNum]-SmokeStartTime)*VSmoke;

					//for(i=0; i<N; i++) {
						/* checking position compared to smoke front */
						tmpr = X[i] - x_smokefront;

						/* center of particle behind smoke front: injured */
						if( tmpr < 0.5*D[i] ) {
							if(Injured[i]==0) {
								Injured[i] = 1;
								// NInjured++; wird anschließend neu berechnet
								V0of[i] = 0.0;
								VX[i] = VY[i] = 0.0;
							}
						}
						/* ahead of front but within its interaction range:
						trying to escape */
						if( (tmpr>=0.5*D[i])&&(tmpr<=R) ) {
							tmpf = A_fire*exp(-(tmpr-0.5*D[i])/B_fire);
							fsmokex[i] += cos(Phi[i])*tmpf;
							fsmokey[i] += sin(Phi[i])*tmpf;
						}
					// }
				}
				break;
			}
		}
	}
}

__global__ void storeOldValues (float* Xprev_d,float* X_d, float* Yprev_d, float*Y_d)
{

    int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
    // printf ("%d\n", i);
	int N = 200;
    if (i <= N) {
        Xprev_d[i] = X_d[i];
        Yprev_d[i] = Y_d[i];
    }
}

__global__ void calcNewValues (float* X_d,float* Y_d,float* VY_d,float* VX_d, float tstep)
{
    int b_ID = blockIdx.x; 		   
	int i =  b_ID * blockDim.x + threadIdx.x;
	
	
    int N = 200;
	
	if (i <= N) {
        X_d[i] += VX_d[i] * tstep;
        Y_d[i] += VY_d[i] * tstep;
    }
}

__global__ void sumUp (const int *summanden, const int countElements, int* sum) {
	float summe = 0;
	int i;
	for (i = 0; i < countElements; i++) {
		summe = summe + summanden[i];
	}
	*sum = summe;
}

void __device__ WallParticleRelation(int iw, int i, float *r, int *can_see, float yCoor, float xCoor, wpoint *WP, parameter *para)
{
    // can_see: whether partice i is within the range of wall iw;  r: distance
	
	float RoomYSize = para-> RoomYSize;
	float R = para -> R;

    switch(iw) {
    case 0: {
            *r = yCoor;
            break;
        }
    case 1: {
            *r = WP[0].x -xCoor;
            break;
        }
    case 2: {
            *r = yCoor-WP[0].y;
            break;
        }
    case 3: {
            *r = xCoor-WP[1].x;
            break;
        }
    case 4: {
            *r = RoomYSize-yCoor;
            break;
        }
    case 5: {
            *r = xCoor-WP[2].x;
            break;
        }
    case 6: {
            *r = WP[2].y-yCoor;
            break;
        }
    case 7: {
            *r = WP[3].x-xCoor;
            break;
        }
    case 8: {
            *r = xCoor;
            break;
        }
    }


    switch(iw) {
    case 0: {
            if(yCoor<=R) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 1: {
            if((xCoor>=WP[0].x-R)&&(xCoor<=WP[0].x)&&(yCoor<=WP[0].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 2: {
            if((xCoor>=WP[0].x)&&(xCoor<=WP[1].x)&&(yCoor<=WP[0].y+R)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 3: {
            if((xCoor>=WP[1].x)&&(xCoor<=WP[1].x+R)&&(yCoor<=WP[1].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 4: {
            if(yCoor>=RoomYSize-R) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 5: {
            if((xCoor>=WP[2].x)&&(xCoor<=WP[2].x+R)&&(yCoor>=WP[2].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 6: {
            if((xCoor>=WP[3].x)&&(xCoor<=WP[2].x)&&(yCoor>=WP[2].y-R)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 7: {
            if((xCoor<=WP[3].x)&&(xCoor>=WP[3].x-R)&&(yCoor>=WP[3].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 8: {
            if(xCoor<=R) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    }
}




void __device__ WallTangForce_FS1( int iw, int i, float r, float *fx, float *fy, float diameter, float VelocityX_Dir, float VelocityY_Dir, parameter *para )
{
float Kappa = para-> Kappa;
#define tmp_delta_r (0.5*diameter-r)

    /* friction forces */
    switch(iw) {
    case 0:
    case 2:
    case 4:
    case 6: {
            *fx = -Kappa*tmp_delta_r*VelocityX_Dir;
            *fy = 0.0;
            break;
        }
    case 1:
    case 3:
    case 5:
    case 7:
    case 8: {
            *fx = 0.0;
            *fy = -Kappa*tmp_delta_r*VelocityY_Dir;
            break;
        }
    }

#undef tmp_delta_r
}

void __device__ WallPsychForce(int iw, int i, float r, float *fx, float *fy, float diameter, parameter *para)
{
	float A = para-> A;
	float B = para -> B;
#define tmp_f (A*exp(-(r-0.5*diameter)/B))

    switch(iw) {
    case 0: {
            *fx = 0.0;
            *fy = tmp_f;
            break;
        }
    case 1: {
            *fx = - tmp_f;
            *fy = 0.0;
            break;
        }
    case 2: {
            *fx = 0.0;
            *fy = tmp_f;
            break;
        }
    case 3: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    case 4: {
            *fx = 0.0;
            *fy = - tmp_f;
            break;
        }
    case 5: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    case 6: {
            *fx = 0.0;
            *fy = - tmp_f;
            break;
        }
    case 7: {
            *fx = - tmp_f;
            *fy = 0.0;
            break;
        }
    case 8: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    }

#undef tmp_f
}

void __device__ WallYoungForce(int iw, int i, float r, float *fx, float *fy, float diameter, parameter *para)
{

float C_Young = para-> C_Young;
#define tmp_f (2.0*C_Young*(0.5*diameter-r))

    switch(iw) {
    case 0: {
            *fx = 0.0;
            *fy = tmp_f;
            break;
        }
    case 1: {
            *fx = - tmp_f;
            *fy = 0.0;
            break;
        }
    case 2: {
            *fx = 0.0;
            *fy = tmp_f;
            break;
        }
    case 3: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    case 4: {
            *fx = 0.0;
            *fy = - tmp_f;
            break;
        }
    case 5: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    case 6: {
            *fx = 0.0;
            *fy = - tmp_f;
            break;
        }
    case 7: {
            *fx = - tmp_f;
            *fy = 0.0;
            break;
        }
    case 8: {
            *fx = tmp_f;
            *fy = 0.0;
            break;
        }
    }

#undef tmp_f
}

void __device__ WPointYoungForce(int iwp, int i, float r, float *fx, float *fy, float xCoor, float yCoor, float diameter, wpoint WP, parameter *para )

{
    /* exerted by wpoint iwp on particle i */
	float C_Young = para-> C_Young;
    float rx,ry;

#define tmp_f_over_r ( 2.0*C_Young*(0.5*diameter-r) / r)

    rx=WP.x-xCoor;
    ry=WP.y-yCoor;
    *fx = - rx * tmp_f_over_r;
    *fy = - ry * tmp_f_over_r;

#undef tmp_f_over_r
}

void __device__ WPointPsychForce(int iwp, int i, float r, float *fx, float *fy, float xCoor, float yCoor, float diameter, wpoint WP, parameter *para )

{
    /* exerted by wpoint iwp on particle i */
	float A = para-> A;
	float B = para-> B;
#define tmp_f_over_r (A*exp(-(r-0.5*diameter)/B)/r)

    *fx = (xCoor-WP.x) * tmp_f_over_r;
    *fy = (yCoor-WP.y) * tmp_f_over_r;

#undef tmp_f_over_r
}


void __device__ WPointParticleRelation(int iwp, int i, float *r, int *can_see, float yCoor, float xCoor, wpoint *WP)
{
    /* can_see: whether partice i is within the range of wpoint iwp r: distance */
	
    *r = sqrt(SQR(WP[iwp].x-xCoor)+SQR(WP[iwp].y-yCoor));

    switch(iwp) {
    case 0: {
            if((xCoor<=WP[0].x)&&(yCoor>=WP[0].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 1: {
            if((xCoor>=WP[1].x)&&(yCoor>=WP[1].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 2: {
            if((xCoor>=WP[2].x)&&(yCoor<=WP[2].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    case 3: {
            if((xCoor<=WP[3].x)&&(yCoor<=WP[3].y)) {
                *can_see=1;
            } else {
                *can_see=0;
            }
            break;
        }
    }
}

void __device__ WPointTangForce_FS1(int iwp, int i, float r, float *fx, float *fy, float xCoor, float yCoor, float diameter, float VelocityX_Dir, float VelocityY_Dir, wpoint WP, parameter *para )

{
	/* exerted by wpoint iwp on particle i */
	float Kappa = para-> Kappa;
    float rx,ry,scal_prod_over_rsqr;

    rx = xCoor-WP.x;
    ry = yCoor-WP.y;
    scal_prod_over_rsqr = (ry*VelocityX_Dir - rx*VelocityY_Dir) / SQR(r);
    *fx = -Kappa * (0.5*diameter-r) * (   ry * scal_prod_over_rsqr );
    *fy = -Kappa * (0.5*diameter-r) * ( - rx * scal_prod_over_rsqr );
}

void Upd(parameter para_h)
{
	
	
    int allocN,i,j,k,l,mx,my,m,j_old,j_new;
    float *fwallx,*fwally,*fwpointx,*fwpointy,*fpairx,*fpairy,
          *fspx,*fspy,*fsumx,*fsumy,*vxnew,*vynew,tstep,tmpr,
          tmp_fpsx,tmp_fpsy,tmp_fyox,tmp_fyoy,tmp_ftax,tmp_ftay,
          tmprsqr,sqrt_fact,ksi,eta,vnew,*ftmagsum,*fsmokex,*fsmokey,
          x_smokefront, tmpf, f_over_r, rx, ry, scal_prod_over_rsqr, 
		  *fcolx,*fcoly;



    /* 0 */
    allocN=N;
    fwallx=vector(0,allocN-1);
    fwally=vector(0,allocN-1);
    
	fwpointx=vector(0,allocN-1);
    fwpointy=vector(0,allocN-1);
    
	fpairx=vector(0,allocN-1);
    fpairy=vector(0,allocN-1);
    
	fsmokex=vector(0,allocN-1);
    fsmokey=vector(0,allocN-1);

    fspx=vector(0,allocN-1);
    fspy=vector(0,allocN-1);
    fsumx=vector(0,allocN-1);
    fsumy=vector(0,allocN-1);
    vxnew=vector(0,allocN-1);
    vynew=vector(0,allocN-1);
    ftmagsum=vector(0,allocN-1);
    fcolx=vector(0,allocN-1);
    fcoly=vector(0,allocN-1);

	// printf("alle normalen Pointer wurden alloziert.\n");
	
	
	
	float *Xprev_d, *Yprev_d, *X_d, *Y_d, *VX_d, *VY_d, *fwallx_d, *fwally_d,*fwpointx_d, *fwpointy_d, *ftmagsum_d, *D_d, *fcolx_d, *fcoly_d, *fsmokex_d, *fsmokey_d, *V0of_d, *SimTime_d, *Phi_d;
	int *Injured_d;
	int *NInjured_d; 
	
	wpoint *WP_d;
	
	parameter *para_d;
	 
	
	
	
	
	int sizeFloatVector = N * sizeof(float);

	hipError_t error; 
    
	error = hipMalloc (&NInjured_d,sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Anzahl der Verletzten
	error = hipMalloc (&Injured_d,N * sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&WP_d,NWP * sizeof(wpoint)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&para_d,sizeof(parameter)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMalloc (&fwpointx_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&fwpointy_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&fwallx_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&fwally_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&ftmagsum_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&D_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMalloc (&fcolx_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&fcoly_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&fsmokex_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&fsmokey_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&V0of_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&SimTime_d, sizeFloatVector);CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&Xprev_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Yprev_d, sizeFloatVector);CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&X_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Y_d,sizeFloatVector);CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&VY_d,sizeFloatVector);CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&VX_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

	error = hipMalloc (&Phi_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	LOG (INFO) << "Device Pointer alloziert.";
	
	
    /* 1 */

    /* 1.0 */
    /* default values */
    tstep = DefaultDeltaT;
    for(i=0; i<N; i++) {
        fwallx[i] = 0.0;
        fwally[i] = 0.0;
        fwpointx[i] = 0.0;
        fwpointy[i] = 0.0;
        fpairx[i] = 0.0;
        fpairy[i] = 0.0;
        fsmokex[i] = 0.0;
        fsmokey[i] = 0.0;

        fspx[i] = 0.0;
        fspy[i] = 0.0;
        fsumx[i] = 0.0;
        fsumy[i] = 0.0;
        ftmagsum[i] = 0.0;
        fcolx[i] = 0.0;
        fcoly[i] = 0.0;
    }
    FW_x=0.0;
	

	// temporäre Kraftvektoren auf dem Device mit 0 initialsieren
	
	
	error = hipMemset(fwallx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemset(fwally_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMemset(fcolx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemset(fcoly_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

	error = hipMemset(fsmokex_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemset(fsmokey_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMemset(fwpointx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemset(fwpointy_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMemset(ftmagsum_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	// error = hipMemcpy(fwallx_d, fwallx, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(fwally_d, fwally, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	// error = hipMemcpy(fcolx_d, fcolx, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(fcoly_d, fcoly, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	// error = hipMemcpy(fwpointx_d, fwpointx, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(fwpointy_d, fwpointy, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	// error = hipMemcpy(ftmagsum_d, ftmagsum, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	LOG (INFO) << "device kraftvektoren initialisiert";
	
	
	// nötige Werte hochkopieren
	
	error = hipMemcpy(D_d, D, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(Injured_d, Injured, N * sizeof(int), hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

	error = hipMemcpy(X_d, X, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(Y_d, Y, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
    error = hipMemcpy(VY_d, VY, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(VX_d, VX, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMemcpy(WP_d, WP, NWP * sizeof(wpoint), hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMemcpy(para_d, &para_h, sizeof(parameter), hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    	
	LOG (INFO) << "Alle Werte wurden hochkopiert";
	
	
	dim3 dimBlock(32); 
	dim3 dimGrid((para_h.N0 + dimBlock.x - 1) / dimBlock.x);
	
	LOG(WARNING) << "Block dimensions: " << dimBlock.x << " " << dimBlock.y << " " << dimBlock.z;
	LOG(WARNING) << "Grid dimensions: " << dimGrid.x << " " << dimGrid.y << " " << dimGrid.z;
	
	
	calcWallForces<<<dimGrid, dimBlock>>> (fwallx_d, fwally_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, WP_d, VX_d, VY_d, para_d); 
	hipDeviceSynchronize();
	// berechnete Werte calcWallForces zurück
	error = hipMemcpy(fwallx, fwallx_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(fwally, fwally_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	// ftmagsum wird im nächsten Kernel nochmal verwendet, darum erst danach zurück
	
	
	
	calcWPointForces <<<dimGrid, dimBlock>>> (fwpointx_d, fwpointy_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, WP_d, VX_d, VY_d, para_d);
	hipDeviceSynchronize();
	// berechnete Werte calcWPointForces zurück
	error = hipMemcpy(fwpointx, fwpointx_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(fwpointy, fwpointy_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(ftmagsum, ftmagsum_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


	
    /* 1.1 */
    /* wall force */
    // for(i=0; i<N; i++) {
        // for(iw=0; iw<NW; iw++) {

            // WallParticleRelation(iw,i,&tmpr,&can_see);
            // if((can_see==1)&&(tmpr<=R)) {

                // /* init */
                // tmp_fpsx = tmp_fpsy = 0.0;
                // tmp_fyox = tmp_fyoy = 0.0;
                // tmp_ftax = tmp_ftay = 0.0;

                // /* psychological force */
                // WallPsychForce(iw,i,tmpr,&tmp_fpsx,&tmp_fpsy);
                // /* Young and tangential forces */
                // if(tmpr<=0.5*D[i]) {
                    // WallYoungForce(iw,i,tmpr,&tmp_fyox,&tmp_fyoy);

                    // WallTangForce_FS1(iw,i,tmpr,&tmp_ftax,&tmp_ftay);

                // }
                // /* summing wall forces */
                // if(Injured[i]==0) {
                    // fwallx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    // fwally[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                // } else { /* ie. if Injured[i]=1 */
                    // fwallx[i] += tmp_fyox + tmp_ftax;
                    // fwally[i] += tmp_fyoy + tmp_ftay;
                // }

                // /* sum of magnitude of touching forces */
                // if(InjurySwitch==1) {
                    // ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                // }

                // /* measuring x component of touching force exerted
                // on walls left and right from exit
                 // -- only in demo mode */

                // if((iw==1)||(iw==7)) {
                    // FW_x -= tmp_fyox + tmp_ftax;
                // }
            // }
        // }
    // }



    /* 1.2 */
    /* wpoint force */
    // for(i=0; i<N; i++) {
        // for(iwp=0; iwp<NWP; iwp++) {

            // WPointParticleRelation(iwp,i,&tmpr,&can_see);
            // if((can_see==1)&&(tmpr<=R)) {

                // /* init */
                // tmp_fpsx = tmp_fpsy = 0.0;
                // tmp_fyox = tmp_fyoy = 0.0;
                // tmp_ftax = tmp_ftay = 0.0;

                // /* computing forces */
                // WPointPsychForce(iwp,i,tmpr,&tmp_fpsx,&tmp_fpsy);
                // if(tmpr<=0.5*D[i]) {
                    // WPointYoungForce(iwp,i,tmpr,&tmp_fyox,&tmp_fyoy);

                    // WPointTangForce_FS1(iwp,i,tmpr,&tmp_ftax,&tmp_ftay);

                // }

                // /* summing forces */
                // if(Injured[i]==0) {
                    // fwpointx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    // fwpointy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                // } else { /* ie. if Injured[i]=1 */
                    // fwpointx[i] += tmp_fyox + tmp_ftax;
                    // fwpointy[i] += tmp_fyoy + tmp_ftay;
                // }

                // /* sum of magnitude of touching forces */
                // if(InjurySwitch==1) {
                    // ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                // }

                // /* measuring x component of touching force exerted
                   // on walls left and right from exit
                   // -- only in demo mode */

                // if((iwp==0)||(iwp==3)) {
                    // FW_x -= tmp_fyox + tmp_ftax;
                // }

            // }
        // }
    // }



    /* 1.3 */
    /* particle-particle forces */
    for(i=0; i<N; i++) {

        j = (int)floor(X[i]*GX/XS) + G * (int)floor(Y[i]*GY/YS);
        for(k=-1; k<=1; k++) {
            for(l=-1; l<=1; l++) {

                mx = j%G+k;
                my = j/G+l;
                if((mx>=0)&&(mx<GX)&&(my>=0)&&(my<GY)) {

                    m = BIndBd[ (mx+GX)%GX + G * (my%GY) ];
                    /* checking each pair of particles only once */
                    while(m>=i) {
                        m = BInd[m];
                    }
                    if(m!=-1) {
                        do {

                            tmprsqr = SQR(X[i]-X[m]) + SQR(Y[i]-Y[m]);
                            if( tmprsqr <= SQR(R) ) {
                                tmpr = sqrt(tmprsqr);

                                /* init */
                                tmp_fpsx = tmp_fpsy = 0.0;
                                tmp_fyox = tmp_fyoy = 0.0;
                                tmp_ftax = tmp_ftay = 0.0;

                                /* pair forces */
                                /* Force(i,m,...) gives the force exerted by m
                                on i, all forces are symmetric now */
                                PP_PsychForce(i,m,tmpr,&tmp_fpsx,&tmp_fpsy);
                                if(tmpr<=0.5*(D[i]+D[m])) {
                                    PP_YoungForce(i,m,tmpr,&tmp_fyox,&tmp_fyoy);
                                    PP_TangForce_FS1(i,m,tmpr,&tmp_ftax,&tmp_ftay);

                                }

                                /* summing forces */
                                if(Injured[i]==0) {
                                    fpairx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                                    fpairy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                                } else { /* ie. if Injured[i]=1 */
                                    fpairx[i] += tmp_fyox + tmp_ftax;
                                    fpairy[i] += tmp_fyoy + tmp_ftay;
                                }
                                if(Injured[m]==0) {
                                    fpairx[m] -= tmp_fpsx + tmp_fyox + tmp_ftax;
                                    fpairy[m] -= tmp_fpsy + tmp_fyoy + tmp_ftay;
                                } else { /* ie. if Injured[m]=1 */
                                    fpairx[m] -= tmp_fyox + tmp_ftax;
                                    fpairy[m] -= tmp_fyoy + tmp_ftay;
                                }

                                /* sum of magnitude of touching forces */
                                if(InjurySwitch==1) {
                                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                                    ftmagsum[m] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                                }
                            }

                            m = BInd[m];
                            while(m>=i) {
                                m = BInd[m];
                            }

                        } while(m!=-1);
                    }
                }
            }
        }
    }
	// benötigte Werte für calcColumnForces hochkopieren
	
	error = hipMemcpy(ftmagsum_d, ftmagsum, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	calcColumnForces <<<dimGrid, dimBlock>>> (fcolx_d, fcoly_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, VX_d, VY_d, para_d); 
	CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipGetLastError();
	// printf ("Ergebniss von calcColumnForces: %s \n", hipGetErrorString(error));
	hipDeviceSynchronize();
	
	// berechnete Werte calcColumnForces zurück
	error = hipMemcpy(fcolx, fcolx_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(fcoly, fcoly_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); 
	error = hipMemcpy(ftmagsum, ftmagsum_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); 
	
	
	
	// benötigte Werte für calcInjuryForces hochkopieren 
	
	// error = hipMemcpy(V0of_d, V0of, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(SimTime_d, SimTime, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(Phi_d, Phi, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	
	// calcInjuryForces <<<dimGrid, dimBlock>>> (fsmokex_d, fsmokey_d, VX_d, VY_d, V0of_d, Injured_d,ftmagsum_d, N, UpdNum, SimTime_d, Phi_d, X_d, D_d, para_d);
	// hipDeviceSynchronize ();
	
	
	// berechnete Werte calcInjuryForces zurück
	
	
	// error = hipMemcpy(fsmokex, fsmokex_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(fsmokey, fsmokey_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(VX, VX_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(VY, VY_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(V0of, V0of_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(Injured, Injured_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	
	
	// Anzahl der Verletzten neu bestimmen
	// sumUp<<<1,1>>> (Injured_d,N, NInjured_d); hipDeviceSynchronize(); error = hipGetLastError ();  printf ("Ergebniss sumUp : %s \n",hipGetErrorString(error));
	
	// error = hipMemcpy(&NInjured, NInjured_d, sizeof(int), hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
    /* 1.4
     * column
     */
    // switch(ColumnSwitch) {
    // default:
    // case 0: {
            // for(i=0; i<N; i++) {
                // fcolx[i] = fcoly[i] = 0.0;
            // }
            // break;
        // }
    // case 1: {
            // for(i=0; i<N; i++) {
                // tmprsqr = SQR(X[i]-ColumnCenterX)+SQR(Y[i]-ColumnCenterY);
                // if(tmprsqr<=SQR(R)) {
                    // tmpr=sqrt(tmprsqr);

                    // /* init */
                    // tmp_fpsx = tmp_fpsy = 0.0;
                    // tmp_fyox = tmp_fyoy = 0.0;
                    // tmp_ftax = tmp_ftay = 0.0;

                    // /* computing forces */
                    // /* psychological */
                    // f_over_r = A * exp(-(tmpr-0.5*(D[i]+ColumnD))/B) / tmpr;
                    // tmp_fpsx = (X[i]-ColumnCenterX) * f_over_r;
                    // tmp_fpsy = (Y[i]-ColumnCenterY) * f_over_r;
                    // /* touching */
                    // if(tmpr<=0.5*(D[i]+ColumnD)) {
                        // /* Young */
                        // f_over_r = 2.0*C_Young*(0.5*(D[i]+ColumnD)-tmpr) / tmpr;
                        // tmp_fyox = (X[i]-ColumnCenterX) * f_over_r;
                        // tmp_fyoy = (Y[i]-ColumnCenterY) * f_over_r;
                        // /* friction */
                        // rx = X[i]-ColumnCenterX;
                        // ry = Y[i]-ColumnCenterY;
                        // scal_prod_over_rsqr = (ry*VX[i] - rx*VY[i]) / SQR(tmpr);

                        // tmp_ftax =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
                                     // * (   ry * scal_prod_over_rsqr );
                        // tmp_ftay =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
                                     // * ( - rx * scal_prod_over_rsqr );


                    // }


                    // /* summing forces */
                    // if(Injured[i]==0) {
                        // fcolx[i] = tmp_fpsx + tmp_fyox + tmp_ftax;
                        // fcoly[i] = tmp_fpsy + tmp_fyoy + tmp_ftay;
                    // } else { /* ie. if Injured[i]==1 */
                        // fcolx[i] = tmp_fyox + tmp_ftax;
                        // fcoly[i] = tmp_fyoy + tmp_ftay;
                    // }


                    // /* sum of magnitude of touching forces */
                    // if(InjurySwitch==1) {
                        // ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                    // }
                // }
            // }
            // break;
        // }
    // }



    /* 1.5 */
    /* injuries */

    switch(InjurySwitch) {
    case 0: {
            break;
        }
    case 1: {

            /* case: people crushed */
            for(i=0; i<N; i++) {

                /* newly injured */
                if((ftmagsum[i]>FCrush_over_1m*PI*D[i])&&(Injured[i]==0)) {
                    Injured[i] = 1;
                    NInjured++;
                    V0of[i] = 0.0;
                }
            }
            break;
        }
    case 2:
    case 3: {

            /* case: smoke front */
            if(SimTime[UpdNum]>=SmokeStartTime) {
                x_smokefront = (SimTime[UpdNum]-SmokeStartTime)*VSmoke;

                for(i=0; i<N; i++) {
                    /* checking position compared to smoke front */
                    tmpr = X[i] - x_smokefront;

                    /* center of particle behind smoke front: injured */
                    if( tmpr < 0.5*D[i] ) {
                        if(Injured[i]==0) {
                            Injured[i] = 1;
                            NInjured++;
                            V0of[i] = 0.0;
                            VX[i] = VY[i] = 0.0;
                        }
                    }
                    /* ahead of front but within its interaction range:
                    trying to escape */
                    if( (tmpr>=0.5*D[i])&&(tmpr<=R) ) {
                        tmpf = A_fire*exp(-(tmpr-0.5*D[i])/B_fire);
                        fsmokex[i] += cos(Phi[i])*tmpf;
                        fsmokey[i] += sin(Phi[i])*tmpf;
                    }
                }
            }
            break;
        }
    }



    /* 2 */

    /* 2.1 preparing update of the eq. of motion */

    sqrt_fact = sqrt(tstep/DefaultDeltaT);
    for(i=0; i<N; i++) {

        /* self-propelling */
        fspx[i] = 1/Tau * (V0of[i]*cos(Phi[i]) - VX[i]);
        fspy[i] = 1/Tau * (V0of[i]*sin(Phi[i]) - VY[i]);

        /* noise */
        if(GaTh!=0.0) {
            ksi = GaussRand(GaMe, GaTh, GaCM);
            eta = 2.0*PI * rand() / (RAND_MAX+1.0);
        } else {
            ksi=0.0;
            eta=0.0;
        }


        /* sum of forces */
        fsumx[i] =   fspx[i] + fpairx[i] + fwallx[i] + fwpointx[i]
                     + sqrt_fact * ksi * cos(eta);
        fsumy[i] =   fspy[i] + fpairy[i] + fwally[i] + fwpointy[i]
                     + sqrt_fact * ksi * sin(eta);


        /* adding smoke force */
        if((InjurySwitch==2)||(InjurySwitch==3)) {
            fsumx[i] += fsmokex[i];
            fsumy[i] += fsmokey[i];
        }
        /* adding force of column */
        switch(ColumnSwitch) {
        default:
        case 0: {
                break;
            }
        case 1: {
                fsumx[i] += fcolx[i];
                fsumy[i] += fcoly[i];
                break;
            }
        }


        /* time step adjustment for velocity change */
        EulTStep( &tstep, sqrt(SQR(fsumx[i])+SQR(fsumy[i])) );


        /* new velocity */
        if(  (Injured[i]==1)
                &&((InjurySwitch==1)||(InjurySwitch==3))
          ) {
            vxnew[i] = 0.0;
            vynew[i] = 0.0;
        } else {
            vxnew[i] = VX[i] + fsumx[i] * tstep;
            vynew[i] = VY[i] + fsumy[i] * tstep;
        }


        /* checking new velocity */
        vnew = sqrt( SQR(vxnew[i]) + SQR(vynew[i]) );
        if(vnew > Vmax) {
            vxnew[i] = vxnew[i]/vnew * Vmax;
            vynew[i] = vynew[i]/vnew * Vmax;
        }
    }




    /* 3 */
    

    // alte Werte hochkopieren
    error = hipMemcpy(X_d, X, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Y_d, Y, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(VY_d, VY, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(VX_d, VX, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


    storeOldValues <<<dimGrid, dimBlock>>> (Xprev_d,X_d,Yprev_d,Y_d);
    hipDeviceSynchronize();


    calcNewValues <<<dimGrid, dimBlock>>> (X_d,Y_d,VY_d,VX_d,tstep);
    hipDeviceSynchronize();

    // neue Werte zurückkopieren
    error = hipMemcpy(Xprev, Xprev_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Yprev, Yprev_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(X, X_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Y, Y_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    for(i=0; i<N; i++) {

        /* .1 wurde ersetzt durch storeOldValues */
        // Xprev[i] = X[i];
        // Yprev[i] = Y[i];

        /* .2 wurde erstzt durch calcNewValues*/
        // X[i] += VX[i] * tstep;
        // Y[i] += VY[i] * tstep;

        if((Xprev[i]>RoomXSize)&&(X[i]<=RoomXSize)) {
            NInRoom++;
        }
        if((Xprev[i]<=RoomXSize)&&(X[i]>RoomXSize)) {
            NInRoom--;

        }
    }


    /* .3 and .4 */
    for(i=0; i<N; i++) {

        /* (a) if the particle is on the board, its book-keeping
           arrays are modified only if its block has changed during
           the last update
           (b) if the particle is off-board, it will be removed */


        /* a */
        if(X[i]<XS) {
            j_old =   (int)floor(Xprev[i]*GX/XS)
                      + G*(int)floor(Yprev[i]*GY/YS);
            j_new = (int)floor(X[i]*GX/XS) + G*(int)floor(Y[i]*GY/YS);
            if( j_new != j_old ) {

                /* deleting particle i from its old block */
                j = j_old;
                if(BIndBd[j]==i) {
                    BIndBd[j] = BInd[i];
                } else {
                    j = BIndBd[j];
                    while(BInd[j]!=i) {
                        j = BInd[j];
                    }
                    BInd[j] = BInd[i];
                }


                /* inserting particle i into its new block */
                j = j_new;
                if(BIndBd[j]==-1) {
                    BIndBd[j] = i;
                    BInd[i] = -1;
                } else {
                    j = BIndBd[j];
                    while(BInd[j]!=-1) {
                        j = BInd[j];
                    }
                    BInd[j] = i;
                    BInd[i] = -1;
                }
            }
        } else {
            RemoveParticle( &N, i );
            i--;
        }
    }



    /* 4 */

    /* 4.1 */
    E[UpdNum+1] = 0.0;
    for(i=0; i<N; i++) {
        E[UpdNum+1] += VX[i] * cos(Phi[i]) + VY[i] * sin(Phi[i]);
    }
    if(N>0) {
        E[UpdNum+1] /= N;
    }


    /* 4.2 */
    for(i=0; i<N; i++) {
        VX[i] = vxnew[i];
        VY[i] = vynew[i];
        V[i] = sqrt(SQR(VX[i])+SQR(VY[i]));
        Vdir[i] = atan2(VY[i],VX[i]);
        Phi[i] = DirectionOfExit( i );
    }


    /* 4.3 */
    SimTime[UpdNum+1] = SimTime[UpdNum] + tstep;
    UpdNum++;


    /*
    if(NInjured>0){
    fprintf(stdout,"t[%d]=%g\n",UpdNum,SimTime[UpdNum]);
    fflush(stdout);
    }
    */


    /* 5 */
    free_vector(fwallx,0,allocN-1);
    free_vector(fwally,0,allocN-1);
    free_vector(fwpointx,0,allocN-1);
    free_vector(fwpointy,0,allocN-1);
    free_vector(fpairx,0,allocN-1);
    free_vector(fpairy,0,allocN-1);
    free_vector(fsmokex,0,allocN-1);
    free_vector(fsmokey,0,allocN-1);

    free_vector(fspx,0,allocN-1);
    free_vector(fspy,0,allocN-1);
    free_vector(fsumx,0,allocN-1);
    free_vector(fsumy,0,allocN-1);
    free_vector(vxnew,0,allocN-1);
    free_vector(vynew,0,allocN-1);
    free_vector(ftmagsum,0,allocN-1);
    free_vector(fcolx,0,allocN-1);
    free_vector(fcoly,0,allocN-1);


    /* 6 */
}
