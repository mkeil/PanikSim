#include "hip/hip_runtime.h"
#pragma once

#include <glog/logging.h>
#include <stdio.h>

#include "types.h"
#include "kernels.h"
#include "deviceFunc.h"
#include "update.h"
#include "prepareParameter.h"




void Upd(parameter para_h)
{
	
    int allocN,i,j,k,l,mx,my,m,j_old,j_new;
    float *fwallx,*fwally,*fwpointx,*fwpointy,*fpairx,*fpairy,
          *fspx,*fspy,*fsumx,*fsumy,*vxnew,*vynew,tstep,tmpr,
          tmp_fpsx,tmp_fpsy,tmp_fyox,tmp_fyoy,tmp_ftax,tmp_ftay,
          tmprsqr,sqrt_fact,ksi,eta,vnew,*ftmagsum,*fsmokex,*fsmokey,
          x_smokefront, tmpf, f_over_r, rx, ry, scal_prod_over_rsqr, 
		  *fcolx,*fcoly;



    /* 0 */
    allocN=N;
    fwallx=vector(0,allocN-1);
    fwally=vector(0,allocN-1);
    
	fwpointx=vector(0,allocN-1);
    fwpointy=vector(0,allocN-1);
    
	fpairx=vector(0,allocN-1);
    fpairy=vector(0,allocN-1);
    
	fsmokex=vector(0,allocN-1);
    fsmokey=vector(0,allocN-1);

    fspx=vector(0,allocN-1);
    fspy=vector(0,allocN-1);
    fsumx=vector(0,allocN-1);
    fsumy=vector(0,allocN-1);
    vxnew=vector(0,allocN-1);
    vynew=vector(0,allocN-1);
    ftmagsum=vector(0,allocN-1);
    fcolx=vector(0,allocN-1);
    fcoly=vector(0,allocN-1);

	// printf("alle normalen Pointer wurden alloziert.\n");
	
	
	
	float *Xprev_d, *Yprev_d, *X_d, *Y_d, *VX_d, *VY_d, *fwallx_d, *fwally_d,*fwpointx_d, *fwpointy_d, *ftmagsum_d, *D_d, *fcolx_d, *fcoly_d, *fsmokex_d, *fsmokey_d, *V0of_d, *SimTime_d, *Phi_d;
	int *Injured_d;
	int *NInjured_d; 
	
	wpoint *WP_d;
	
	parameter *para_d;
	 
	
	
	
	
	int sizeFloatVector = N * sizeof(float);

	hipError_t error; 
    
	error = hipMalloc (&NInjured_d,sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); // Anzahl der Verletzten
	error = hipMalloc (&Injured_d,N * sizeof(int)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&WP_d,NWP * sizeof(wpoint)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&para_d,sizeof(parameter)); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMalloc (&fwpointx_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&fwpointy_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&fwallx_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&fwally_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&ftmagsum_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&D_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMalloc (&fcolx_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&fcoly_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&fsmokex_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMalloc (&fsmokey_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&V0of_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&SimTime_d, sizeFloatVector);CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMalloc (&Xprev_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Yprev_d, sizeFloatVector);CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&X_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&Y_d,sizeFloatVector);CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMalloc (&VY_d,sizeFloatVector);CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMalloc (&VX_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

	error = hipMalloc (&Phi_d,sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	LOG (INFO) << "Device Pointer alloziert.";
	
	
    /* 1 */

    /* 1.0 */
    /* default values */
    tstep = DefaultDeltaT;
    for(i=0; i<N; i++) {
        fwallx[i] = 0.0;
        fwally[i] = 0.0;
        fwpointx[i] = 0.0;
        fwpointy[i] = 0.0;
        fpairx[i] = 0.0;
        fpairy[i] = 0.0;
        fsmokex[i] = 0.0;
        fsmokey[i] = 0.0;

        fspx[i] = 0.0;
        fspy[i] = 0.0;
        fsumx[i] = 0.0;
        fsumy[i] = 0.0;
        ftmagsum[i] = 0.0;
        fcolx[i] = 0.0;
        fcoly[i] = 0.0;
    }
    FW_x=0.0;
	

	// temporäre Kraftvektoren auf dem Device mit 0 initialsieren
	
	
	error = hipMemset(fwallx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemset(fwally_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMemset(fcolx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemset(fcoly_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

	error = hipMemset(fsmokex_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemset(fsmokey_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMemset(fwpointx_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemset(fwpointy_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	error = hipMemset(ftmagsum_d, 0, sizeFloatVector); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	// error = hipMemcpy(fwallx_d, fwallx, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(fwally_d, fwally, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	// error = hipMemcpy(fcolx_d, fcolx, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(fcoly_d, fcoly, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	// error = hipMemcpy(fwpointx_d, fwpointx, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	// error = hipMemcpy(fwpointy_d, fwpointy, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	// error = hipMemcpy(ftmagsum_d, ftmagsum, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	LOG (INFO) << "device kraftvektoren initialisiert";
	
	
	// nötige Werte hochkopieren
	
	error = hipMemcpy(D_d, D, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(Injured_d, Injured, N * sizeof(int), hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

	error = hipMemcpy(X_d, X, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(Y_d, Y, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
    error = hipMemcpy(VY_d, VY, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(VX_d, VX, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	error = hipMemcpy(WP_d, WP, NWP * sizeof(wpoint), hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
		
	error = hipMemcpy(para_d, &para_h, sizeof(parameter), hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    	
	LOG (INFO) << "Alle Werte wurden hochkopiert";
	
	
	dim3 dimBlock(32); 
	dim3 dimGrid((para_h.N0 + dimBlock.x - 1) / dimBlock.x);
	
	// LOG(WARNING) << "Block dimensions: " << dimBlock.x << " " << dimBlock.y << " " << dimBlock.z;
	// LOG(WARNING) << "Grid dimensions: " << dimGrid.x << " " << dimGrid.y << " " << dimGrid.z;
	
	calcWallForces<<<dimGrid, dimBlock>>> (fwallx_d, fwally_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, WP_d, VX_d, VY_d, para_d, N, NW); 
	hipDeviceSynchronize();
	// berechnete Werte calcWallForces zurück
	error = hipMemcpy(fwallx, fwallx_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(fwally, fwally_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	// ftmagsum wird im nächsten Kernel nochmal verwendet, darum erst danach zurück
	
	calcWPointForces <<<dimGrid, dimBlock>>> (fwpointx_d, fwpointy_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, WP_d, VX_d, VY_d, para_d, N, NWP);
	hipDeviceSynchronize();
	// berechnete Werte calcWPointForces zurück
	error = hipMemcpy(fwpointx, fwpointx_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(fwpointy, fwpointy_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(ftmagsum, ftmagsum_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


	
    /* 1.1 */
    /* wall force */
    // for(i=0; i<N; i++) {
        // for(iw=0; iw<NW; iw++) {

            // WallParticleRelation(iw,i,&tmpr,&can_see);
            // if((can_see==1)&&(tmpr<=R)) {

                // /* init */
                // tmp_fpsx = tmp_fpsy = 0.0;
                // tmp_fyox = tmp_fyoy = 0.0;
                // tmp_ftax = tmp_ftay = 0.0;

                // /* psychological force */
                // WallPsychForce(iw,i,tmpr,&tmp_fpsx,&tmp_fpsy);
                // /* Young and tangential forces */
                // if(tmpr<=0.5*D[i]) {
                    // WallYoungForce(iw,i,tmpr,&tmp_fyox,&tmp_fyoy);

                    // WallTangForce_FS1(iw,i,tmpr,&tmp_ftax,&tmp_ftay);

                // }
                // /* summing wall forces */
                // if(Injured[i]==0) {
                    // fwallx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    // fwally[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                // } else { /* ie. if Injured[i]=1 */
                    // fwallx[i] += tmp_fyox + tmp_ftax;
                    // fwally[i] += tmp_fyoy + tmp_ftay;
                // }

                // /* sum of magnitude of touching forces */
                // if(InjurySwitch==1) {
                    // ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                // }

                // /* measuring x component of touching force exerted
                // on walls left and right from exit
                 // -- only in demo mode */

                // if((iw==1)||(iw==7)) {
                    // FW_x -= tmp_fyox + tmp_ftax;
                // }
            // }
        // }
    // }



    /* 1.2 */
    /* wpoint force */
    // for(i=0; i<N; i++) {
        // for(iwp=0; iwp<NWP; iwp++) {

            // WPointParticleRelation(iwp,i,&tmpr,&can_see);
            // if((can_see==1)&&(tmpr<=R)) {

                // /* init */
                // tmp_fpsx = tmp_fpsy = 0.0;
                // tmp_fyox = tmp_fyoy = 0.0;
                // tmp_ftax = tmp_ftay = 0.0;

                // /* computing forces */
                // WPointPsychForce(iwp,i,tmpr,&tmp_fpsx,&tmp_fpsy);
                // if(tmpr<=0.5*D[i]) {
                    // WPointYoungForce(iwp,i,tmpr,&tmp_fyox,&tmp_fyoy);

                    // WPointTangForce_FS1(iwp,i,tmpr,&tmp_ftax,&tmp_ftay);

                // }

                // /* summing forces */
                // if(Injured[i]==0) {
                    // fwpointx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                    // fwpointy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                // } else { /* ie. if Injured[i]=1 */
                    // fwpointx[i] += tmp_fyox + tmp_ftax;
                    // fwpointy[i] += tmp_fyoy + tmp_ftay;
                // }

                // /* sum of magnitude of touching forces */
                // if(InjurySwitch==1) {
                    // ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                // }

                // /* measuring x component of touching force exerted
                   // on walls left and right from exit
                   // -- only in demo mode */

                // if((iwp==0)||(iwp==3)) {
                    // FW_x -= tmp_fyox + tmp_ftax;
                // }

            // }
        // }
    // }



    /* 1.3 */
    /* particle-particle forces */
    for(i=0; i<N; i++) {

        j = (int)floor(X[i]*GX/XS) + G * (int)floor(Y[i]*GY/YS);
        for(k=-1; k<=1; k++) {
            for(l=-1; l<=1; l++) {

                mx = j%G+k;
                my = j/G+l;
                if((mx>=0)&&(mx<GX)&&(my>=0)&&(my<GY)) {

                    m = BIndBd[ (mx+GX)%GX + G * (my%GY) ];
                    /* checking each pair of particles only once */
                    while(m>=i) {
                        m = BInd[m];
                    }
                    if(m!=-1) {
                        do {

                            tmprsqr = SQR(X[i]-X[m]) + SQR(Y[i]-Y[m]);
                            if( tmprsqr <= SQR(R) ) {
                                tmpr = sqrt(tmprsqr);

                                /* init */
                                tmp_fpsx = tmp_fpsy = 0.0;
                                tmp_fyox = tmp_fyoy = 0.0;
                                tmp_ftax = tmp_ftay = 0.0;

                                /* pair forces */
                                /* Force(i,m,...) gives the force exerted by m
                                on i, all forces are symmetric now */
                                PP_PsychForce(i,m,tmpr,&tmp_fpsx,&tmp_fpsy);
                                if(tmpr<=0.5*(D[i]+D[m])) {
                                    PP_YoungForce(i,m,tmpr,&tmp_fyox,&tmp_fyoy);
                                    PP_TangForce_FS1(i,m,tmpr,&tmp_ftax,&tmp_ftay);

                                }

                                /* summing forces */
                                if(Injured[i]==0) {
                                    fpairx[i] += tmp_fpsx + tmp_fyox + tmp_ftax;
                                    fpairy[i] += tmp_fpsy + tmp_fyoy + tmp_ftay;
                                } else { /* ie. if Injured[i]=1 */
                                    fpairx[i] += tmp_fyox + tmp_ftax;
                                    fpairy[i] += tmp_fyoy + tmp_ftay;
                                }
                                if(Injured[m]==0) {
                                    fpairx[m] -= tmp_fpsx + tmp_fyox + tmp_ftax;
                                    fpairy[m] -= tmp_fpsy + tmp_fyoy + tmp_ftay;
                                } else { /* ie. if Injured[m]=1 */
                                    fpairx[m] -= tmp_fyox + tmp_ftax;
                                    fpairy[m] -= tmp_fyoy + tmp_ftay;
                                }

                                /* sum of magnitude of touching forces */
                                if(InjurySwitch==1) {
                                    ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                                    ftmagsum[m] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                                }
                            }

                            m = BInd[m];
                            while(m>=i) {
                                m = BInd[m];
                            }

                        } while(m!=-1);
                    }
                }
            }
        }
    }
	// benötigte Werte für calcColumnForces hochkopieren
	
	error = hipMemcpy(ftmagsum_d, ftmagsum, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	calcColumnForces <<<dimGrid, dimBlock>>> (fcolx_d, fcoly_d, ftmagsum_d, D_d, Injured_d, X_d, Y_d, VX_d, VY_d, para_d, N); 
	
	CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); error = hipGetLastError();
	// printf ("Ergebniss von calcColumnForces: %s \n", hipGetErrorString(error));
	hipDeviceSynchronize();
	
	// berechnete Werte calcColumnForces zurück
	error = hipMemcpy(fcolx, fcolx_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(fcoly, fcoly_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); 
	
	
	
	
	// benötigte Werte für calcInjuryForces hochkopieren 
	
	error = hipMemcpy(V0of_d, V0of, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(SimTime_d, SimTime, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(Phi_d, Phi, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
	
	
	calcInjuryForces <<<dimGrid, dimBlock>>> (fsmokex_d, fsmokey_d, VX_d, VY_d, V0of_d, Injured_d,ftmagsum_d, N, UpdNum, SimTime_d, Phi_d, X_d, D_d, para_d);
	hipDeviceSynchronize ();
	
	
	// berechnete Werte calcInjuryForces zurück
	
	error = hipMemcpy(fsmokex, fsmokex_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(fsmokey, fsmokey_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(VX, VX_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(VY, VY_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(V0of, V0of_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(Injured, Injured_d, N * sizeof(int), hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	error = hipMemcpy(ftmagsum, ftmagsum_d, sizeFloatVector, hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error); 
	
	
	
	// Anzahl der Verletzten neu bestimmen
	// sumUp<<<1,1>>> (Injured_d,N, NInjured_d); hipDeviceSynchronize(); error = hipGetLastError ();  printf ("Ergebniss sumUp : %s \n",hipGetErrorString(error));
	
	// error = hipMemcpy(&NInjured, NInjured_d, sizeof(int), hipMemcpyDeviceToHost); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
	
    /* 1.4
     * column
     */
    // switch(ColumnSwitch) {
    // default:
    // case 0: {
            // for(i=0; i<N; i++) {
                // fcolx[i] = fcoly[i] = 0.0;
            // }
            // break;
        // }
    // case 1: {
            // for(i=0; i<N; i++) {
                // tmprsqr = SQR(X[i]-ColumnCenterX)+SQR(Y[i]-ColumnCenterY);
                // if(tmprsqr<=SQR(R)) {
                    // tmpr=sqrt(tmprsqr);

                    // /* init */
                    // tmp_fpsx = tmp_fpsy = 0.0;
                    // tmp_fyox = tmp_fyoy = 0.0;
                    // tmp_ftax = tmp_ftay = 0.0;

                    // /* computing forces */
                    // /* psychological */
                    // f_over_r = A * exp(-(tmpr-0.5*(D[i]+ColumnD))/B) / tmpr;
                    // tmp_fpsx = (X[i]-ColumnCenterX) * f_over_r;
                    // tmp_fpsy = (Y[i]-ColumnCenterY) * f_over_r;
                    // /* touching */
                    // if(tmpr<=0.5*(D[i]+ColumnD)) {
                        // /* Young */
                        // f_over_r = 2.0*C_Young*(0.5*(D[i]+ColumnD)-tmpr) / tmpr;
                        // tmp_fyox = (X[i]-ColumnCenterX) * f_over_r;
                        // tmp_fyoy = (Y[i]-ColumnCenterY) * f_over_r;
                        // /* friction */
                        // rx = X[i]-ColumnCenterX;
                        // ry = Y[i]-ColumnCenterY;
                        // scal_prod_over_rsqr = (ry*VX[i] - rx*VY[i]) / SQR(tmpr);

                        // tmp_ftax =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
                                     // * (   ry * scal_prod_over_rsqr );
                        // tmp_ftay =   -Kappa * (0.5*(D[i]+ColumnD)-tmpr)
                                     // * ( - rx * scal_prod_over_rsqr );


                    // }


                    // /* summing forces */
                    // if(Injured[i]==0) {
                        // fcolx[i] = tmp_fpsx + tmp_fyox + tmp_ftax;
                        // fcoly[i] = tmp_fpsy + tmp_fyoy + tmp_ftay;
                    // } else { /* ie. if Injured[i]==1 */
                        // fcolx[i] = tmp_fyox + tmp_ftax;
                        // fcoly[i] = tmp_fyoy + tmp_ftay;
                    // }


                    // /* sum of magnitude of touching forces */
                    // if(InjurySwitch==1) {
                        // ftmagsum[i] += sqrt(SQR(tmp_fyox)+SQR(tmp_fyoy));
                    // }
                // }
            // }
            // break;
        // }
    // }



    /* 1.5 */
    /* injuries */

    // switch(InjurySwitch) {
    // case 0: {
            // break;
        // }
    // case 1: {

            // /* case: people crushed */
            // for(i=0; i<N; i++) {

                // /* newly injured */
                // if((ftmagsum[i]>FCrush_over_1m*PI*D[i])&&(Injured[i]==0)) {
                    // Injured[i] = 1;
                    // NInjured++;
                    // V0of[i] = 0.0;
                // }
            // }
            // break;
        // }
    // case 2:
    // case 3: {

            // /* case: smoke front */
            // if(SimTime[UpdNum]>=SmokeStartTime) {
                // x_smokefront = (SimTime[UpdNum]-SmokeStartTime)*VSmoke;

                // for(i=0; i<N; i++) {
                    // /* checking position compared to smoke front */
                    // tmpr = X[i] - x_smokefront;

                    // /* center of particle behind smoke front: injured */
                    // if( tmpr < 0.5*D[i] ) {
                        // if(Injured[i]==0) {
                            // Injured[i] = 1;
                            // NInjured++;
                            // V0of[i] = 0.0;
                            // VX[i] = VY[i] = 0.0;
                        // }
                    // }
                    // /* ahead of front but within its interaction range:
                    // trying to escape */
                    // if( (tmpr>=0.5*D[i])&&(tmpr<=R) ) {
                        // tmpf = A_fire*exp(-(tmpr-0.5*D[i])/B_fire);
                        // fsmokex[i] += cos(Phi[i])*tmpf;
                        // fsmokey[i] += sin(Phi[i])*tmpf;
                    // }
                // }
            // }
            // break;
        // }
    // }



    /* 2 */

    /* 2.1 preparing update of the eq. of motion */

    sqrt_fact = sqrt(tstep/DefaultDeltaT);
    for(i=0; i<N; i++) {

        /* self-propelling */
        fspx[i] = 1/Tau * (V0of[i]*cos(Phi[i]) - VX[i]);
        fspy[i] = 1/Tau * (V0of[i]*sin(Phi[i]) - VY[i]);

        /* noise */
        if(GaTh!=0.0) {
            ksi = GaussRand(GaMe, GaTh, GaCM);
            eta = 2.0*PI * rand() / (RAND_MAX+1.0);
        } else {
            ksi=0.0;
            eta=0.0;
        }


        /* sum of forces */
        fsumx[i] =   fspx[i] + fpairx[i] + fwallx[i] + fwpointx[i]
                     + sqrt_fact * ksi * cos(eta);
        fsumy[i] =   fspy[i] + fpairy[i] + fwally[i] + fwpointy[i]
                     + sqrt_fact * ksi * sin(eta);


        /* adding smoke force */
        if((InjurySwitch==2)||(InjurySwitch==3)) {
            fsumx[i] += fsmokex[i];
            fsumy[i] += fsmokey[i];
        }
        /* adding force of column */
        switch(ColumnSwitch) {
        default:
        case 0: {
                break;
            }
        case 1: {
                fsumx[i] += fcolx[i];
                fsumy[i] += fcoly[i];
                break;
            }
        }


        /* time step adjustment for velocity change */
        EulTStep( &tstep, sqrt(SQR(fsumx[i])+SQR(fsumy[i])) );


        /* new velocity */
        if(  (Injured[i]==1)
                &&((InjurySwitch==1)||(InjurySwitch==3))
          ) {
            vxnew[i] = 0.0;
            vynew[i] = 0.0;
        } else {
            vxnew[i] = VX[i] + fsumx[i] * tstep;
            vynew[i] = VY[i] + fsumy[i] * tstep;
        }


        /* checking new velocity */
        vnew = sqrt( SQR(vxnew[i]) + SQR(vynew[i]) );
        if(vnew > Vmax) {
            vxnew[i] = vxnew[i]/vnew * Vmax;
            vynew[i] = vynew[i]/vnew * Vmax;
        }
    }




    /* 3 */
    

    // alte Werte hochkopieren
    error = hipMemcpy(X_d, X, sizeFloatVector, hipMemcpyHostToDevice); CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Y_d, Y, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(VY_d, VY, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(VX_d, VX, sizeFloatVector, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

	
    storeOldValues <<<dimGrid, dimBlock>>> (Xprev_d,X_d,Yprev_d,Y_d, N);
    hipDeviceSynchronize();

	
    calcNewValues <<<dimGrid, dimBlock>>> (X_d,Y_d,VY_d,VX_d,tstep, N);
    hipDeviceSynchronize();

    // neue Werte zurückkopieren
    error = hipMemcpy(Xprev, Xprev_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Yprev, Yprev_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(X, X_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    error = hipMemcpy(Y, Y_d, sizeFloatVector, hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    for(i=0; i<N; i++) {

        /* .1 wurde ersetzt durch storeOldValues */
        // Xprev[i] = X[i];
        // Yprev[i] = Y[i];

        /* .2 wurde erstzt durch calcNewValues*/
        // X[i] += VX[i] * tstep;
        // Y[i] += VY[i] * tstep;

        if((Xprev[i]>RoomXSize)&&(X[i]<=RoomXSize)) {
            NInRoom++;
        }
        if((Xprev[i]<=RoomXSize)&&(X[i]>RoomXSize)) {
            NInRoom--;

        }
    }


    /* .3 and .4 */
    for(i=0; i<N; i++) {

        /* (a) if the particle is on the board, its book-keeping
           arrays are modified only if its block has changed during
           the last update
           (b) if the particle is off-board, it will be removed */


        /* a */
        if(X[i]<XS) {
            j_old =   (int)floor(Xprev[i]*GX/XS)
                      + G*(int)floor(Yprev[i]*GY/YS);
            j_new = (int)floor(X[i]*GX/XS) + G*(int)floor(Y[i]*GY/YS);
            if( j_new != j_old ) {

                /* deleting particle i from its old block */
                j = j_old;
                if(BIndBd[j]==i) {
                    BIndBd[j] = BInd[i];
                } else {
                    j = BIndBd[j];
                    while(BInd[j]!=i) {
                        j = BInd[j];
                    }
                    BInd[j] = BInd[i];
                }


                /* inserting particle i into its new block */
                j = j_new;
                if(BIndBd[j]==-1) {
                    BIndBd[j] = i;
                    BInd[i] = -1;
                } else {
                    j = BIndBd[j];
                    while(BInd[j]!=-1) {
                        j = BInd[j];
                    }
                    BInd[j] = i;
                    BInd[i] = -1;
                }
            }
        } else {
            RemoveParticle( &N, i );
            i--;
        }
    }



    /* 4 */

    /* 4.1 */
    E[UpdNum+1] = 0.0;
    for(i=0; i<N; i++) {
        E[UpdNum+1] += VX[i] * cos(Phi[i]) + VY[i] * sin(Phi[i]);
    }
    if(N>0) {
        E[UpdNum+1] /= N;
    }


    /* 4.2 */
    for(i=0; i<N; i++) {
        VX[i] = vxnew[i];
        VY[i] = vynew[i];
        V[i] = sqrt(SQR(VX[i])+SQR(VY[i]));
        Vdir[i] = atan2(VY[i],VX[i]);
        Phi[i] = DirectionOfExit( i );
    }


    /* 4.3 */
    SimTime[UpdNum+1] = SimTime[UpdNum] + tstep;
    UpdNum++;


    /*
    if(NInjured>0){
    fprintf(stdout,"t[%d]=%g\n",UpdNum,SimTime[UpdNum]);
    fflush(stdout);
    }
    */


    /* 5 */
    free_vector(fwallx,0,allocN-1);
    free_vector(fwally,0,allocN-1);
    free_vector(fwpointx,0,allocN-1);
    free_vector(fwpointy,0,allocN-1);
    free_vector(fpairx,0,allocN-1);
    free_vector(fpairy,0,allocN-1);
    free_vector(fsmokex,0,allocN-1);
    free_vector(fsmokey,0,allocN-1);

    free_vector(fspx,0,allocN-1);
    free_vector(fspy,0,allocN-1);
    free_vector(fsumx,0,allocN-1);
    free_vector(fsumy,0,allocN-1);
    free_vector(vxnew,0,allocN-1);
    free_vector(vynew,0,allocN-1);
    free_vector(ftmagsum,0,allocN-1);
    free_vector(fcolx,0,allocN-1);
    free_vector(fcoly,0,allocN-1);


    /* 6 */
}
